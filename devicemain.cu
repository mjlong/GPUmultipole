#include "CPUComplex.h"
#include "CComplex.h"
#include "multipole_data.h"
#include "multipole.h"
#include "simulation.h"

/*
  To compile host and device codes separately, 
  this "main" file works as interface 
  allocating device memory, transfering data and partitioning computation sources
*/


void printdevice();

void anyvalue(struct multipoledata data, unsigned setgridx, unsigned setblockx, unsigned num_src, unsigned devstep){
  unsigned gridx, blockx, gridsize;
  float timems = 0.0;
  unsigned *cnt, *blockcnt;
  unsigned int active,i;
  double *hostarray, *devicearray;
  MemStruct HostMem, DeviceMem;
  hipEvent_t start, stop;
  gpuErrchk(hipEventCreate(&start));
  gpuErrchk(hipEventCreate(&stop));
  // printdevice();
  gridx = setgridx;
  blockx = setblockx;
  dim3 dimBlock(gridx, 1);
  dim3 dimGrid(blockx, 1, 1);
  gridsize = gridx*blockx;
  gpuErrchk(hipMalloc((void**)&devicearray, 4*gridsize*sizeof(double)));
  gpuErrchk(hipMalloc((void**)&(DeviceMem.nInfo), gridsize*sizeof(NeutronInfoStruct)));
  gpuErrchk(hipMalloc((void**)&(DeviceMem.thread_active), gridsize*sizeof(unsigned int)));
  HostMem.thread_active = (unsigned int *)malloc(gridsize*sizeof(unsigned int));
  gpuErrchk(hipMalloc((void**)&(DeviceMem.num_terminated_neutrons), sizeof(unsigned int)));
  HostMem.num_terminated_neutrons = (unsigned int *)malloc(sizeof(unsigned int));
  HostMem.num_terminated_neutrons[0] = 0u;
  gpuErrchk(hipMemcpy(DeviceMem.num_terminated_neutrons, HostMem.num_terminated_neutrons, sizeof(unsigned int), hipMemcpyHostToDevice));
  gpuErrchk(hipMalloc((void**)&(DeviceMem.tally), gridsize*sizeof(TallyStruct)));
  gpuErrchk(hipMalloc((void**)&(blockcnt), gridx*sizeof(unsigned int)));
  hostarray = (double*)malloc(4*gridsize*sizeof(double));
  cnt      = (unsigned*)malloc(gridx*sizeof(unsigned));

  multipole U238(data); //host multipoledata to device
  initialize<<<dimBlock, dimGrid>>>(DeviceMem, 2000.0);//1.95093e4);
  //  hipDeviceSynchronize();
  /*
    Note: shared memory size is in unit of Bybe
    And the address can be referred in form of p = pshared + offset
  */
  gpuErrchk(hipEventRecord(start, 0));

  active = 1u;

  while (active){
    history<<<dimBlock, dimGrid>>>(U238, DeviceMem, num_src, devstep);
    gpuErrchk(hipMemcpy(HostMem.thread_active, DeviceMem.thread_active, gridsize*sizeof(unsigned int), hipMemcpyDeviceToHost));
    active = 0u;
    for (i = 0; i < blockx; i++){
      active += HostMem.thread_active[i];
    }
  }
  remaining<<<dimBlock, dimGrid>>>(U238, devicearray, DeviceMem);

  gpuErrchk(hipEventRecord(stop, 0));
  gpuErrchk(hipEventSynchronize(stop));
  gpuErrchk(hipEventElapsedTime(&timems, start, stop));

  printf("time elapsed:%3.1f ms\n", timems);
 
  gpuErrchk(hipMemcpy(hostarray, devicearray, 4*gridsize*sizeof(double), hipMemcpyDeviceToHost));

  unsigned ints=0, sharedmem;
  ints = blockx;
  sharedmem = ints*sizeof(int);
  statistics<<<dimBlock, dimGrid, sharedmem>>>(DeviceMem.tally, blockcnt);
  gpuErrchk(hipMemcpy(cnt, blockcnt, gridx*sizeof(unsigned), hipMemcpyDeviceToHost));

  for(int i=0;i<gridsize;i++){
    printf("%.15e %.15e %.15e %.15e",
	   hostarray[4*i],
	   hostarray[4*i+1],
	   hostarray[4*i+2],
	   hostarray[4*i+3]);
    if(hostarray[4*i]<0)
      printf("error-:%d \n",i);
    else{
      if(hostarray[4*i]>=2000.0)
	printf("error+:%d \n",i);
      else
	printf("\n");
    }
  }

  unsigned sum = 0;
  for (int i=0;i<gridx;i++){
    printf("%4d\n",cnt[i]);
    sum += cnt[i];
  }
  printf("time elapsed:%g mus\n", timems*1000/sum);

  FILE *fp=NULL;
  fp = fopen("timelog","a+");
  gpuErrchk(hipMemcpy(HostMem.num_terminated_neutrons, 
		       DeviceMem.num_terminated_neutrons, 
		       sizeof(unsigned int), 
		       hipMemcpyDeviceToHost));
  fprintf(fp,"%-4d,%-4d,%-.6f,%-5.2f M,%-4d,%-2d M\n", gridx, blockx,timems*1000/sum, *HostMem.num_terminated_neutrons/1000000.0f, devstep, num_src/1000000);
  fclose(fp);
  //hipEventRecord(stop, 0);
  //hipEventSynchronize(stop);
  //hipEventElapsedTime(&timems, start, stop);

  gpuErrchk(hipEventDestroy(start));
  gpuErrchk(hipEventDestroy(stop));

  gpuErrchk(hipFree(devicearray));
  gpuErrchk(hipFree(DeviceMem.nInfo));
  gpuErrchk(hipFree(DeviceMem.thread_active));
  gpuErrchk(hipFree(DeviceMem.tally));
  U238.release_pointer();

  free(hostarray);
  free(cnt);
  free(HostMem.thread_active);
  free(HostMem.num_terminated_neutrons);
  return;
}


void printdevice(){
  hipDeviceProp_t prop; 
  int count;
  hipGetDeviceCount(&count);
  for (int i=0; i<count; i++){
    hipGetDeviceProperties( &prop, i );
    printf( "   --- General Information for device %d ---\n", i );
    printf( "Name:  %s\n", prop.name );
    printf( "Compute capability:  %d.%d\n", prop.major, prop.minor );
    printf( "Clock rate:  %d\n", prop.clockRate );
    printf( "Device copy overlap:  " );
    if (prop.deviceOverlap)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n");
    printf( "Kernel execution timeout :  " );
    if (prop.kernelExecTimeoutEnabled)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n" );
    
    printf( "   --- Memory Information for device %d ---\n", i );
    printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
    printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
    printf( "Max mem pitch:  %ld\n", prop.memPitch );
    printf( "Texture Alignment:  %ld\n", prop.textureAlignment );
    
    printf( "   --- MP Information for device %d ---\n", i );
    printf( "Multiprocessor count:  %d\n",
	    prop.multiProcessorCount );
    printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
    printf( "Registers per mp:  %d\n", prop.regsPerBlock );
    printf( "Threads in warp:  %d\n", prop.warpSize );
    printf( "Max threads per block:  %d\n",
	    prop.maxThreadsPerBlock );
    printf( "Max thread dimensions:  (%d, %d, %d)\n",
	    prop.maxThreadsDim[0], prop.maxThreadsDim[1],
	    prop.maxThreadsDim[2] );
    printf( "Max grid dimensions:  (%d, %d, %d)\n",
	    prop.maxGridSize[0], prop.maxGridSize[1],
	    prop.maxGridSize[2] );
    printf( "\n" );
  }


}
