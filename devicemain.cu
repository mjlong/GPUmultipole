#include "CPUComplex.h"
#include "CComplex.h"
#include "multipole_data.h"
#include "multipole.h"
#include "simulation.h"

/*
  To compile host and device codes separately, 
  this "main" file works as interface 
  allocating device memory, transfering data and partitioning computation sources
*/


void printdevice();

void anyvalue(struct multipoledata data, int setgridx, int setblockx){
  unsigned gridx, blockx, gridsize;
  unsigned ints=0, doubles=0, sharedmem;
  float timems = 0.0;
  unsigned *cnt;
  double *hostarray, *devicearray;
  struct neutronInfo Info;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  // printdevice();
  gridx = setgridx;
  blockx = setblockx;
  dim3 dimBlock(gridx, 1);
  dim3 dimGrid(blockx, 1, 1);
  gridsize = gridx*blockx;
  hipMalloc((void**)&devicearray, 4*gridsize*sizeof(double));
  hipMalloc((void**)&(Info.rndState), gridsize*sizeof(hiprandState));
  hipMalloc((void**)&(Info.energy), gridsize*sizeof(double));
  hipMalloc((void**)&(Info.tally), gridx*sizeof(unsigned));
  hostarray = (double*)malloc(4*gridsize*sizeof(double));
  cnt      = (unsigned*)malloc(gridx*sizeof(unsigned));

  multipole U238(data); //host multipoledata to device
  initialize<<<dimBlock, dimGrid>>>(Info, 2000.0);//1.95093e4);
  //  hipDeviceSynchronize();
  /*
    Note: shared memory size is in unit of Bybe
    And the address can be referred in form of p = pshared + offset
  */
  ints = blockx;
  sharedmem = doubles*sizeof(double)+ints*sizeof(int);
  hipEventRecord(start, 0);
  history<<<dimBlock, dimGrid, sharedmem>>>(U238, devicearray, Info);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&timems, start, stop);

  printf("time elapsed:%3.1f ms\n", timems);
 
  hipMemcpy(hostarray, devicearray, 4*gridsize*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(cnt, Info.tally, gridx*sizeof(unsigned), hipMemcpyDeviceToHost);

  for(int i=0;i<gridsize;i++){
    printf("%.15e %.15e %.15e %.15e\n",
	   hostarray[4*i],
	   hostarray[4*i+1],
	   hostarray[4*i+2],
	   hostarray[4*i+3]);
  }

  unsigned sum = 0;
  for (int i=0;i<gridx;i++){
    //printf("%4d\n",cnt[i]);
    sum += cnt[i];
  }
  printf("time elapsed:%g mus\n", timems*1000/sum);

  FILE *fp=NULL;
  fp = fopen("timelog","a+");
  fprintf(fp,"%3d,%3d,%g    \n", gridx, blockx, timems*1000/sum);
  fclose(fp);
  //hipEventRecord(stop, 0);
  //hipEventSynchronize(stop);
  //hipEventElapsedTime(&timems, start, stop);

  //hipEventDestroy(start);
  //hipEventDestroy(stop);

  hipFree(devicearray);
  hipFree(Info.energy);
  hipFree(Info.tally);
  hipFree(Info.rndState);

  free(hostarray);
  free(cnt);
  return;
}


void printdevice(){
  hipDeviceProp_t prop; 
  int count;
  hipGetDeviceCount(&count);
  for (int i=0; i<count; i++){
    hipGetDeviceProperties( &prop, i );
    printf( "   --- General Information for device %d ---\n", i );
    printf( "Name:  %s\n", prop.name );
    printf( "Compute capability:  %d.%d\n", prop.major, prop.minor );
    printf( "Clock rate:  %d\n", prop.clockRate );
    printf( "Device copy overlap:  " );
    if (prop.deviceOverlap)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n");
    printf( "Kernel execution timeout :  " );
    if (prop.kernelExecTimeoutEnabled)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n" );
    
    printf( "   --- Memory Information for device %d ---\n", i );
    printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
    printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
    printf( "Max mem pitch:  %ld\n", prop.memPitch );
    printf( "Texture Alignment:  %ld\n", prop.textureAlignment );
    
    printf( "   --- MP Information for device %d ---\n", i );
    printf( "Multiprocessor count:  %d\n",
	    prop.multiProcessorCount );
    printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
    printf( "Registers per mp:  %d\n", prop.regsPerBlock );
    printf( "Threads in warp:  %d\n", prop.warpSize );
    printf( "Max threads per block:  %d\n",
	    prop.maxThreadsPerBlock );
    printf( "Max thread dimensions:  (%d, %d, %d)\n",
	    prop.maxThreadsDim[0], prop.maxThreadsDim[1],
	    prop.maxThreadsDim[2] );
    printf( "Max grid dimensions:  (%d, %d, %d)\n",
	    prop.maxGridSize[0], prop.maxGridSize[1],
	    prop.maxGridSize[2] );
    printf( "\n" );
  }


}
