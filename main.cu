#include "CPUComplex.h"
#include "CComplex.h"
#include "multipole_data.h"
#include "multipole.h"
#include "simulation.h"

/*
  To compile host and device codes separately, 
  this "main" file works as interface 
  allocating device memory, transfering data and partitioning computation sources
*/


void printdevice();

void anyvalue(struct multipoledata data, int *value, double *d1, double *d2){
  unsigned gridx, gridy, blockx, blocky, blockz, blocknum, gridsize;
  unsigned ints=0, floats=0, doubles=0, sharedmem;
  double *hostarray, *devicearray, *tally;
  struct neutronInfo Info;
  //  printdevice();
  gridx = 4;
  gridy = 1;
  blockx = 16;
  blocky = 1;
  blockz = 1;
  dim3 dimBlock(gridx, gridy);
  dim3 dimGrid(blockx, blocky, blockz);
  blocknum = gridx*gridy; 
  gridsize = gridx*gridy*blockx*blocky*blockz;
  hipMalloc((void**)&devicearray, 7*gridsize*sizeof(double));
  hipMalloc((void**)&(Info.rndState), gridsize*sizeof(hiprandState));
  hipMalloc((void**)&(Info.energy), gridsize*sizeof(double));
  hipMalloc((void**)&(Info.tally), blocknum*sizeof(double));
  hostarray = (double*)malloc(7*gridsize*sizeof(double));
  tally     = (double*)malloc(blocknum*sizeof(double));

  multipole U238(data); //host multipoledata to device


  initialize<<<dimBlock, dimGrid>>>(Info, 1000.0);
  //  hipDeviceSynchronize();

  /*
    Note: shared memory size is in unit of Bybe
    And the address can be referred in form of p = pshared + offset
  */
  doubles = blockx*blocky*blockz;
  sharedmem = doubles*sizeof(double)+floats*sizeof(float)+ints*sizeof(int);
  history<<<dimBlock, dimGrid, sharedmem>>>(U238, devicearray, Info);

  hipMemcpy(hostarray, devicearray, 7*gridsize*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(tally, Info.tally, blocknum*sizeof(double), hipMemcpyDeviceToHost);

  for(int i=0;i<gridsize;i++){
    printf("%8.4f %8.5e %8.5e %8.5e %8.5e %8.5e %8.5e\n",
	   hostarray[7*i],
	   hostarray[7*i+1],
	   hostarray[7*i+2],
	   hostarray[7*i+3],
	   hostarray[7*i+4],
	   hostarray[7*i+5],
	   hostarray[7*i+6]);
  }
  for (int i=0;i<blocknum;i++)
    printf("%2.1f\n",tally[i]);


  return;
}


void printdevice(){
  hipDeviceProp_t prop; 
  int count;
  hipGetDeviceCount(&count);
  for (int i=0; i<count; i++){
    hipGetDeviceProperties( &prop, i );
    printf( "   --- General Information for device %d ---\n", i );
    printf( "Name:  %s\n", prop.name );
    printf( "Compute capability:  %d.%d\n", prop.major, prop.minor );
    printf( "Clock rate:  %d\n", prop.clockRate );
    printf( "Device copy overlap:  " );
    if (prop.deviceOverlap)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n");
    printf( "Kernel execution timeout :  " );
    if (prop.kernelExecTimeoutEnabled)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n" );
    
    printf( "   --- Memory Information for device %d ---\n", i );
    printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
    printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
    printf( "Max mem pitch:  %ld\n", prop.memPitch );
    printf( "Texture Alignment:  %ld\n", prop.textureAlignment );
    
    printf( "   --- MP Information for device %d ---\n", i );
    printf( "Multiprocessor count:  %d\n",
	    prop.multiProcessorCount );
    printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
    printf( "Registers per mp:  %d\n", prop.regsPerBlock );
    printf( "Threads in warp:  %d\n", prop.warpSize );
    printf( "Max threads per block:  %d\n",
	    prop.maxThreadsPerBlock );
    printf( "Max thread dimensions:  (%d, %d, %d)\n",
	    prop.maxThreadsDim[0], prop.maxThreadsDim[1],
	    prop.maxThreadsDim[2] );
    printf( "Max grid dimensions:  (%d, %d, %d)\n",
	    prop.maxGridSize[0], prop.maxGridSize[1],
	    prop.maxGridSize[2] );
    printf( "\n" );
  }


}
