#include "hip/hip_runtime.h"
#include "CPUComplex.h"
#include "CComplex.h"
#include "multipole_data.h"
#include "multipole.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__ void history(hiprandState *rndStates, unsigned *);



void anyvalue(struct multipoledata data, int *value, double *d1, double *d2){
  hiprandState *rndStates;
  unsigned gridx, gridy, blockx, blocky, blockz, gridsize;
  unsigned *hostarray, *devicearray;
  gridx = 3;
  gridy = 2;
  blockx = 4;
  blocky = 5;
  blockz = 6;
  dim3 dimBlock(gridx, gridy);
  dim3 dimGrid(blockx, blocky, blockz);
  gridsize = gridx*gridy*blockx*blocky*blockz;
  hipMalloc((void**)&rndStates, gridsize*sizeof(hiprandState));
  hipMalloc((void**)&devicearray, gridsize*sizeof(unsigned int));
  hostarray = (unsigned*)malloc(gridsize*sizeof(unsigned int));
  multipole U238(data); //host multipoledata to device
  history<<<dimBlock, dimGrid>>>(rndStates, devicearray);
  hipMemcpy(hostarray, devicearray, gridsize*sizeof(unsigned int), hipMemcpyDeviceToHost);
  for(int i=0;i<gridsize;i++)
    printf("%5d  %5d\n",i,hostarray[i]);
  return;
}


__global__ void history(hiprandState *rndStates, unsigned *devicearray){
  //TODO:this is one scheme to match threads to 1D array, 
  //try others when real simulation structure becomes clear
  int id = 
    (blockDim.x*blockDim.y*blockDim.z)*
    (blockIdx.y*gridDim.x+blockIdx.x)+
    (blockDim.x*blockDim.y)*threadIdx.z+
    blockDim.x*threadIdx.y+
    threadIdx.x;
  bool live=true;
  double energy = 1.0;
  double rnd = 0.0;
  while(live){
    energy = energy * rnd;
    live = (energy>1.0e-4);
  }
  devicearray[id] = id;
}
