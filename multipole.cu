#include "hip/hip_runtime.h"
#include "multipole.h"
multipole::multipole(struct multipoledata data){
  size_t size;
  /*
    allocate and assign integers
  */
  size = sizeof(int);
  hipMalloc((void**)&dev_integers, 6*size);
  hipMemcpy(dev_integers+MODE,    &(data.mode), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_integers+WINDOWS, &(data.windows), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_integers+FITORDER, &(data.fitorder), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_integers+NUML, &(data.numL), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_integers+FISSION, &(data.fissionable), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_integers+LENGTH, &(data.length), size, hipMemcpyHostToDevice);

  /*
    allocate and assign doubles
  */
  size = sizeof(double);
  hipMalloc((void**)&dev_doubles,  4*size);
  hipMemcpy(dev_doubles+STARTE, &(data.startE), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_doubles+ENDE,   &(data.endE), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_doubles+SPACING,&(data.spacing), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_doubles+SQRTAWR, &(data.sqrtAWR), size, hipMemcpyHostToDevice);

  /*
    allocate and assign arrays
  */
  size = data.length*(MP_RF+data.fissionable)*2*sizeof(double);
  hipMalloc((void**)&mpdata, size);
  hipMemcpy(mpdata, data.mpdata, size, hipMemcpyHostToDevice);

  size = data.length*sizeof(unsigned);
  hipMalloc((void**)&l_value, size);
  hipMemcpy(l_value, data.l_value, size, hipMemcpyHostToDevice);

  size = data.numL*sizeof(double);
  hipMalloc((void**)&pseudo_rho, size);
  hipMemcpy(pseudo_rho, data.pseudo_rho, size, hipMemcpyHostToDevice);


  size = data.windows*sizeof(int);
  hipMalloc((void**)&w_start, size);
  hipMemcpy(w_start, data.w_start, size, hipMemcpyHostToDevice);
  hipMalloc((void**)&w_end, size);
  hipMemcpy(w_end, data.w_end, size, hipMemcpyHostToDevice);

  size = (FIT_F+data.fissionable)*(data.fitorder+1)*data.windows*sizeof(double);
  hipMalloc((void**)&fit, size);
  hipMemcpy(fit, data.fit, size, hipMemcpyHostToDevice);


}


multipole::~multipole(){
  hipFree(dev_integers);
  hipFree(dev_doubles);
  hipFree(mpdata);
  hipFree(l_value);
  hipFree(pseudo_rho);
  hipFree(w_start);
  hipFree(w_end);
  hipFree(fit);
}
__device__  void multipole::xs_eval_fast(double E, double sqrtKT, 
			double &sigT, double &sigA, double &sigF){
  int    iP, iC, iW, startW, endW;
  double *twophi;
  double sqrtE = sqrt(E);
  double power, DOPP, DOPP_ECOEF;
  CComplex w_val;
  /*
  if(1==mode)
    iW = (int)(sqrtE - sqrt(startE))/spacing;
  else if(2==mode)
    iW = (int)(log(E) - log(startE))/spacing;
  else
    iW = (int)( E - startE )/spacing;
  startW = w_start[iW];
  endW   = w_end[iW];
  if(startW <= endW)
    fill_factors(sqrtE,twophi);
  sigT = 0.0;
  sigA = 0.0;
  sigF = 0.0;
  //polynomial fitting
  for (iC=0;iC<=fitorder;iC++){
    power = pow(E,iC);
    sigT += fit[findex(FIT_T, iC, iW)]*power;
    sigA += fit[findex(FIT_A, iC, iW)]*power;
    if(MP_FISS == fissionable)
      sigF += fit[findex(FIT_F, iC, iW)]*power;
  }
  //Faddeeva evaluation in advance
  DOPP = sqrtAWR/sqrtKT;
  DOPP_ECOEF = DOPP/sqrt(PI);
  for(iP=startW;iP<=endW;iP++){
    Z_array[iP-startW] = (sqrtE - mpdata[pindex(MP_EA,iP)])*DOPP;
    W_array[iP-startW] = Faddeeva::w(Z_array[iP-startW])*DOPP_ECOEF;
  }

  //evaluating
  for(iP=startW;iP<=endW;iP++){
    sigT += real(mpdata[pindex(MP_RT,iP)]*sigT_factor[l_value[iP]-1]*W_array[iP-startW]);
    sigA += real(mpdata[pindex(MP_RA,iP)]*W_array[iP-startW]);
    if(MP_FISS == fissionable)
      sigF += real(mpdata[pindex(MP_RF,iP)]*W_array[iP-startW]);
  }
  */

}

__device__  void multipole::xs_eval_fast(double E,  
			double &sigT, double &sigA, double &sigF){
  int    iP, iC, iW, startW, endW;
  double *twophi;
  double sqrtE = sqrt(E);
  double power;
  CComplex PSIIKI, CDUM1, w_val;
  /*
  twophi = (double*)malloc(sizeof(double)*numL);
 
  if(1==mode)
    iW = (int)(sqrtE - sqrt(startE))/spacing;
  else if(2==mode)
    iW = (int)(log(E) - log(startE))/spacing;
  else
    iW = (int)( E - startE )/spacing;
  startW = w_start[iW];
  endW   = w_end[iW];
  if(startW <= endW)
    fill_factors(sqrtE,twophi);
  sigT = 0.0;
  sigA = 0.0;
  sigF = 0.0;
  //polynomial fitting
  for (iC=0;iC<=fitorder;iC++){
    power = pow(E,iC);
    sigT += fit[findex(FIT_T, iC, iW)]*power;
    sigA += fit[findex(FIT_A, iC, iW)]*power;
    if(MP_FISS == fissionable)
      sigF += fit[findex(FIT_F, iC, iW)]*power;
  }
  //Faddeeva evaluation in advance

  //evaluating
  for(iP=startW;iP<=endW;iP++){
    PSIIKI = -ONEI/(mpdata[pindex(MP_EA,iP)] - sqrtE);
    CDUM1  = PSIIKI / E;
    sigT += real(mpdata[pindex(MP_RT,iP)]*CDUM1*sigT_factor[l_value[iP]-1]);
    sigA += real(mpdata[pindex(MP_RA,iP)]*CDUM1);
    if(MP_FISS == fissionable)
      sigF += real(mpdata[pindex(MP_RF,iP)]*CDUM1);
  }
  free(twophi);
  */
}


__host__ __device__  int multipole::findex(int type, int iC, int iW){
  //  return windows*(fitorder+1)*type+windows*iC+iW;
  return 0;
}

__host__ __device__  int multipole::pindex(int type, int iP){
  //  return length*type + iP;
  return 0;
}

//TODO: here just continue the initilization scheme, it deserves trying make some values shared
__device__ void multipole::fill_factors(double sqrtE, double *twophi, CComplex *sigT_factor){
  int iL;
  double arg;

  for(iL = 0; iL<dev_integers[NUML]; iL++){
    twophi[iL] = pseudo_rho[iL] * sqrtE; 
    if(2==iL)
      twophi[iL] -= atan(twophi[iL]);
    else if(3==iL){
      arg = 3.0*twophi[iL] / (3.0 - twophi[iL]*twophi[iL]);
      twophi[iL] -= atan(arg);
    }
    else if(4==iL){
      arg = twophi[iL]*(15.0 - twophi[iL]*twophi[iL])/(15.0 - 6.0*twophi[iL]*twophi[iL]);
      twophi[iL] -= atan(arg);
    }
    twophi[iL] *= 2.0;
    sigT_factor[iL] = CComplex(cos(twophi[iL]), -sin(twophi[iL]));
  }

}
