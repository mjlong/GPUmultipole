#include "hip/hip_runtime.h"
#include "multipole.h"
multipole::multipole(struct multipoledata data){
  size_t size;
  /*
    allocate and assign integers
  */
  size = sizeof(int);
  hipMalloc((void**)&dev_integers, 4*size);
  hipMemcpy(dev_integers+MODE,    &(data.mode), size, hipMemcpyHostToDevice);
  //hipMemcpy(dev_integers+WINDOWS, &(data.windows), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_integers+FITORDER, &(data.fitorder), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_integers+NUML, &(data.numL), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_integers+FISSIONABLE, &(data.fissionable), size, hipMemcpyHostToDevice);
  //hipMemcpy(dev_integers+LENGTH, &(data.length), size, hipMemcpyHostToDevice);

  /*
    allocate and assign doubles
  */
  size = sizeof(double);
  hipMalloc((void**)&dev_doubles,  3*size);
  hipMemcpy(dev_doubles+STARTE, &(data.startE), size, hipMemcpyHostToDevice);
  //hipMemcpy(dev_doubles+ENDE,   &(data.endE), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_doubles+SPACING,&(data.spacing), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_doubles+SQRTAWR, &(data.sqrtAWR), size, hipMemcpyHostToDevice);

  /*
    allocate and assign arrays
  */
  size = data.length*(MP_RF+data.fissionable)*2*sizeof(double);
  hipMalloc((void**)&mpdata, size);
  hipMemcpy(mpdata, data.mpdata, size, hipMemcpyHostToDevice);

  size = data.length*sizeof(unsigned);
  hipMalloc((void**)&l_value, size);
  hipMemcpy(l_value, data.l_value, size, hipMemcpyHostToDevice);

  size = data.numL*sizeof(double);
  hipMalloc((void**)&pseudo_rho, size);
  hipMemcpy(pseudo_rho, data.pseudo_rho, size, hipMemcpyHostToDevice);


  size = data.windows*sizeof(int);
  hipMalloc((void**)&w_start, size);
  hipMemcpy(w_start, data.w_start, size, hipMemcpyHostToDevice);
  hipMalloc((void**)&w_end, size);
  hipMemcpy(w_end, data.w_end, size, hipMemcpyHostToDevice);

  size = (FIT_F+data.fissionable)*(data.fitorder+1)*data.windows*sizeof(double);
  hipMalloc((void**)&fit, size);
  hipMemcpy(fit, data.fit, size, hipMemcpyHostToDevice);
}


multipole::~multipole(){
  hipFree(dev_integers);
  hipFree(dev_doubles);
  hipFree(mpdata);
  hipFree(l_value);
  hipFree(pseudo_rho);
  hipFree(w_start);
  hipFree(w_end);
  hipFree(fit);
}
__device__  void multipole::xs_eval_fast(double E, double sqrtKT, 
					 double &sigT, double &sigA, double &sigF, 
					 CComplex* sigT_factor, unsigned blocksize){
  /* Copy variables to local memory for efficiency */ 
  int mode        = dev_integers[MODE];
  int    iP, iC, iW, startW, endW;
  double spacing = dev_doubles[SPACING];
  double startE  = dev_doubles[STARTE];
  double sqrtE = sqrt(E);
  if(1==mode)
    iW = (int)((sqrtE - sqrt(startE))/spacing);
  else if(2==mode)
    iW = (int)((log(E) - log(startE))/spacing);
  else
    iW = (int)(( E - startE )/spacing);
  int fitorder    = dev_integers[FITORDER];
  int numL        = dev_integers[NUML];
  int fissionable = dev_integers[FISSIONABLE];
  //int length      = dev_integers[LENGTH];
  //int windows     = dev_integers[WINDOWS];
  //TODO:if length,windows are really not needed, remove them from dev_integers[] array

  //TODO:I've not found wat to allocate for a thread
  // 4 = maximum numL, consistent with max 3==iL in fill_factors()
  //double twophi[4];
  //CComplex sigT_factor[4];

  double sqrtAWR = dev_doubles[SQRTAWR];

  
  double power, DOPP, DOPP_ECOEF;
  CComplex w_val;

  startW = w_start[iW];
  endW   = w_end[iW];

  if(startW <= endW)
    fill_factors(sqrtE,numL,sigT_factor, blocksize);
  sigT = 0.0;
  sigA = 0.0;
  sigF = 0.0;
  //polynomial fitting
  for (iC=0;iC<=fitorder;iC++){
    power = pow(E,iC*0.5-1.0);
    sigT += fit[findex(iW,iC,FIT_T,fitorder+1,2+fissionable)]*power;
    sigA += fit[findex(iW,iC,FIT_A,fitorder+1,2+fissionable)]*power;
    if(MP_FISS == fissionable)
      sigF += fit[findex(iW,iC,FIT_F,fitorder+1,2+fissionable)]*power;
  }

  DOPP = sqrtAWR/sqrtKT;
  DOPP_ECOEF = DOPP/E*sqrt(PI);

  for(iP=startW;iP<=endW;iP++){
    w_val = Faddeeva::w((sqrtE - mpdata[pindex(iP-1,MP_EA)])*DOPP)*DOPP_ECOEF;
    sigT += real(mpdata[pindex(iP-1,MP_RT)]*sigT_factor[(l_value[iP-1]-1)*blocksize]*w_val);	    
    sigA += real(mpdata[pindex(iP-1,MP_RA)]*w_val);                              
    if(MP_FISS == fissionable)
      sigF += real(mpdata[pindex(iP-1,MP_RF)]*w_val);
  }

}

__device__  void multipole::xs_eval_fast(double E,  
					 double &sigT, double &sigA, double &sigF,
					 CComplex *sigT_factor, unsigned blocksize){
  /* Copy variables to local memory for efficiency */ 
  int mode        = dev_integers[MODE];
  int fitorder    = dev_integers[FITORDER];
  int fissionable = dev_integers[FISSIONABLE];
  //int length      = dev_integers[LENGTH];
  //int windows     = dev_integers[WINDOWS];
  int numL        = dev_integers[NUML];
  //size_t size;
  double spacing = dev_doubles[SPACING];
  double startE  = dev_doubles[STARTE];
  
  int    iP, iC, iW, startW, endW;


  double sqrtE = sqrt(E);
  double power;
  CComplex PSIIKI, CDUM1, w_val;

  if(1==mode)
    iW = (int)((sqrtE - sqrt(startE))/spacing);
  else if(2==mode)
    iW = (int)((log(E) - log(startE))/spacing);
  else
    iW = (int)(( E - startE )/spacing);
  startW = w_start[iW];
  endW   = w_end[iW];
  if(startW <= endW)
    fill_factors(sqrtE,numL,sigT_factor, blocksize);
  sigT = 0.0;
  sigA = 0.0;
  sigF = 0.0;
  //polynomial fitting
  for (iC=0;iC<=fitorder;iC++){
    power = pow(E,iC);
    sigT += fit[findex(iW,iC,FIT_T,fitorder+1,2+fissionable)]*power;
    sigA += fit[findex(iW,iC,FIT_A,fitorder+1,2+fissionable)]*power;
    if(MP_FISS == fissionable)
      sigF += fit[findex(iW,iC,FIT_F,fitorder+1,2+fissionable)]*power;
  }
  //Faddeeva evaluation in advance

  //evaluating
  for(iP=startW;iP<=endW;iP++){
    PSIIKI = -ONEI/(mpdata[pindex(iP-1,MP_EA)] - sqrtE);
    CDUM1  = PSIIKI / E;
    sigT += real(mpdata[pindex(iP-1,MP_RT)]*CDUM1*sigT_factor[(l_value[iP-1]-1)*blocksize]);
    sigA += real(mpdata[pindex(iP-1,MP_RA)]*CDUM1);
    if(MP_FISS == fissionable)
      sigF += real(mpdata[pindex(iP-1,MP_RF)]*CDUM1);
  }
  
}



__host__ __device__ int multipole::findex(int iW, int iC, int type, int orders, int types){
  return iW*orders*types + iC*types + type; 
}

__host__ __device__ int multipole::pindex(int iP, int type){
  return iP*4 + type;
}

//TODO: here just continue the initilization scheme, it deserves trying make some values shared
__device__ void multipole::fill_factors(double sqrtE, int numL, CComplex *sigT_factor, unsigned blocksize){
  int iL;
  double arg;
  double twophi;

  for(iL = 0; iL<numL; iL++){
    twophi = pseudo_rho[iL] * sqrtE; 
    if(1==iL)
      twophi -= atan(twophi);
    else if(2==iL){
      arg = 3.0*twophi / (3.0 - twophi*twophi);
      twophi -= atan(arg);
    }
    else if(3==iL){
      arg = twophi*(15.0 - twophi*twophi)/(15.0 - 6.0*twophi*twophi);
      twophi -= atan(arg);
    }
    twophi *= 2.0;
    sigT_factor[iL*blocksize] = CComplex(cos(twophi), -sin(twophi));
  }

}
