#include "hip/hip_runtime.h"
#include "multipole.h"
multipole::multipole(struct multipoledata data){
  size_t size;
  /*
    allocate and assign integers
  */
  size = sizeof(int);
  hipMalloc((void**)&dev_integers, 6*size);
  hipMemcpy(dev_integers+MODE,    &(data.mode), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_integers+WINDOWS, &(data.windows), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_integers+FITORDER, &(data.fitorder), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_integers+NUML, &(data.numL), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_integers+FISSIONABLE, &(data.fissionable), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_integers+LENGTH, &(data.length), size, hipMemcpyHostToDevice);

  /*
    allocate and assign doubles
  */
  size = sizeof(double);
  hipMalloc((void**)&dev_doubles,  4*size);
  hipMemcpy(dev_doubles+STARTE, &(data.startE), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_doubles+ENDE,   &(data.endE), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_doubles+SPACING,&(data.spacing), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_doubles+SQRTAWR, &(data.sqrtAWR), size, hipMemcpyHostToDevice);

  /*
    allocate and assign arrays
  */
  size = data.length*(MP_RF+data.fissionable)*2*sizeof(double);
  hipMalloc((void**)&mpdata, size);
  hipMemcpy(mpdata, data.mpdata, size, hipMemcpyHostToDevice);

  size = data.length*sizeof(unsigned);
  hipMalloc((void**)&l_value, size);
  hipMemcpy(l_value, data.l_value, size, hipMemcpyHostToDevice);

  size = data.numL*sizeof(double);
  hipMalloc((void**)&pseudo_rho, size);
  hipMemcpy(pseudo_rho, data.pseudo_rho, size, hipMemcpyHostToDevice);


  size = data.windows*sizeof(int);
  hipMalloc((void**)&w_start, size);
  hipMemcpy(w_start, data.w_start, size, hipMemcpyHostToDevice);
  hipMalloc((void**)&w_end, size);
  hipMemcpy(w_end, data.w_end, size, hipMemcpyHostToDevice);

  size = (FIT_F+data.fissionable)*(data.fitorder+1)*data.windows*sizeof(double);
  hipMalloc((void**)&fit, size);
  hipMemcpy(fit, data.fit, size, hipMemcpyHostToDevice);

  /*
    Following lines allocate Z_array, W_array for the "in advance" scheme
  */

  int maxwindow=0;
  int cnt;
  int iW;
  for(iW=0;iW<data.windows;iW++){
    cnt = data.w_end[iW]-data.w_start[iW]+1;
    if(cnt>maxwindow)
      maxwindow = cnt;
  }
  size = maxwindow*2*sizeof(double);
  hipMalloc((void**)&Z_array, size);
  hipMalloc((void**)&W_array, size);

}


multipole::~multipole(){
  hipFree(dev_integers);
  hipFree(dev_doubles);
  hipFree(mpdata);
  hipFree(l_value);
  hipFree(pseudo_rho);
  hipFree(w_start);
  hipFree(w_end);
  hipFree(fit);
}
__device__  void multipole::xs_eval_fast(double E, double sqrtKT, 
			double &sigT, double &sigA, double &sigF){
  /* Copy variables to local memory for efficiency */ 
  int mode        = dev_integers[MODE];
  int fitorder    = dev_integers[FITORDER];
  int fissionable = dev_integers[FISSIONABLE];
  int length      = dev_integers[LENGTH];
  int windows     = dev_integers[WINDOWS];

  double spacing = dev_doubles[SPACING];
  double startE  = dev_doubles[STARTE];
  double sqrtAWR = dev_doubles[SQRTAWR];

  int    iP, iC, iW, startW, endW;
  //TODO:I've not found wat to allocate for a thread
  // 5 = maximum numL, consistent with max 4==iL in fill_factors()
  double twophi[5];
  CComplex sigT_factor[5];
  double sqrtE = sqrt(E);
  double power, DOPP, DOPP_ECOEF;
  CComplex w_val;

  if(1==mode)
    iW = (int)(sqrtE - sqrt(startE))/spacing;
  else if(2==mode)
    iW = (int)(log(E) - log(startE))/spacing;
  else
    iW = (int)( E - startE )/spacing;
  startW = w_start[iW];
  endW   = w_end[iW];
  if(startW <= endW)
    fill_factors(sqrtE,twophi,sigT_factor);
  sigT = 0.0;
  sigA = 0.0;
  sigF = 0.0;
  //polynomial fitting
  for (iC=0;iC<=fitorder;iC++){
    power = pow(E,iC);
    sigT += fit[findex(FIT_T, iC, iW,fitorder,windows)]*power;
    sigA += fit[findex(FIT_A, iC, iW,fitorder,windows)]*power;
    if(MP_FISS == fissionable)
      sigF += fit[findex(FIT_F, iC, iW,fitorder,windows)]*power;
  }
  //Faddeeva evaluation in advance
  //TODO: Test whether in advance evaluation is faster
  DOPP = sqrtAWR/sqrtKT;
  DOPP_ECOEF = DOPP/sqrt(PI);
  for(iP=startW;iP<=endW;iP++){
    Z_array[iP-startW] = (sqrtE - mpdata[pindex(MP_EA,iP-1,length)])*DOPP;
    W_array[iP-startW] = Faddeeva::w(Z_array[iP-startW])*DOPP_ECOEF;
  }

  //evaluating
  for(iP=startW;iP<=endW;iP++){
    sigT += real(mpdata[pindex(MP_RT,iP-1,length)]*sigT_factor[l_value[iP-1]-1]*W_array[iP-startW]);
    sigA += real(mpdata[pindex(MP_RA,iP-1,length)]*W_array[iP-startW]);
    if(MP_FISS == fissionable)
      sigF += real(mpdata[pindex(MP_RF,iP-1,length)]*W_array[iP-startW]);
  }


}

__device__  void multipole::xs_eval_fast(double E,  
			double &sigT, double &sigA, double &sigF){
  /* Copy variables to local memory for efficiency */ 
  int mode        = dev_integers[MODE];
  int fitorder    = dev_integers[FITORDER];
  int fissionable = dev_integers[FISSIONABLE];
  int length      = dev_integers[LENGTH];
  int windows     = dev_integers[WINDOWS];
  int numL        = dev_integers[NUML];
  size_t size;
  double spacing = dev_doubles[SPACING];
  double startE  = dev_doubles[STARTE];
  
  int    iP, iC, iW, startW, endW;
  //TODO:I've not found wat to allocate for a thread
  // 5 = maximum numL, consistent with max 4==iL in fill_factors()
  double *twophi;
  CComplex *sigT_factor;

  double sqrtE = sqrt(E);
  double power;
  CComplex PSIIKI, CDUM1, w_val;

  if(1==mode)
    iW = (int)(sqrtE - sqrt(startE))/spacing;
  else if(2==mode)
    iW = (int)(log(E) - log(startE))/spacing;
  else
    iW = (int)( E - startE )/spacing;
  startW = w_start[iW];
  endW   = w_end[iW];
  size = numL*sizeof(double);
  twophi = (double*)malloc(size);
  sigT_factor = (CComplex*)malloc(2*size);
  if(startW <= endW)
    fill_factors(sqrtE,twophi,sigT_factor);
  sigT = 0.0;
  sigA = 0.0;
  sigF = 0.0;
  //polynomial fitting
  for (iC=0;iC<=fitorder;iC++){
    power = pow(E,iC);
    sigT += fit[findex(FIT_T, iC, iW,fitorder,windows)]*power;
    sigA += fit[findex(FIT_A, iC, iW,fitorder,windows)]*power;
    if(MP_FISS == fissionable)
      sigF += fit[findex(FIT_F, iC, iW,fitorder,windows)]*power;
  }
  //Faddeeva evaluation in advance

  //evaluating
  for(iP=startW;iP<=endW;iP++){
    PSIIKI = -ONEI/(mpdata[pindex(MP_EA,iP-1,length)] - sqrtE);
    CDUM1  = PSIIKI / E;
    sigT += real(mpdata[pindex(MP_RT,iP-1,length)]*CDUM1*sigT_factor[l_value[iP-1]-1]);
    sigA += real(mpdata[pindex(MP_RA,iP-1,length)]*CDUM1);
    if(MP_FISS == fissionable)
      sigF += real(mpdata[pindex(MP_RF,iP-1,length)]*CDUM1);
  }
  free(twophi);
  
}


__host__ __device__  int multipole::findex(int type, int iC, int iW, int fitorder, int windows){
  return windows*(fitorder+1)*type+windows*iC+iW;
}

__host__ __device__  int multipole::pindex(int type, int iP, int length){
  return length*type + iP;
}

//TODO: here just continue the initilization scheme, it deserves trying make some values shared
__device__ void multipole::fill_factors(double sqrtE, double *twophi, CComplex *sigT_factor){
  int iL;
  double arg;

  for(iL = 0; iL<dev_integers[NUML]; iL++){
    twophi[iL] = pseudo_rho[iL] * sqrtE; 
    if(2==iL)
      twophi[iL] -= atan(twophi[iL]);
    else if(3==iL){
      arg = 3.0*twophi[iL] / (3.0 - twophi[iL]*twophi[iL]);
      twophi[iL] -= atan(arg);
    }
    else if(4==iL){
      arg = twophi[iL]*(15.0 - twophi[iL]*twophi[iL])/(15.0 - 6.0*twophi[iL]*twophi[iL]);
      twophi[iL] -= atan(arg);
    }
    twophi[iL] *= 2.0;
    sigT_factor[iL] = CComplex(cos(twophi[iL]), -sin(twophi[iL]));
  }

}
