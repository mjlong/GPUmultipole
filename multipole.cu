#include "hip/hip_runtime.h"
#include "multipole.h"
#if defined(__QUICKWT)
//TODO: consider quickw must use float here
// it deserves try double later
//texture<float2> tex_wtable;
/*static __inline__ __device__ CComplex<float> texfetch_complex8(texture<float2> t, int i){
  float2 v = tex1Dfetch(t,i);
  return CComplex<float>(v.x, v.y);
  }*/

texture<float2, 2> tex_wtable;
static __inline__ __device__ CComplex<float> texfetch_complex8(texture<float2,2> t, int i, int j){
  float2 v = tex2D(t, i, j);
  return CComplex<float>(v.x, v.y);
}

#endif

#if defined(__QUICKW)
multipole::multipole(struct multipoledata data, CComplex<CMPTYPE>* wtable){
#else
multipole::multipole(struct multipoledata data){
#endif
  size_t size;
  /*
    allocate and assign integers
  */
  size = sizeof(unsigned);
  hipMalloc((void**)&dev_integers, 4*size);
  hipMemcpy(dev_integers+MODE,    &(data.mode), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_integers+FITORDER, &(data.fitorder), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_integers+NUML, &(data.numL), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_integers+FISSIONABLE, &(data.fissionable), size, hipMemcpyHostToDevice);

  /*
    allocate and assign doubles
  */
  size = sizeof(CMPTYPE);
  hipMalloc((void**)&dev_doubles,  3*size);
  hipMemcpy(dev_doubles+STARTE, &(data.startE), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_doubles+SPACING,&(data.spacing), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_doubles+SQRTAWR, &(data.sqrtAWR), size, hipMemcpyHostToDevice);

  /*
    allocate and assign arrays
  */
  size = data.length*(MP_RF+data.fissionable)*2*sizeof(CMPTYPE);
  hipMalloc((void**)&mpdata, size);
  hipMemcpy(mpdata, data.mpdata, size, hipMemcpyHostToDevice);

  size = data.length*sizeof(unsigned);
  hipMalloc((void**)&l_value, size);
  hipMemcpy(l_value, data.l_value, size, hipMemcpyHostToDevice);

  size = data.numL*sizeof(CMPTYPE);
  hipMalloc((void**)&pseudo_rho, size);
  hipMemcpy(pseudo_rho, data.pseudo_rho, size, hipMemcpyHostToDevice);


  size = data.windows*sizeof(int);
  hipMalloc((void**)&w_start, size);
  hipMemcpy(w_start, data.w_start, size, hipMemcpyHostToDevice);
  hipMalloc((void**)&w_end, size);
  hipMemcpy(w_end, data.w_end, size, hipMemcpyHostToDevice);

  size = (FIT_F+data.fissionable)*(data.fitorder+1)*data.windows*sizeof(CMPTYPE);
  hipMalloc((void**)&fit, size);
  hipMemcpy(fit, data.fit, size, hipMemcpyHostToDevice);
  
#if defined(__QUICKWT)
  //hipBindTexture(NULL, tex_wtable, wtable, LENGTH*LENGTH*sizeof(CMPTYPE)*2);
  hipChannelFormatDesc desc = hipCreateChannelDesc<float2>();
  hipBindTexture2D(NULL, tex_wtable, wtable, desc, LENGTH, LENGTH, sizeof(float2)*LENGTH);
#endif

#if defined(__QUICKWG)
  table = wtable;  
#endif

}


multipole::~multipole(){
}

void multipole::release_pointer(){
  gpuErrchk(hipFree(dev_integers));
  gpuErrchk(hipFree(dev_doubles));
  gpuErrchk(hipFree(mpdata));
  gpuErrchk(hipFree(l_value));
  gpuErrchk(hipFree(pseudo_rho));
  gpuErrchk(hipFree(w_start));
  gpuErrchk(hipFree(w_end));
  gpuErrchk(hipFree(fit));
#if defined(__QUICKWT)
  hipUnbindTexture(tex_wtable);
#endif
}

// xs eval with MIT Faddeeva()
#if defined(__MITW) || defined(__QUICKW)
__device__  void multipole::xs_eval_fast(CMPTYPE E, CMPTYPE sqrtKT, 
			                 CMPTYPE &sigT, CMPTYPE &sigA, CMPTYPE &sigF){

  // Copy variables to local memory for efficiency 
  unsigned mode        = dev_integers[MODE];
  int    iP, iC, iW, startW, endW;
  CMPTYPE spacing = dev_doubles[SPACING];
  CMPTYPE startE  = dev_doubles[STARTE];
  CMPTYPE sqrtE = sqrt(E);
  if(1==mode)
    iW = (int)((sqrtE - sqrt(startE))/spacing);
  else if(2==mode)
    iW = (int)((log(E) - log(startE))/spacing);
  else
    iW = (int)(( E - startE )/spacing);
  unsigned fitorder    = dev_integers[FITORDER];
  unsigned numL        = dev_integers[NUML];
  unsigned fissionable = dev_integers[FISSIONABLE];

  CMPTYPE sqrtAWR = dev_doubles[SQRTAWR];
  CMPTYPE power, DOPP, DOPP_ECOEF;
  CComplex<CMPTYPE> w_val;

  startW = w_start[iW];
  endW   = w_end[iW];
  CComplex<CMPTYPE> sigT_factor[4];
  //CComplex sigtfactor;
  if(startW <= endW)
    fill_factors(sqrtE,numL,sigT_factor);
  sigT = 0.0;
  sigA = 0.0;
  sigF = 0.0;
  //polynomial fitting

  for (iC=0;iC<=fitorder;iC++){
    power = (CMPTYPE)pow((double)E,(double)iC*0.5-1.0);
    sigT += fit[findex(iW,iC,FIT_T,fitorder+1,2+fissionable)]*power;
    sigA += fit[findex(iW,iC,FIT_A,fitorder+1,2+fissionable)]*power;
    if(MP_FISS == fissionable)
      sigF += fit[findex(iW,iC,FIT_F,fitorder+1,2+fissionable)]*power;
  }

  DOPP = sqrtAWR/sqrtKT;
  DOPP_ECOEF = DOPP/E*sqrt(PI);

  for(iP=startW;iP<=endW;iP++){
    //sigtfactor = sigT_factor[l_value[iP-1]-1];
    //w_val = (sqrtE - mpdata[pindex(iP-1,MP_EA)])*DOPP*DOPP_ECOEF;
#if defined(__QUICKWT)
    CComplex<CMPTYPE> z = (sqrtE - mpdata[pindex(iP-1,MP_EA)])*DOPP;
    CMPTYPE p = 10.0*abs(real(z));
    CMPTYPE q = 10.0*imag(z);
    int     l = (int)p + 1;
    int     m = (int)q + 1;
    /*    w_val = w_function(z, 
		       texfetch_complex8(tex_wtable, (m-1)*LENGTH+l),
		       texfetch_complex8(tex_wtable, m*LENGTH + l-1),
		       texfetch_complex8(tex_wtable, m*LENGTH + l  ),
		       texfetch_complex8(tex_wtable, m*LENGTH + l+1),
		       texfetch_complex8(tex_wtable, (m+1)*LENGTH+l),
		       texfetch_complex8(tex_wtable, (m+1)*LENGTH+l+1),
		       p, q)*DOPP_ECOEF;
    */
    w_val = w_function(z, 
		       texfetch_complex8(tex_wtable, m-1, l  ),
		       texfetch_complex8(tex_wtable, m  , l-1),
		       texfetch_complex8(tex_wtable, m  , l  ),
		       texfetch_complex8(tex_wtable, m  , l+1),
		       texfetch_complex8(tex_wtable, m+1, l  ),
		       texfetch_complex8(tex_wtable, m+1, l+1),
		       p, q)*DOPP_ECOEF;
#endif
		       
#if defined(__QUICKWG)
    w_val = w_function((sqrtE - mpdata[pindex(iP-1,MP_EA)])*DOPP,table)*DOPP_ECOEF;
#endif

#if defined(__MITW)
    w_val = Faddeeva::w((sqrtE - mpdata[pindex(iP-1,MP_EA)])*DOPP,0.0)*DOPP_ECOEF;
#endif
    sigT += real(mpdata[pindex(iP-1,MP_RT)]*sigT_factor[l_value[iP-1]-1]*w_val);//sigtfactor);	    
    sigA += real(mpdata[pindex(iP-1,MP_RA)]*w_val);                              
    if(MP_FISS == fissionable)
      sigF += real(mpdata[pindex(iP-1,MP_RF)]*w_val);
  }

}
#endif

//xs eval with Quick W()
/*
__device__  void multipole::xs_eval_fast(CMPTYPE E, CMPTYPE sqrtKT, CComplex<CMPTYPE> *table, 
			                 CMPTYPE &sigT, CMPTYPE &sigA, CMPTYPE &sigF){

  // Copy variables to local memory for efficiency 
  unsigned mode        = dev_integers[MODE];
  int    iP, iC, iW, startW, endW;
  CMPTYPE spacing = dev_doubles[SPACING];
  CMPTYPE startE  = dev_doubles[STARTE];
  CMPTYPE sqrtE = sqrt(E);
  if(1==mode)
    iW = (int)((sqrtE - sqrt(startE))/spacing);
  else if(2==mode)
    iW = (int)((log(E) - log(startE))/spacing);
  else
    iW = (int)(( E - startE )/spacing);
  unsigned fitorder    = dev_integers[FITORDER];
  unsigned numL        = dev_integers[NUML];
  unsigned fissionable = dev_integers[FISSIONABLE];

  CMPTYPE sqrtAWR = dev_doubles[SQRTAWR];
  CMPTYPE power, DOPP, DOPP_ECOEF;
  CComplex<CMPTYPE> w_val;

  startW = w_start[iW];
  endW   = w_end[iW];
  CComplex<CMPTYPE> sigT_factor[4];
  //CComplex sigtfactor;
  if(startW <= endW)
    fill_factors(sqrtE,numL,sigT_factor);
  sigT = 0.0;
  sigA = 0.0;
  sigF = 0.0;
  //polynomial fitting

  for (iC=0;iC<=fitorder;iC++){
    power = (CMPTYPE)pow((double)E,(double)iC*0.5-1.0);
    sigT += fit[findex(iW,iC,FIT_T,fitorder+1,2+fissionable)]*power;
    sigA += fit[findex(iW,iC,FIT_A,fitorder+1,2+fissionable)]*power;
    if(MP_FISS == fissionable)
      sigF += fit[findex(iW,iC,FIT_F,fitorder+1,2+fissionable)]*power;
  }

  DOPP = sqrtAWR/sqrtKT;
  DOPP_ECOEF = DOPP/E*sqrt(PI);

  for(iP=startW;iP<=endW;iP++){
    //sigtfactor = sigT_factor[l_value[iP-1]-1];
    //w_val = (sqrtE - mpdata[pindex(iP-1,MP_EA)])*DOPP*DOPP_ECOEF;
    w_val = w_function((sqrtE - mpdata[pindex(iP-1,MP_EA)])*DOPP,table)*DOPP_ECOEF;
    sigT += real(mpdata[pindex(iP-1,MP_RT)]*sigT_factor[l_value[iP-1]-1]*w_val);//sigtfactor);	    
    sigA += real(mpdata[pindex(iP-1,MP_RA)]*w_val);                              
    if(MP_FISS == fissionable)
      sigF += real(mpdata[pindex(iP-1,MP_RF)]*w_val);
  }

}
*/

//xs eval at 0K
__device__  void multipole::xs_eval_fast(CMPTYPE E,  
                        	 	 CMPTYPE &sigT, CMPTYPE &sigA, CMPTYPE &sigF){

  // Copy variables to local memory for efficiency 
  unsigned mode        = dev_integers[MODE];
  int    iP, iC, iW, startW, endW;
  CMPTYPE spacing = dev_doubles[SPACING];
  CMPTYPE startE  = dev_doubles[STARTE];
  CMPTYPE sqrtE = sqrt(E);
  if(1==mode)
    iW = (int)((sqrtE - sqrt(startE))/spacing);
  else if(2==mode)
    iW = (int)((log(E) - log(startE))/spacing);
  else
    iW = (int)(( E - startE )/spacing);
  unsigned fitorder    = dev_integers[FITORDER];
  unsigned fissionable = dev_integers[FISSIONABLE];
  unsigned numL        = dev_integers[NUML];

  CMPTYPE power;
  CComplex<CMPTYPE> PSIIKI, CDUM1, w_val;

 
  startW = w_start[iW];
  endW   = w_end[iW];
  CComplex<CMPTYPE> sigT_factor[4];
  //CComplex sigtfactor;
  if(startW <= endW)
    fill_factors(sqrtE,numL,sigT_factor);
  sigT = 0.0;
  sigA = 0.0;
  sigF = 0.0;
  //polynomial fitting

  for (iC=0;iC<=fitorder;iC++){
    power = (CMPTYPE)pow((double)E,(double)iC*0.5-1.0);
    sigT += fit[findex(iW,iC,FIT_T,fitorder+1,2+fissionable)]*power;
    sigA += fit[findex(iW,iC,FIT_A,fitorder+1,2+fissionable)]*power;
    if(MP_FISS == fissionable)
      sigF += fit[findex(iW,iC,FIT_F,fitorder+1,2+fissionable)]*power;
  }




  for(iP=startW;iP<=endW;iP++){
    //sigtfactor = sigT_factor[l_value[iP-1]-1];
    PSIIKI = -ONEI/(mpdata[pindex(iP-1,MP_EA)] - sqrtE);
    CDUM1  = PSIIKI / E;
    sigT += real(mpdata[pindex(iP-1,MP_RT)]*CDUM1*sigT_factor[l_value[iP-1]-1]);//sigtfactor);
    sigA += real(mpdata[pindex(iP-1,MP_RA)]*CDUM1);
    if(MP_FISS == fissionable)
      sigF += real(mpdata[pindex(iP-1,MP_RF)]*CDUM1);
  }
  
}

//xs eval at 0k but sampled to sqrtKT
/*
__device__  void multipole::xs_eval_fast(CMPTYPE E, CMPTYPE sqrtKT, CMPTYPE rnd, 
                        	 	 CMPTYPE &sigT, CMPTYPE &sigA, CMPTYPE &sigF){

  // Copy variables to local memory for efficiency 
  unsigned mode        = dev_integers[MODE];
  int    iP, iC, iW, startW, endW;
  CMPTYPE spacing = dev_doubles[SPACING];
  CMPTYPE startE  = dev_doubles[STARTE];
  CMPTYPE sqrtAWR = dev_doubles[SQRTAWR];

  E = E + rnd * sqrtKT * sqrt(0.5) / sqrtAWR;
  CMPTYPE sqrtE = sqrt(E);
  if(1==mode)
    iW = (int)((sqrtE - sqrt(startE))/spacing);
  else if(2==mode)
    iW = (int)((log(E) - log(startE))/spacing);
  else
    iW = (int)(( E - startE )/spacing);
  unsigned fitorder    = dev_integers[FITORDER];
  unsigned fissionable = dev_integers[FISSIONABLE];
  unsigned numL        = dev_integers[NUML];

  CMPTYPE power;
  CComplex<CMPTYPE> PSIIKI, CDUM1, w_val;
 
  startW = w_start[iW];
  endW   = w_end[iW];
  CComplex<CMPTYPE> sigT_factor[4];
  //CComplex sigtfactor;
  if(startW <= endW)
    fill_factors(sqrtE,numL,sigT_factor);
  sigT = 0.0;
  sigA = 0.0;
  sigF = 0.0;
  //polynomial fitting

  for (iC=0;iC<=fitorder;iC++){
    power = (CMPTYPE)pow((double)E,(double)iC*0.5-1.0);
    sigT += fit[findex(iW,iC,FIT_T,fitorder+1,2+fissionable)]*power;
    sigA += fit[findex(iW,iC,FIT_A,fitorder+1,2+fissionable)]*power;
    if(MP_FISS == fissionable)
      sigF += fit[findex(iW,iC,FIT_F,fitorder+1,2+fissionable)]*power;
  }




  for(iP=startW;iP<=endW;iP++){
    //sigtfactor = sigT_factor[l_value[iP-1]-1];
    PSIIKI = -ONEI/(mpdata[pindex(iP-1,MP_EA)] - sqrtE);
    CDUM1  = PSIIKI / E;
    sigT += real(mpdata[pindex(iP-1,MP_RT)]*CDUM1*sigT_factor[l_value[iP-1]-1]);//sigtfactor);
    sigA += real(mpdata[pindex(iP-1,MP_RA)]*CDUM1);
    if(MP_FISS == fissionable)
      sigF += real(mpdata[pindex(iP-1,MP_RF)]*CDUM1);
  }
  
}
*/

__host__ __device__ int multipole::findex(int iW, int iC, int type, int orders, int types){
  return iW*orders*types + iC*types + type; 
}

__host__ __device__ int multipole::pindex(int iP, int type){
  return iP*4 + type;
}

__device__ void multipole::fill_factors(CMPTYPE sqrtE, int numL, 
                                        CComplex<CMPTYPE> *sigT_factor){
  int iL;
  CMPTYPE arg;
  CMPTYPE twophi; 
  
  for(iL = 0; iL<numL; iL++){
    twophi = pseudo_rho[iL] * sqrtE; 
    if(1==iL)
      twophi -= atan(twophi);
    else if(2==iL){
      arg = 3.0*twophi / (3.0 - twophi*twophi);
      twophi -= atan(arg);
    }
    else if(3==iL){
      arg = twophi*(15.0 - twophi*twophi)/(15.0 - 6.0*twophi*twophi);
      twophi -= atan(arg);
    }
    twophi *= 2.0;
    sigT_factor[iL] = CComplex<CMPTYPE>(cos(twophi), -sin(twophi));
  }

}
