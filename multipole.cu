#include "hip/hip_runtime.h"
#include "multipole.h"
multipole::multipole(struct multipoledata data){
  size_t size;
  /*
    allocate and assign integers
  */
  size = sizeof(unsigned);
  hipMalloc((void**)&dev_integers, 4*size);
  hipMemcpy(dev_integers+MODE,    &(data.mode), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_integers+FITORDER, &(data.fitorder), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_integers+NUML, &(data.numL), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_integers+FISSIONABLE, &(data.fissionable), size, hipMemcpyHostToDevice);

  /*
    allocate and assign doubles
  */
  size = sizeof(double);
  hipMalloc((void**)&dev_doubles,  3*size);
  hipMemcpy(dev_doubles+STARTE, &(data.startE), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_doubles+SPACING,&(data.spacing), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_doubles+SQRTAWR, &(data.sqrtAWR), size, hipMemcpyHostToDevice);

  /*
    allocate and assign arrays
  */
  size = data.length*(MP_RF+data.fissionable)*2*sizeof(double);
  hipMalloc((void**)&mpdata, size);
  hipMemcpy(mpdata, data.mpdata, size, hipMemcpyHostToDevice);

  size = data.length*sizeof(unsigned);
  hipMalloc((void**)&l_value, size);
  hipMemcpy(l_value, data.l_value, size, hipMemcpyHostToDevice);

  size = data.numL*sizeof(double);
  hipMalloc((void**)&pseudo_rho, size);
  hipMemcpy(pseudo_rho, data.pseudo_rho, size, hipMemcpyHostToDevice);


  size = data.windows*sizeof(int);
  hipMalloc((void**)&w_start, size);
  hipMemcpy(w_start, data.w_start, size, hipMemcpyHostToDevice);
  hipMalloc((void**)&w_end, size);
  hipMemcpy(w_end, data.w_end, size, hipMemcpyHostToDevice);

  size = (FIT_F+data.fissionable)*(data.fitorder+1)*data.windows*sizeof(double);
  hipMalloc((void**)&fit, size);
  hipMemcpy(fit, data.fit, size, hipMemcpyHostToDevice);
}


multipole::~multipole(){
  gpuErrchk(hipFree(dev_integers));
  hipFree(dev_doubles);
  hipFree(mpdata);
  hipFree(l_value);
  hipFree(pseudo_rho);
  hipFree(w_start);
  hipFree(w_end);
  hipFree(fit);
}
__device__  void multipole::xs_eval_fast(double E, double sqrtKT, 
			                 double &sigT, double &sigA, double &sigF){

  /* Copy variables to local memory for efficiency */ 
  unsigned mode        = dev_integers[MODE];
  int    iP, iC, iW, startW, endW;
  double spacing = dev_doubles[SPACING];
  double startE  = dev_doubles[STARTE];
  double sqrtE = sqrt(E);
  if(1==mode)
    iW = (int)((sqrtE - sqrt(startE))/spacing);
  else if(2==mode)
    iW = (int)((log(E) - log(startE))/spacing);
  else
    iW = (int)(( E - startE )/spacing);
  unsigned fitorder    = dev_integers[FITORDER];
  unsigned numL        = dev_integers[NUML];
  unsigned fissionable = dev_integers[FISSIONABLE];

  double sqrtAWR = dev_doubles[SQRTAWR];
  double power, DOPP, DOPP_ECOEF;
  CComplex w_val;

  startW = w_start[iW];
  endW   = w_end[iW];
  CComplex sigT_factor[4];
  if(startW <= endW)
    fill_factors(sqrtE,numL,sigT_factor);
  sigT = 0.0;
  sigA = 0.0;
  sigF = 0.0;
  //polynomial fitting
  for (iC=0;iC<=fitorder;iC++){
    power = pow(E,iC*0.5-1.0);
    sigT += fit[findex(iW,iC,FIT_T,fitorder+1,2+fissionable)]*power;
    sigA += fit[findex(iW,iC,FIT_A,fitorder+1,2+fissionable)]*power;
    if(MP_FISS == fissionable)
      sigF += fit[findex(iW,iC,FIT_F,fitorder+1,2+fissionable)]*power;
  }

  DOPP = sqrtAWR/sqrtKT;
  DOPP_ECOEF = DOPP/E*sqrt(PI);

  for(iP=startW;iP<=endW;iP++){
    w_val = Faddeeva::w((sqrtE - mpdata[pindex(iP-1,MP_EA)])*DOPP)*DOPP_ECOEF;
    sigT += real(mpdata[pindex(iP-1,MP_RT)]*sigT_factor[l_value[iP-1]-1]*w_val);	    
    sigA += real(mpdata[pindex(iP-1,MP_RA)]*w_val);                              
    if(MP_FISS == fissionable)
      sigF += real(mpdata[pindex(iP-1,MP_RF)]*w_val);
  }

}

__device__  void multipole::xs_eval_fast(double E,  
                        	 	 double &sigT, double &sigA, double &sigF){

  /* Copy variables to local memory for efficiency */ 
  unsigned mode        = dev_integers[MODE];
  unsigned fitorder    = dev_integers[FITORDER];
  unsigned fissionable = dev_integers[FISSIONABLE];
  unsigned numL        = dev_integers[NUML];

  double spacing = dev_doubles[SPACING];
  double startE  = dev_doubles[STARTE];
  
  int    iP, iC, iW, startW, endW;
  CComplex sigT_factor[4];
  double sqrtE = sqrt(E);
  double power;
  CComplex PSIIKI, CDUM1, w_val;

  if(1==mode)
    iW = (int)((sqrtE - sqrt(startE))/spacing);
  else if(2==mode)
    iW = (int)((log(E) - log(startE))/spacing);
  else
    iW = (int)(( E - startE )/spacing);
  startW = w_start[iW];
  endW   = w_end[iW];
  if(startW <= endW)
    fill_factors(sqrtE,numL,sigT_factor);
  sigT = 0.0;
  sigA = 0.0;
  sigF = 0.0;
  //polynomial fitting
  for (iC=0;iC<=fitorder;iC++){
    power = pow(E,iC);
    sigT += fit[findex(iW,iC,FIT_T,fitorder+1,2+fissionable)]*power;
    sigA += fit[findex(iW,iC,FIT_A,fitorder+1,2+fissionable)]*power;
    if(MP_FISS == fissionable)
      sigF += fit[findex(iW,iC,FIT_F,fitorder+1,2+fissionable)]*power;
  }

  for(iP=startW;iP<=endW;iP++){
    PSIIKI = -ONEI/(mpdata[pindex(iP-1,MP_EA)] - sqrtE);
    CDUM1  = PSIIKI / E;
    sigT += real(mpdata[pindex(iP-1,MP_RT)]*CDUM1*sigT_factor[l_value[iP-1]-1]);
    sigA += real(mpdata[pindex(iP-1,MP_RA)]*CDUM1);
    if(MP_FISS == fissionable)
      sigF += real(mpdata[pindex(iP-1,MP_RF)]*CDUM1);
  }
  
}



__host__ __device__ int multipole::findex(int iW, int iC, int type, int orders, int types){
  return iW*orders*types + iC*types + type; 
}

__host__ __device__ int multipole::pindex(int iP, int type){
  return iP*4 + type;
}

__device__ void multipole::fill_factors(double sqrtE, int numL, 
                                        CComplex *sigT_factor){
  int iL;
  double arg;
  double twophi; 
  
  for(iL = 0; iL<numL; iL++){
    twophi = pseudo_rho[iL] * sqrtE; 
    if(1==iL)
      twophi -= atan(twophi);
    else if(2==iL){
      arg = 3.0*twophi / (3.0 - twophi*twophi);
      twophi -= atan(arg);
    }
    else if(3==iL){
      arg = twophi*(15.0 - twophi*twophi)/(15.0 - 6.0*twophi*twophi);
      twophi -= atan(arg);
    }
    twophi *= 2.0;
    sigT_factor[iL] = CComplex(cos(twophi), -sin(twophi));
  }

}
