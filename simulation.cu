#include "hip/hip_runtime.h"
#include "simulation.h"

__global__ void initialize(neutronInfo Info, double energy){
  //  int id = ((blockDim.x*blockDim.y*blockDim.z)*(blockIdx.y*gridDim.x+blockIdx.x)+(blockDim.x*blockDim.y)*threadIdx.z+blockDim.x*threadIdx.y+threadIdx.x);//THREADID;
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  Info.energy[id] = energy; //id+1.0; //(id + 1)*1.63*energy*0.001;// 

}

__global__ void history(multipole U238, double *devicearray, struct neutronInfo Info){
  //__global__ void history(multipole U238, struct neutronInfo Info){
  //TODO:this is one scheme to match threads to 1D array, 
  //try others when real simulation structure becomes clear
  int id = blockDim.x * blockIdx.x + threadIdx.x;//THREADID;

  bool live=true;
  double localenergy;
  double rnd;
  double sigT, sigA, sigF;
  extern __shared__ double shared[];
  //size of shared[] is given as 3rd parameter while launching the kernel
  /* Each thread gets same seed, a different sequence number, no offset */
  hiprand_init(1234, id, 0, &Info.rndState[id]);

  /* Copy state to local memory for efficiency */ 
  hiprandState localState = Info.rndState[id];

  localenergy = Info.energy[id];
  unsigned cnt = 0;
  unsigned *tally = (unsigned*)shared;
  int idl = threadIdx.x;
  int idb = blockIdx.x;
  int blocksize = blockDim.x * blockDim.y * blockDim.z;
  /*
    shift shared memory for double twophi[MAXNUML] and complex sigT_factor[MAXNUML]
  */
  //TODO: tailor to accomodate more than two isotopes
  double *sharedpole = shared + (blocksize>>1) + idl*MAXNUML*3;
  while(live){
    rnd = hiprand_uniform(&localState);
    U238.xs_eval_fast(localenergy, sqrt(300.0*KB), sigT, sigA, sigF, sharedpole);
    localenergy = localenergy * rnd;
    live = (localenergy>10.0);
    cnt = cnt + 1;
    //live = false;
  }

  devicearray[4*id]=localenergy;
  devicearray[4*id+1]=sigT;
  devicearray[4*id+2]=sigA;
  devicearray[4*id+3]=sigF;

  /* Copy state back to global memory */ 
  Info.rndState[id] = localState; 

  /*reduce tally*/
  int i;
  tally[idl] = cnt;
  __syncthreads();
  i = blocksize>>1;
  while(i){
    if(idl<i)
      tally[idl] += tally[idl+i];
    __syncthreads();
    i=i>>1;
  }
  if(0==idl){
    //reduction scheme depends on tally type
    //following is to count moderation times
    Info.ntally.cnt[idb] = tally[0];
  }

}


