#include "hip/hip_runtime.h"
#include "simulation.h" 

__device__ void launch(NeutronInfoStruct* pInfo,int id, double energy){
  pInfo[id].energy = energy;
}

__global__ void initialize(MemStruct pInfo, double energy){
  //int id = ((blockDim.x*blockDim.y*blockDim.z)*(blockIdx.y*gridDim.x+blockIdx.x)+(blockDim.x*blockDim.y)*threadIdx.z+blockDim.x*threadIdx.y+threadIdx.x);//THREADID;
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  launch(pInfo.nInfo, id, energy);
  //pInfo[id].energy = energy; //id+1.0; //(id + 1)*1.63*energy*0.001;// 
  pInfo.thread_active[id] = 1u;
  pInfo.tally[id].cnt = 0;

}

__global__ void history(multipole U238, double *devicearray, NeutronInfoStruct *pInfo, TallyStruct *pTally, int last){
  //TODO:this is one scheme to match threads to 1D array, 
  //try others when real simulation structure becomes clear
  int id = blockDim.x * blockIdx.x + threadIdx.x;

  bool live=true;
  double localenergy;
  double rnd;
  double sigT, sigA, sigF;

  /* Each thread gets same seed, a different sequence number, no offset */
  hiprand_init(1234, id, 0, &(pInfo[id].rndState));

  /* Copy state to local memory for efficiency */ 
  hiprandState localState = pInfo[id].rndState;

  localenergy = pInfo[id].energy;
  unsigned cnt = 0;

    while(live){
      rnd = hiprand_uniform(&localState);
      U238.xs_eval_fast(localenergy, sqrt(300.0*KB), sigT, sigA, sigF);
      localenergy = localenergy * rnd;
      live = (localenergy>1.0);
      cnt = cnt + 1;
      //live = false;
    }
   
  if(last){
  devicearray[4*id]=localenergy/rnd;
  devicearray[4*id+1]=sigT;
  devicearray[4*id+2]=sigA;
  devicearray[4*id+3]=sigF;
  }
  
  /* Copy state back to global memory */ 
  pInfo[id].rndState = localState; 
  pTally[id].cnt += cnt; 
}



__global__ void statistics(TallyStruct *threadtally, unsigned* cnt){
  /*reduce tally*/
  /*TODO:
    alternatives:
    1. only count for a block, saving global memory (acceess)
    2. count for each thread, saving time in thread wait
  */
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned idl = threadIdx.x;
  extern __shared__ unsigned shared[];
  //size of shared[] is given as 3rd parameter while launching the kernel
  int i;
  shared[idl] = threadtally[id].cnt;
  __syncthreads();
  i = blockDim.x>>1;
  while(i){
    if(idl<i)
      shared[idl] += shared[idl+i];
    __syncthreads();
    i=i>>1;
  }
  if(0==idl){
    //reduction scheme depends on tally type
    //following is to count moderation times
    cnt[blockIdx.x] = shared[0];
  }
  
}
