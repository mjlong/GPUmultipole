#include "hip/hip_runtime.h"
#include "simulation.h"

__global__ void initialize(neutronInfo Info, double energy){
  int id = ((blockDim.x*blockDim.y*blockDim.z)*(blockIdx.y*gridDim.x+blockIdx.x)+(blockDim.x*blockDim.y)*threadIdx.z+blockDim.x*threadIdx.y+threadIdx.x);//THREADID;
  Info.energy[id] = energy;//(id + 1)*1.63*energy*0.001;

}

__global__ void history(multipole U238, double *devicearray, struct neutronInfo Info){
  int i;

  //TODO:this is one scheme to match threads to 1D array, 
  //try others when real simulation structure becomes clear
  int id = ((blockDim.x*blockDim.y*blockDim.z)*(blockIdx.y*gridDim.x+blockIdx.x)+(blockDim.x*blockDim.y)*threadIdx.z+blockDim.x*threadIdx.y+threadIdx.x);//THREADID;

  int idl = 
    (blockDim.x*blockDim.y)*threadIdx.z+
    blockDim.x*threadIdx.y+
    threadIdx.x;

  int idb = 
    blockIdx.y*gridDim.x+blockIdx.x;
  int blocksize = blockDim.x * blockDim.y * blockDim.z;

  bool live=true;
  double energy;
  double rnd;
  double sigT, sigA, sigF;
  extern __shared__ double shared[];
  //size of shared[] is given as 3rd parameter while launching the kernel
  double *tally = &shared[0];

  /* Each thread gets same seed, a different sequence number, no offset */
  hiprand_init(1234, id, 0, &Info.rndState[id]);

  /* Copy state to local memory for efficiency */ 
  hiprandState localState = Info.rndState[id];

  energy = Info.energy[id];
  devicearray[4*id]=energy;
  while(live){
    rnd = hiprand_uniform(&localState);
    U238.xs_eval_fast(energy, sqrt(300.0*KB), sigT, sigA, sigF);
    energy = energy * rnd;
    live = (energy>1.0);
  }
   
  devicearray[4*id+1]=sigT;
  devicearray[4*id+2]=sigA;
  devicearray[4*id+3]=sigF;
  
  /* Copy state back to global memory */ 
  Info.rndState[id] = localState; 

  /*reduce tally*/
  tally[idl] = (double)(rnd<0.5);
  __syncthreads();
  i = blocksize>>1;
  while(i){
    if(idl<i)
      tally[idl] += tally[idl+i];
    __syncthreads();
    i=i>>1;
  }
  if(0==idl){
    Info.tally[idb] = tally[0]/blocksize;
  }
}


