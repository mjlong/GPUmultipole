#include "CPUComplex.h"
#include "CComplex.h"
#include "simulation.h"
#include "manmemory.h"

#include "devicebridge.h"
/*
  To compile host and device codes separately, 
  this "main" file works as interface 
  allocating device memory, transfering data and partitioning computation sources
*/

void initialize_neutrons(unsigned gridx, unsigned blockx,MemStruct DeviceMem){
  initialize<<<gridx, blockx>>>(DeviceMem);
}

void start_neutrons(unsigned gridx, unsigned blockx, material mat, multipole mp_data, MemStruct DeviceMem, unsigned num_src){
    history<<<gridx, blockx, blockx*sizeof(unsigned)>>>(mat, mp_data, DeviceMem, num_src);
} 

unsigned count_neutrons(unsigned gridx, unsigned blockx, MemStruct DeviceMem, MemStruct HostMem, unsigned num_src){
//count terminated neutrons 
  unsigned active;
  reduce_sum_plus<<<1, gridx, gridx*sizeof(unsigned)>>>(DeviceMem.block_terminated_neutrons, DeviceMem.num_terminated_neutrons);
  gpuErrchk(hipMemcpy(HostMem.num_terminated_neutrons,DeviceMem.num_terminated_neutrons,sizeof(unsigned int), hipMemcpyDeviceToHost));
  active = HostMem.num_terminated_neutrons[0] + gridx*blockx < num_src;  
  return active;
}

unsigned count_lives(unsigned gridx, unsigned blockx, MemStruct DeviceMem, MemStruct HostMem){
//count neutrons still marked "live"
  unsigned active;
  reduce_sum_plus<<<1, gridx, gridx*sizeof(unsigned)>>>(DeviceMem.block_terminated_neutrons, DeviceMem.num_terminated_neutrons);
  reduce_sum_equal<<<gridx,blockx,blockx*sizeof(unsigned)>>>(DeviceMem.nInfo.live, DeviceMem.block_terminated_neutrons);
  reduce_sum_equal<<<1,gridx, gridx*sizeof(unsigned)>>>(DeviceMem.block_terminated_neutrons, DeviceMem.num_live_neutrons);
  gpuErrchk(hipMemcpy(&active, DeviceMem.num_live_neutrons, sizeof(unsigned), hipMemcpyDeviceToHost));  
  return active;
}

void sort_prepare(unsigned gridx, unsigned blockx,MemStruct DeviceMem, material mat){
  update_sort_key<<<gridx, blockx>>>(DeviceMem, mat);
}

void transport_neutrons(unsigned gridx, unsigned blockx,MemStruct DeviceMem, material mat, unsigned renew){
  transport<<<gridx, blockx>>>(DeviceMem, mat,renew);
}

void print_results(unsigned gridx, unsigned blockx, unsigned num_src, unsigned num_bin, MemStruct DeviceMem, MemStruct HostMem, unsigned* blockcnt,unsigned* cnt, float timems){
  
  unsigned *d_cnt, *h_cnt;
  gpuErrchk(hipMalloc((void**)&d_cnt, num_bin*sizeof(unsigned)));
  h_cnt = (unsigned*)malloc(num_bin*sizeof(unsigned));
  for(int i=0;i<num_bin;i++){
    reduce_sum_plus<<<gridx, blockx, blockx*sizeof(int)>>>(DeviceMem.tally.cnt+i*gridx*blockx, blockcnt+i*gridx);
  }
  for(int i=0;i<num_bin;i++){
    reduce_sum_equal<<<1, gridx, gridx*sizeof(int)>>>(blockcnt+i*gridx, d_cnt+i);
  }
  gpuErrchk(hipMemcpy(h_cnt,d_cnt,sizeof(unsigned)*num_bin, hipMemcpyDeviceToHost));

  gpuErrchk(hipMemcpy(cnt, blockcnt, num_bin*gridx*sizeof(unsigned), hipMemcpyDeviceToHost));

/*print collision cnt and time*/
  unsigned sum = 0;
  for (int i=0;i<gridx;i++){
    for(int j=0;j<num_bin;j++){ 
      printf("%4d ",cnt[i+j*gridx]);
      sum += cnt[i+j*gridx];
    }
    printf("\n");
  }
  for(int j=0;j<num_bin;j++){ 
    printf("%4d ",h_cnt[j]);
  }
  printf("\n");
  printf("time elapsed:%g mus\n", timems*1000/sum);
  
  free(h_cnt);
  gpuErrchk(hipFree(d_cnt));
  FILE *fp=NULL;
  fp = fopen("timelog","a+");
  gpuErrchk(hipMemcpy(HostMem.num_terminated_neutrons, 
		       DeviceMem.num_terminated_neutrons, 
		       sizeof(unsigned int), 
		       hipMemcpyDeviceToHost));
  fprintf(fp,"%-4d,%-4d,%-.6f,%-8d,%-4d,%-2d M\n", gridx, blockx,timems*1000/sum, *HostMem.num_terminated_neutrons, 1, num_src/1000000);
  fclose(fp);
}

void printdevice(){
  hipDeviceProp_t prop; 
  int count;
  hipGetDeviceCount(&count);
  printf("num of devices=%d\n",count);
  for (int i=0; i<count; i++){
    hipGetDeviceProperties( &prop, i );
    printf( "   --- General Information for device %d ---\n", i );
    printf( "Name:  %s\n", prop.name );
    printf( "Compute capability:  %d.%d\n", prop.major, prop.minor );
    printf( "Clock rate:  %d\n", prop.clockRate );
    printf( "Device copy overlap:  " );
    if (prop.deviceOverlap)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n");
    printf( "Kernel execution timeout :  " );
    if (prop.kernelExecTimeoutEnabled)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n" );
    
    printf( "   --- Memory Information for device %d ---\n", i );
    printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
    printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
    printf( "Max mem pitch:  %ld\n", prop.memPitch );
    printf( "Texture Alignment:  %ld\n", prop.textureAlignment );
    
    printf( "   --- MP Information for device %d ---\n", i );
    printf( "Multiprocessor count:  %d\n",
	    prop.multiProcessorCount );
    printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
    printf( "Registers per mp:  %d\n", prop.regsPerBlock );
    printf( "Threads in warp:  %d\n", prop.warpSize );
    printf( "Max threads per block:  %d\n",
	    prop.maxThreadsPerBlock );
    printf( "Max thread dimensions:  (%d, %d, %d)\n",
	    prop.maxThreadsDim[0], prop.maxThreadsDim[1],
	    prop.maxThreadsDim[2] );
    printf( "Max grid dimensions:  (%d, %d, %d)\n",
	    prop.maxGridSize[0], prop.maxGridSize[1],
	    prop.maxGridSize[2] );
    printf( "\n" );
  }


}
