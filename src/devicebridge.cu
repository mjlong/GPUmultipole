#include "hip/hip_runtime.h"
#include "simulation.h"
#include "manmemory.h"

#include "devicebridge.h"
/*
  To compile host and device codes separately, 
  this "main" file works as interface 
  allocating device memory, transfering data and partitioning computation sources
*/
void initialize_neutrons_fix(unsigned gridx, unsigned blockx,MemStruct DeviceMem,float width,int ubat){
  srand (time(NULL));
  int i=0;
  for(i=0;i<ubat;i++){
    fixsrc_sample<<<gridx, blockx>>>(DeviceMem,width,i*gridx*blockx);
  }
  //gpuErrchk(hipDeviceSynchronize());  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_x+gridx*blockx*ubat,DeviceMem.nInfo.pos_x,sizeof(float)*gridx*blockx*ubat, hipMemcpyDeviceToDevice));    
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_y+gridx*blockx*ubat,DeviceMem.nInfo.pos_y,sizeof(float)*gridx*blockx*ubat, hipMemcpyDeviceToDevice));    
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_z+gridx*blockx*ubat,DeviceMem.nInfo.pos_z,sizeof(float)*gridx*blockx*ubat, hipMemcpyDeviceToDevice));    
}

void initialize_neutrons_active(MemStruct DeviceMem, MemStruct HostMem, unsigned num_src){
  int i = HostMem.bank.cursor_end[0]-1;
  int j;
  //find the boundary of generation -1 in the delayed bank
  while(-1==HostMem.bank.generation_of_birth[i]){    i--;  }
  i++; 
  //use source in delayed bank with generation_of_birth=-1 as initial source for 1st active generation
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_x+num_src,HostMem.bank.x+i,sizeof(float)*(HostMem.bank.cursor_end[0]-i), hipMemcpyHostToDevice)); 
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_y+num_src,HostMem.bank.y+i,sizeof(float)*(HostMem.bank.cursor_end[0]-i), hipMemcpyHostToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_z+num_src,HostMem.bank.z+i,sizeof(float)*(HostMem.bank.cursor_end[0]-i), hipMemcpyHostToDevice));  
  //pull from start of the delayed bank to fill the 1st active generation
  j = num_src-(HostMem.bank.cursor_end[0]-i);  // j=number of first few neutrons to be pulled to 1st generation
  j = (j>0)*j + (j<=0)*0; 
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_x+num_src+(HostMem.bank.cursor_end[0]-i),HostMem.bank.x,sizeof(float)*j, hipMemcpyHostToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_y+num_src+(HostMem.bank.cursor_end[0]-i),HostMem.bank.y,sizeof(float)*j, hipMemcpyHostToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_z+num_src+(HostMem.bank.cursor_end[0]-i),HostMem.bank.z,sizeof(float)*j, hipMemcpyHostToDevice));  
  HostMem.bank.cursor_available[0]=j; 
  //since the first j neutrons has been pulled, their time_of_use level must be increased
  while(j>0){
    (HostMem.bank.time_of_use[--j])++;
  }
  
}

void initialize_neutrons_active_not_src(unsigned gridx, unsigned blockx,MemStruct DeviceMem, int num_seg, int seed){
  int i=0;
  for(i=0;i<num_seg;i++){
    initialize_without_src<<<gridx, blockx>>>(DeviceMem,i*gridx*blockx,seed);
  }
  //gpuErrchk(hipDeviceSynchronize());  
}


void initialize_neutrons(unsigned gridx, unsigned blockx,MemStruct DeviceMem,float width,int banksize,int ubat, int seed){
  int i=0;
  for(i=0;i<ubat;i++){
  //  printf("init... %d:%d/%d\n",i*gridx*blockx,(i+1)*gridx*blockx,banksize);
    initialize<<<gridx, blockx>>>(DeviceMem,width,banksize,i*gridx*blockx,seed);
  }
  //gpuErrchk(hipDeviceSynchronize());  
#if defined(__3D)
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_x+gridx*blockx*ubat,DeviceMem.nInfo.pos_x,sizeof(float)*gridx*blockx*ubat, hipMemcpyDeviceToDevice));    
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_y+gridx*blockx*ubat,DeviceMem.nInfo.pos_y,sizeof(float)*gridx*blockx*ubat, hipMemcpyDeviceToDevice));    
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_z+gridx*blockx*ubat,DeviceMem.nInfo.pos_z,sizeof(float)*gridx*blockx*ubat, hipMemcpyDeviceToDevice));    
#endif
#if defined(__1D)
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_x+gridx*blockx*ubat,DeviceMem.nInfo.pos_x,sizeof(float)*gridx*blockx*ubat, hipMemcpyDeviceToDevice));    
#endif
}

#if defined(__SCATTERPLOT)
void copyinitial(MemStruct DeviceMem, MemStruct HostMem, unsigned gridsize){
  gpuErrchk(hipMemcpy(HostMem.nInfo.pos_x,DeviceMem.nInfo.pos_x,sizeof(float)*gridsize, hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(HostMem.nInfo.pos_y,DeviceMem.nInfo.pos_y,sizeof(float)*gridsize, hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(HostMem.nInfo.pos_z,DeviceMem.nInfo.pos_z,sizeof(float)*gridsize, hipMemcpyDeviceToHost)); 
  gpuErrchk(hipMemcpy(HostMem.nInfo.live,DeviceMem.nInfo.live,sizeof(int)*gridsize, hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(HostMem.nInfo.energy,DeviceMem.nInfo.energy,sizeof(CMPTYPE)*gridsize, hipMemcpyDeviceToHost));  
}
#endif

void resetcount(MemStruct DeviceMem){
  unsigned x=0;
  gpuErrchk(hipMemcpy(DeviceMem.num_terminated_neutrons,&x,sizeof(unsigned), hipMemcpyHostToDevice));  
}
#if defined(__1D)
#if defined(__MTALLY)||(__FTALLY)
unsigned setbank(MemStruct DeviceMem, MemStruct HostMem, int gridsize, int tnum_bins){
  float* y2 = (float*)malloc(sizeof(float)*gridsize);
  float* x2 = (float*)malloc(sizeof(float)*gridsize*2);
  int* sid1 = (int*)malloc(sizeof(int)*gridsize);
#if defined(__MTALLY)
  int* sid2 = (int*)malloc(sizeof(int)*gridsize*2);
#endif
  gpuErrchk(hipMemcpy(y2,DeviceMem.nInfo.pos_y,sizeof(float)*gridsize, hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(sid1,DeviceMem.nInfo.imat,sizeof(int )*gridsize, hipMemcpyDeviceToHost));  
  int sid;
  float y; 
  unsigned j=0;
  for(int i=0;i<gridsize;i++){
    y = y2[i]; sid = sid1[i];
    HostMem.batcnt[sid]++;
    if(0!=y){
#if defined(__MTALLY)
      sid = sid/tnum_bins;
      if(y>0){sid2[j]=sid; x2[j++]=y;sid2[j]=sid;x2[j++]=y;sid2[j]=sid;x2[j++]=y;}
      else{sid2[j]=sid; x2[j++]=0-y;sid2[j]=sid; x2[j++]=0-y;}
#else
      if(y>0){x2[j++]=y;x2[j++]=y;x2[j++]=y;}
      else{x2[j++]=0-y;x2[j++]=0-y;}
#endif
    }
  }
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_x+gridsize,x2,sizeof(float)*gridsize*2, hipMemcpyHostToDevice));  
  free(x2);
  free(y2);
#if defined(__MTALLY)
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.imat+gridsize,sid2,sizeof(int)*gridsize*2, hipMemcpyHostToDevice));  
  free(sid2);
#endif
  free(sid1);
  return j;
}
#else
unsigned setbank(MemStruct DeviceMem, MemStruct HostMem, int gridsize){
  float* y2 = (float*)malloc(sizeof(float)*gridsize);
  float* x2 = (float*)malloc(sizeof(float)*gridsize*2);
  gpuErrchk(hipMemcpy(y2,DeviceMem.nInfo.pos_y,sizeof(float)*gridsize, hipMemcpyDeviceToHost));  
  float y; 
  unsigned j=0;
  for(int i=0;i<gridsize;i++){
    y = y2[i]; 
    if(0!=y){
      if(y>0){
	//number=3;
        x2[j++]=y;
	x2[j++]=y;
	x2[j++]=y;
      }
      else{
	//number=2;
	x2[j++]=0-y;
	x2[j++]=0-y;
      }
    }
  }
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_x+gridsize,x2,sizeof(float)*gridsize*2, hipMemcpyHostToDevice));  
  free(x2);
  free(y2);
  return j;
}
#endif//__MTALLY
#endif//__1D
#if defined(__3D)
//==============================================================================
//================setbank_converge() ===========================================
//1. Update fission sites in the phase of converging fission source
//2. The update follows traditional method, multiplicity is not treated
unsigned setbank_converge(MemStruct DeviceMem, MemStruct HostMem, int gridsize){
  float* x2 = (float*)malloc(sizeof(float)*gridsize*2);
  float* y2 = (float*)malloc(sizeof(float)*gridsize*2);
  float* z2 = (float*)malloc(sizeof(float)*gridsize*2);
  gpuErrchk(hipMemcpy(HostMem.nInfo.pos_x,DeviceMem.nInfo.pos_x,sizeof(float)*gridsize, hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(HostMem.nInfo.pos_y,DeviceMem.nInfo.pos_y,sizeof(float)*gridsize, hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(HostMem.nInfo.pos_z,DeviceMem.nInfo.pos_z,sizeof(float)*gridsize, hipMemcpyDeviceToHost));  
  memset(HostMem.nInfo.live,0,sizeof(int)*gridsize);
  gpuErrchk(hipMemcpy(HostMem.nInfo.live, DeviceMem.nInfo.live ,sizeof(int)*gridsize,   hipMemcpyDeviceToHost));  
  int live;  unsigned j=0;int k=0;
  for(int i=0;i<gridsize;i++){
    live = HostMem.nInfo.live[i];
    //if(live<4){
    for(k=0;k<live;k++){//live=2 or 3
      if(j>(gridsize*2)) {printf("live=%d,j=%d,i=%d/%d,overflow\n",live,j,i,gridsize);exit(-1);}
      //else{
      x2[j]=HostMem.nInfo.pos_x[i];
      y2[j]=HostMem.nInfo.pos_y[i];
      z2[j]=HostMem.nInfo.pos_z[i];
      j++;
      //}
    }
    //}
  }
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_x+gridsize,x2,sizeof(float)*gridsize*2, hipMemcpyHostToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_y+gridsize,y2,sizeof(float)*gridsize*2, hipMemcpyHostToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_z+gridsize,z2,sizeof(float)*gridsize*2, hipMemcpyHostToDevice));  
  free(x2);  free(y2);  free(z2);
  return j;
}
//=====================end function setbank_converge() =========================


//==============================================================================
//================setbank_prepbank() ===========================================
//1. Update fission sites in the phase of preparing delayed fission bank =======
//2. The update follows traditional method, multiplicity is not treated ========
//3. But only the unique neutrons are stored into the bank =====================
unsigned setbank_prepbank(MemStruct DeviceMem, MemStruct HostMem, int gridsize, unsigned ibat){
  float* x2 = (float*)malloc(sizeof(float)*gridsize*2);
  float* y2 = (float*)malloc(sizeof(float)*gridsize*2);
  float* z2 = (float*)malloc(sizeof(float)*gridsize*2);
  gpuErrchk(hipMemcpy(HostMem.nInfo.pos_x,DeviceMem.nInfo.pos_x,sizeof(float)*gridsize, hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(HostMem.nInfo.pos_y,DeviceMem.nInfo.pos_y,sizeof(float)*gridsize, hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(HostMem.nInfo.pos_z,DeviceMem.nInfo.pos_z,sizeof(float)*gridsize, hipMemcpyDeviceToHost));  
  memset(HostMem.nInfo.live,0,sizeof(int)*gridsize);
  gpuErrchk(hipMemcpy(HostMem.nInfo.live, DeviceMem.nInfo.live ,sizeof(int)*gridsize,   hipMemcpyDeviceToHost));  
  int live;  unsigned j=0;int k=0;
  unsigned cursor = HostMem.bank.cursor_end[0];
  for(int i=0;i<gridsize;i++){
    live = HostMem.nInfo.live[i];
    //==========If fissioned, fission site generates neutrons for the next generation ==============
    for(k=0;k<live;k++){//live=2 or 3
      if(j>(gridsize*2)) {printf("live=%d,j=%d,i=%d/%d,overflow\n",live,j,i,gridsize);exit(-1);}
      //else{
      x2[j]=HostMem.nInfo.pos_x[i];
      y2[j]=HostMem.nInfo.pos_y[i];
      z2[j]=HostMem.nInfo.pos_z[i];
      j++;
      //}
    }
    //==========If fissioned, fission site also generates neutrons into the delay bank =============
    if(0<live){
      HostMem.bank.x[cursor] = HostMem.nInfo.pos_x[i];
      HostMem.bank.y[cursor] = HostMem.nInfo.pos_y[i];
      HostMem.bank.z[cursor] = HostMem.nInfo.pos_z[i];
      HostMem.bank.generation_of_birth[cursor] = ibat;
      HostMem.bank.time_of_use[cursor] = 0; 
      cursor++; 
    }
  }
  HostMem.bank.cursor_end[0] = cursor;
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_x+gridsize,x2,sizeof(float)*gridsize*2, hipMemcpyHostToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_y+gridsize,y2,sizeof(float)*gridsize*2, hipMemcpyHostToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_z+gridsize,z2,sizeof(float)*gridsize*2, hipMemcpyHostToDevice));  
  free(x2);    free(y2);    free(z2);
  return j;
}
//=====================end function setbank_prepbank() =========================

void bank_print(MemStruct HostMem){
  printf("[==TOU==]");
  int end=HostMem.bank.cursor_end[0]; 
  int ava=HostMem.bank.cursor_available[0];
  int sta=HostMem.bank.cursor_start[0]; 
  int saf=HostMem.bank.cursor_safe[0]; 
  int sag=HostMem.bank.cursor_safe[1]; 
  int siz=HostMem.bank.size[0]; 
  for(int i=0; i<siz; i++){
    printf("[%3d", HostMem.bank.time_of_use[i]);
    if(end==i) printf("e"); 
    if(ava==i) printf("a");
    if(sta==i) printf("i");
    if(saf==i) printf("s");
    if(sag==i) printf("t");
    printf("],");
  }
  printf("\n[==GOB==]");
  for(int i=0; i<siz; i++){
    printf("[%3d", HostMem.bank.generation_of_birth[i]);
    if(end==i) printf("e"); 
    if(ava==i) printf("a");
    if(sta==i) printf("i");
    if(saf==i) printf("s");
    if(sag==i) printf("t");
    printf("],");
  }
  printf("\n");
}
void setbank_active_out(unsigned ibat, MemStruct DeviceMem, MemStruct HostMem, int banksize, unsigned jstart){
  jstart = (jstart>=banksize)*banksize + (jstart<banksize)*jstart;
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_x+banksize,DeviceMem.nInfo.pos_x+2*banksize,sizeof(float)*(jstart), hipMemcpyDeviceToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_y+banksize,DeviceMem.nInfo.pos_y+2*banksize,sizeof(float)*(jstart), hipMemcpyDeviceToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_z+banksize,DeviceMem.nInfo.pos_z+2*banksize,sizeof(float)*(jstart), hipMemcpyDeviceToDevice));  
  if(jstart>=banksize) return;

  float* x2 = (float*)malloc(sizeof(float)*(banksize-jstart));
  float* y2 = (float*)malloc(sizeof(float)*(banksize-jstart));
  float* z2 = (float*)malloc(sizeof(float)*(banksize-jstart));

  //========== sources of delayed fission bank contribute to next generation =======================
  //j-jstart is the number of newly fissioned neutrons of the current segment
  bank_pull(ibat,HostMem,x2,y2,z2,banksize-jstart); 

  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_x+banksize+jstart,x2,sizeof(float)*(banksize-jstart), hipMemcpyHostToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_y+banksize+jstart,y2,sizeof(float)*(banksize-jstart), hipMemcpyHostToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_z+banksize+jstart,z2,sizeof(float)*(banksize-jstart), hipMemcpyHostToDevice));  
  free(x2);  free(y2);  free(z2);
}

void bank_pull(unsigned ibat, MemStruct HostMem, float *x2, float *y2, float* z2, unsigned num_required_neutrons){
  //count is the count of so far pulled neutrons
  unsigned count = 0; 
  unsigned i0=HostMem.bank.cursor_start[0];
  unsigned range = HostMem.bank.size[0]*(HostMem.bank.size[0]==HostMem.bank.cursor_end[0]) + HostMem.bank.cursor_end[0]*(HostMem.bank.size[0]!=HostMem.bank.cursor_end[0]); 
  
  //if(28<=ibat){printf("i_ava=%d,i_s=%d,range=%d\n",i,HostMem.bank.cursor_safe[0], range);}
  while((count<num_required_neutrons)){
    //if(28<=ibat){printf("i=%d,j=%d\n",i,j);}
    x2[count]=HostMem.bank.x[(i0+count)%range];
    y2[count]=HostMem.bank.y[(i0+count)%range];
    z2[count]=HostMem.bank.z[(i0+count)%range];
    (HostMem.bank.time_of_use[(i0+count)%range])++;
    count++;
  }
  HostMem.bank.cursor_available[0]=(i0+count)%range;
}


unsigned setbank_active_in(unsigned ibat, MemStruct DeviceMem, MemStruct HostMem, int gridsize, int banksize, unsigned jstart, int shift){
  float* x2 = (float*)malloc(sizeof(float)*gridsize);
  float* y2 = (float*)malloc(sizeof(float)*gridsize);
  float* z2 = (float*)malloc(sizeof(float)*gridsize);
  int* fission_sites = (int*)malloc(sizeof(int)*gridsize);
  gpuErrchk(hipMemcpy(fission_sites,DeviceMem.nInfo.imat+shift,sizeof(int )*gridsize, hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(HostMem.nInfo.pos_x,DeviceMem.nInfo.pos_x+shift,sizeof(float)*gridsize, hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(HostMem.nInfo.pos_y,DeviceMem.nInfo.pos_y+shift,sizeof(float)*gridsize, hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(HostMem.nInfo.pos_z,DeviceMem.nInfo.pos_z+shift,sizeof(float)*gridsize, hipMemcpyDeviceToHost));  
  memset(HostMem.nInfo.live,0,sizeof(int)*gridsize);
  gpuErrchk(hipMemcpy(HostMem.nInfo.live, DeviceMem.nInfo.live +shift,sizeof(int)*gridsize,   hipMemcpyDeviceToHost));  
  int live;  unsigned j=jstart;int k=0; int fission_site;
  for(int i=0;i<gridsize;i++){
    live = HostMem.nInfo.live[i];
    fission_site = fission_sites[i];
    HostMem.batcnt[fission_site]+= (1*(0!=live));
    if(live>1){
    //========== sources of last generation constribute to next generation =========================
    x2[j-jstart]=HostMem.nInfo.pos_x[i];
    y2[j-jstart]=HostMem.nInfo.pos_y[i];
    z2[j-jstart]=HostMem.nInfo.pos_z[i];
    j++;
    }
  }
  //========== sources of last generation constribute to delayed fission bank ====================
  bank_push(ibat, HostMem,x2,y2,z2,j-jstart);


  k = (j>banksize)*banksize + (j<=banksize)*j;
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_x+2*banksize+jstart,x2,sizeof(float)*(k-jstart), hipMemcpyHostToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_y+2*banksize+jstart,y2,sizeof(float)*(k-jstart), hipMemcpyHostToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_z+2*banksize+jstart,z2,sizeof(float)*(k-jstart), hipMemcpyHostToDevice));  
  gpuErrchk(hipDeviceSynchronize());  

  free(fission_sites);
  free(x2);  free(y2);  free(z2);
  return j;
}

void bank_push(unsigned ibat, MemStruct HostMem,float* x2, float* y2, float* z2, unsigned num_new_neutron){
  int ix=0; 
  int i_end = HostMem.bank.cursor_end[0];
  int i_limit;
  //If the bank has not been filled yet, fill it from cursor_end[0]
  while( (i_end<HostMem.bank.size[0]) && (ix<num_new_neutron)  ){
    HostMem.bank.x[i_end] = x2[ix];
    HostMem.bank.y[i_end] = y2[ix];
    HostMem.bank.z[i_end] = z2[ix];
    HostMem.bank.generation_of_birth[i_end] = ibat;
    HostMem.bank.time_of_use[i_end] = 0; 
    ix++; 
    i_end++; 
  }
  HostMem.bank.cursor_end[0]=i_end;
  //Once the bank is full, cursor_end==cursor_size
  //num_new_neutron neutrons are filled into the position interval [cursor_start,cursor_available]
  //Even if the pull process was looped, neutrons are pushed most to cursor_available because the bank needs old neutrons

  i_limit =  HostMem.bank.cursor_available[0] ;

  i_end = HostMem.bank.cursor_start[0];
  //if(28<=ibat){    printf("push starting point=%d\n",i_end);  }
  while(  (ix<num_new_neutron)  &&  (i_end!=i_limit)   ){
    HostMem.bank.x[i_end] = x2[ix];
    HostMem.bank.y[i_end] = y2[ix];
    HostMem.bank.z[i_end] = z2[ix];
    HostMem.bank.generation_of_birth[i_end] = ibat;
    HostMem.bank.time_of_use[i_end] = 0; 
    ix++; 
    i_end=(i_end+1)%HostMem.bank.size[0]; 
  }
  
  HostMem.bank.cursor_start[0] = i_end;
}


void set_cursor_safe(MemStruct HostMem, unsigned ibat){
  unsigned i = HostMem.bank.cursor_start[0]; 
  unsigned j=0; 
  while(   ((ibat-HostMem.bank.generation_of_birth[i])>=HostMem.bank.delta_safe[0]) && (j<min(HostMem.bank.size[0],HostMem.bank.cursor_end[0]))  ){
    i++;
    i = i%HostMem.bank.size[0];
    j++;
  }
  HostMem.bank.cursor_safe[0]=i;

  i = HostMem.bank.cursor_start[0]; 
  j = 0; 
  while(   ((ibat-1-HostMem.bank.generation_of_birth[i])>=HostMem.bank.delta_safe[0]) && (j<min(HostMem.bank.size[0],HostMem.bank.cursor_end[0]))  ){
    i++;
    i = i%HostMem.bank.size[0];
    j++;
  }
  HostMem.bank.cursor_safe[1]=i;

}


#if defined(__FTALLY2)
//==============================================================================
//=============setbank() of __FTALLY2 samples to meet the \mu===================
unsigned setbank(MemStruct DeviceMem, MemStruct HostMem, int gridsize, int banksize, unsigned jstart, int shift){
  float* x2 = (float*)malloc(sizeof(float)*gridsize*2);
  float* y2 = (float*)malloc(sizeof(float)*gridsize*2);
  float* z2 = (float*)malloc(sizeof(float)*gridsize*2);
  int* sid1 = (int*)malloc(sizeof(int)*gridsize);
  gpuErrchk(hipMemcpy(sid1,DeviceMem.nInfo.imat,sizeof(int )*gridsize, hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(HostMem.nInfo.pos_x,DeviceMem.nInfo.pos_x+shift,sizeof(float)*gridsize, hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(HostMem.nInfo.pos_y,DeviceMem.nInfo.pos_y+shift,sizeof(float)*gridsize, hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(HostMem.nInfo.pos_z,DeviceMem.nInfo.pos_z+shift,sizeof(float)*gridsize, hipMemcpyDeviceToHost));  
  memset(HostMem.nInfo.live,0,sizeof(int)*gridsize);
  gpuErrchk(hipMemcpy(HostMem.nInfo.live, DeviceMem.nInfo.live +shift,sizeof(int)*gridsize,   hipMemcpyDeviceToHost));  
  //jfor(int i=0;i<gridsize;i++){
  //j  printf("i=%d,live=%d,imat[i]=%d\n",i,HostMem.nInfo.live[i],sid1[i]);
  //j}
  int live;  unsigned j=jstart;int k=0; int sid;
  for(int i=0;i<gridsize;i++){
    live = HostMem.nInfo.live[i];
    sid = sid1[i];
    HostMem.batcnt[sid]+= (1*(0!=live));
    if(live>1){
    x2[j-jstart]=HostMem.nInfo.pos_x[i];
    y2[j-jstart]=HostMem.nInfo.pos_y[i];
    z2[j-jstart]=HostMem.nInfo.pos_z[i];
    j++;
    }
    for(k=0;k<live-1;k++){//live=2 or 3
      if((j-jstart)>(gridsize*2)) {printf("live=%d,j=%d,i=%d/%d,overflow\n",live,j,i,gridsize);exit(-1);}
      //else{
      x2[j-jstart]=rand()*1.0/RAND_MAX*HostMem.wdspp[0];
      y2[j-jstart]=rand()*1.0/RAND_MAX*HostMem.wdspp[0];
      z2[j-jstart]=rand()*1.0/RAND_MAX*HostMem.wdspp[0];
      j++;
      //}
    }
  }
  
  k = (j>banksize)*banksize + (j<=banksize)*j;
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_x+2*banksize+jstart,x2,sizeof(float)*(k-jstart), hipMemcpyHostToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_y+2*banksize+jstart,y2,sizeof(float)*(k-jstart), hipMemcpyHostToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_z+2*banksize+jstart,z2,sizeof(float)*(k-jstart), hipMemcpyHostToDevice));  
  gpuErrchk(hipDeviceSynchronize());  
  free(sid1);
  free(x2);  free(y2);  free(z2);

  return j;
}
//==============================================================================
//=============== setbank() of __FTALLY2 samples to satisfy num_src ============
void setbank2(MemStruct DeviceMem, MemStruct HostMem, int banksize, unsigned jstart){
  jstart = (jstart>=banksize)*banksize + (jstart<banksize)*jstart;
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_x+banksize,DeviceMem.nInfo.pos_x+2*banksize,sizeof(float)*(jstart), hipMemcpyDeviceToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_y+banksize,DeviceMem.nInfo.pos_y+2*banksize,sizeof(float)*(jstart), hipMemcpyDeviceToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_z+banksize,DeviceMem.nInfo.pos_z+2*banksize,sizeof(float)*(jstart), hipMemcpyDeviceToDevice));  
  
  if(jstart>=banksize) return;
  float* x2 = (float*)malloc(sizeof(float)*(banksize-jstart));
  float* y2 = (float*)malloc(sizeof(float)*(banksize-jstart));
  float* z2 = (float*)malloc(sizeof(float)*(banksize-jstart));

  int k; 
  for(k=jstart;k<banksize;k++){
    x2[k-jstart]=rand()*1.0/RAND_MAX*HostMem.wdspp[0];
    y2[k-jstart]=rand()*1.0/RAND_MAX*HostMem.wdspp[0];
    z2[k-jstart]=rand()*1.0/RAND_MAX*HostMem.wdspp[0];
  }

  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_x+banksize+jstart,x2,sizeof(float)*(banksize-jstart), hipMemcpyHostToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_y+banksize+jstart,y2,sizeof(float)*(banksize-jstart), hipMemcpyHostToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_z+banksize+jstart,z2,sizeof(float)*(banksize-jstart), hipMemcpyHostToDevice));  
  free(x2);  free(y2);  free(z2);
}

#endif
#endif


int count_pop(int *live, int gridsize){
  int sum = 0;
  for(int i=0;i<gridsize;i++)
    sum += (0!=live[i]);
  return sum;
}
#if defined(__1D)
void start_neutrons(unsigned gridx, unsigned blockx, MemStruct DeviceMem, unsigned ubat,unsigned num_src,unsigned banksize, unsigned tnum_bin){
  int i=0;
  for(i=0;i<ubat;i++){//num_src is important as loop index, but useless in history<<<>>>
#if defined(__MTALLY)
    //printf("i=(%d/%d)",i,ubat);
#endif
    history<<<gridx, blockx/*, blockx*sizeof(unsigned)*/>>>(DeviceMem, num_src,i*gridx*blockx,banksize);
  }
  gpuErrchk(hipDeviceSynchronize());  

#if defined(__MTALLY)
  //printf("\n");
#endif
}
#endif

#if defined(__3D)
//==============================================================================
//=========start_neutrons_active() launches kernels for active generations =====
//1. At least for __FTALLY2, source and delay bank can be performed segment-wise
unsigned start_neutrons_active(unsigned ibat, unsigned gridx, unsigned blockx, MemStruct DeviceMem, unsigned num_seg, unsigned banksize, unsigned tnum_bin, MemStruct HostMem){
  int i=0; int j=0; 
  for(i=0;i<num_seg;i++){
    gpuErrchk(hipMemset(DeviceMem.nInfo.imat, 0, gridx*blockx*sizeof(int)));
    history<<<gridx, blockx/*, blockx*sizeof(unsigned)*/>>>(DeviceMem, gridx*blockx*num_seg,i*gridx*blockx,banksize,1);
    gpuErrchk(hipDeviceSynchronize()); 
    //printf("before %dth setbank_in,j=%d:\n",i,j);    bank_print(HostMem);
    j = setbank_active_in(ibat, DeviceMem, HostMem, gridx*blockx,gridx*blockx*num_seg,j,i*gridx*blockx); 
    //printf("after  %dth setbank_in,j=%d:\n",i,j);    bank_print(HostMem);
  }
  setbank_active_out(ibat, DeviceMem, HostMem, gridx*blockx*num_seg, j);
  //printf("after setbank_out:\n");    bank_print(HostMem);  
  return j;
}


void     start_neutrons(unsigned gridx, unsigned blockx, MemStruct DeviceMem, unsigned ubat,unsigned num_src,unsigned banksize, unsigned tnum_bin){

  int i=0; 
  for(i=0;i<ubat;i++){//num_src is important as loop index, but useless in history<<<>>>
#if defined(__CTALLY2)
    gpuErrchk(hipMemset(DeviceMem.cnt2_t, 0, tnum_bin*gridx*blockx*sizeof(int)));
#endif
    history<<<gridx, blockx/*, blockx*sizeof(unsigned)*/>>>(DeviceMem, num_src,i*gridx*blockx,banksize, 0);
    gpuErrchk(hipDeviceSynchronize()); 
    //printf("%d/%d done\n",i,ubat);
  }
}

void check(unsigned gridx, unsigned blockx, MemStruct DeviceMem, unsigned ubat){
  int i=0;
  printf("start of check\n");
  for(i=0;i<ubat;i++){
    preview_live<<<gridx, blockx>>>(DeviceMem, i*gridx*blockx);
  }
}
#endif

//Original branches of start_neutron() for 1D,3D,ref,vac and steady, transient
//void start_neutrons(unsigned gridx, unsigned blockx, MemStruct DeviceMem, unsigned num_src,unsigned active,unsigned banksize){
//#if defined(__3D)&&!defined(__TRAN)
//  history_3d_ref<<<gridx, blockx/*, blockx*sizeof(unsigned)*/>>>(DeviceMem, num_src,active,banksize);
//#endif
//} 
//
//
unsigned count_neutrons(unsigned gridx, unsigned blockx, MemStruct DeviceMem, MemStruct HostMem, unsigned num_src){
//count terminated neutrons 
  unsigned active;
  reduce_sum_plus<<<1, gridx, gridx*sizeof(int)>>>(DeviceMem.block_terminated_neutrons, DeviceMem.num_terminated_neutrons);
  gpuErrchk(hipMemcpy(HostMem.num_terminated_neutrons,DeviceMem.num_terminated_neutrons,sizeof(int), hipMemcpyDeviceToHost));
  active = HostMem.num_terminated_neutrons[0] + gridx*blockx < num_src;  
#if defined(__PRINTTRACK__)
  printf("[active]%d terminated\n",HostMem.num_terminated_neutrons[0]);
#endif
  return active;
}

unsigned count_lives(unsigned gridx, unsigned blockx, MemStruct DeviceMem, MemStruct HostMem){
//count neutrons still marked "live"
  int active;
  reduce_sum_equal<<<gridx,blockx,blockx*sizeof(int)>>>(DeviceMem.nInfo.live, DeviceMem.block_terminated_neutrons);
  //I made a mistake to reuse block_terminated_neutrons here. 
  //However, as long as blockx<=gridx(size of block_terminated_neutrons), there would be no problem
  reduce_sum_equal<<<1,gridx, gridx*sizeof(int)>>>(DeviceMem.block_terminated_neutrons, DeviceMem.num_live_neutrons);
  gpuErrchk(hipMemcpy(&active, DeviceMem.num_live_neutrons, sizeof(int), hipMemcpyDeviceToHost));  
  return active;
}

void save_results(unsigned ibat, unsigned gridx, unsigned blockx, unsigned num_bin, MemStruct DeviceMem, MemStruct HostMem){
  for(int i=0;i<num_bin;i++){
    reduce_sum_equal<<<gridx, blockx, blockx*sizeof(CMPTYPE)>>>(
                   DeviceMem.tally.cnt+i*gridx*blockx, 
                   DeviceMem.block_spectrum+i*gridx);
  }
  for(int i=0;i<num_bin;i++){
    reduce_sum_equal<<<1, gridx, gridx*sizeof(CMPTYPE)>>>(
                   DeviceMem.block_spectrum+i*gridx, DeviceMem.batcnt+i);
  }
  //printf("%s\n", hipGetErrorString(hipPeekAtLastError()));
  //printf("%s\n", hipGetErrorString(hipDeviceSynchronize()));
  gpuErrchk(hipMemcpy(HostMem.batcnt,DeviceMem.batcnt,sizeof(CMPTYPE)*num_bin, hipMemcpyDeviceToHost));

#if defined(__CTALLY2)
  for(int i=0;i<num_bin;i++){
    reduce_sum_equal<<<gridx, blockx, blockx*sizeof(CMPTYPE)>>>(
                   DeviceMem.tally.cnt2+i*gridx*blockx, 
                   DeviceMem.block_spectrum+i*gridx);
  }
  for(int i=0;i<num_bin;i++){
    reduce_sum_equal<<<1, gridx, gridx*sizeof(CMPTYPE)>>>(
                   DeviceMem.block_spectrum+i*gridx, DeviceMem.batcnt2+i);
  }
  //printf("%s\n", hipGetErrorString(hipPeekAtLastError()));
  //printf("%s\n", hipGetErrorString(hipDeviceSynchronize()));
  gpuErrchk(hipMemcpy(HostMem.batcnt2,DeviceMem.batcnt2,sizeof(CMPTYPE)*num_bin, hipMemcpyDeviceToHost));

#endif
}

void print_results(unsigned meshes, unsigned nbat, double *tally){
  int im,ib;
  for(ib=0;ib<nbat;ib++){
    for(im=0;im<meshes;im++){
      printf("%.5f ",tally[ib*meshes+im]);
    }
    printf("\n");
  }
}
void printdevice(){
  hipDeviceProp_t prop; 
  int count;
  hipGetDeviceCount(&count);
  printf("num of devices=%d\n",count);
  for (int i=0; i<count; i++){
    hipGetDeviceProperties( &prop, i );
    printf( "   --- General Information for device %d ---\n", i );
    printf( "Name:  %s\n", prop.name );
    printf( "Compute capability:  %d.%d\n", prop.major, prop.minor );
    printf( "Clock rate:  %d\n", prop.clockRate );
    printf( "Device copy overlap:  " );
    if (prop.deviceOverlap)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n");
    printf( "Kernel execution timeout :  " );
    if (prop.kernelExecTimeoutEnabled)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n" );
    
    printf( "   --- Memory Information for device %d ---\n", i );
    printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
    printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
    printf( "Max mem pitch:  %ld\n", prop.memPitch );
    printf( "Texture Alignment:  %ld\n", prop.textureAlignment );
    
    printf( "   --- MP Information for device %d ---\n", i );
    printf( "Multiprocessor count:  %d\n",
	    prop.multiProcessorCount );
    printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
    printf( "Registers per mp:  %d\n", prop.regsPerBlock );
    printf( "Threads in warp:  %d\n", prop.warpSize );
    printf( "Max threads per block:  %d\n",
	    prop.maxThreadsPerBlock );
    printf( "Max thread dimensions:  (%d, %d, %d)\n",
	    prop.maxThreadsDim[0], prop.maxThreadsDim[1],
	    prop.maxThreadsDim[2] );
    printf( "Max grid dimensions:  (%d, %d, %d)\n",
	    prop.maxGridSize[0], prop.maxGridSize[1],
	    prop.maxGridSize[2] );
    printf( "\n" );
  }


}
