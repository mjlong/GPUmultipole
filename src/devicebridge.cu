#include "CPUComplex.h"
#include "CComplex.h"
#include "simulation.h"
#include "manmemory.h"

#include "devicebridge.h"



void z2w(CPUComplex<CMPTYPE> *pz, CPUComplex<CMPTYPE>* pw, unsigned numz){
  for(int i=0;i<numz;i++){
    pw[i] = Faddeeva_h::w(pz[i]);
  } 

}

void z2w(CComplex<CMPTYPE> *pz, CComplex<CMPTYPE> *pw, unsigned numz){
  unsigned blocks, threads;
  if(numz<128){
    blocks=1;
    threads = numz;
  }
  else{
    threads = 128; 
    blocks = numz/128;
  }
  z2w_d<<<blocks,threads>>>(pz,pw);  
}
/*
  To compile host and device codes separately, 
  this "main" file works as interface 
  allocating device memory, transfering data and partitioning computation sources
*/

void initialize_neutrons(unsigned gridx, unsigned blockx,MemStruct DeviceMem){
  initialize<<<gridx, blockx>>>(DeviceMem);
}

void start_neutrons(unsigned gridx, unsigned blockx, material mat, multipole mp_data, MemStruct DeviceMem, unsigned num_src,unsigned active){
    history<<<gridx, blockx, blockx*sizeof(unsigned)>>>(mat, mp_data, DeviceMem, num_src,active);
} 

unsigned count_neutrons(unsigned gridx, unsigned blockx, MemStruct DeviceMem, MemStruct HostMem, unsigned num_src){
//count terminated neutrons 
  unsigned active;
  reduce_sum_plus<<<1, gridx, gridx*sizeof(unsigned)>>>(DeviceMem.block_terminated_neutrons, DeviceMem.num_terminated_neutrons);
  gpuErrchk(hipMemcpy(HostMem.num_terminated_neutrons,DeviceMem.num_terminated_neutrons,sizeof(unsigned int), hipMemcpyDeviceToHost));
  active = HostMem.num_terminated_neutrons[0] + gridx*blockx < num_src;  
#if defined(__PRINTTRACK__)
  printf("[active]%d terminated\n",HostMem.num_terminated_neutrons[0]);
#endif
  return active;
}

unsigned count_lives(unsigned gridx, unsigned blockx, MemStruct DeviceMem, MemStruct HostMem){
//count neutrons still marked "live"
  unsigned active;
  reduce_sum_equal<<<gridx,blockx,blockx*sizeof(unsigned)>>>(DeviceMem.nInfo.live, DeviceMem.block_terminated_neutrons);
  //I made a mistake to reuse block_terminated_neutrons here. 
  //However, as long as blockx<=gridx(size of block_terminated_neutrons), there would be no problem
  reduce_sum_equal<<<1,gridx, gridx*sizeof(unsigned)>>>(DeviceMem.block_terminated_neutrons, DeviceMem.num_live_neutrons);
  gpuErrchk(hipMemcpy(&active, DeviceMem.num_live_neutrons, sizeof(unsigned), hipMemcpyDeviceToHost));  
  return active;
}

void sort_prepare(unsigned gridx, unsigned blockx,MemStruct DeviceMem, material mat){
  update_sort_key<<<gridx, blockx>>>(DeviceMem, mat);
}

void transport_neutrons(unsigned gridx, unsigned blockx,MemStruct DeviceMem, material mat, unsigned renew){
  transport<<<gridx, blockx>>>(DeviceMem, mat,renew);
}

void print_results(unsigned gridx, unsigned blockx, unsigned num_src, unsigned num_bin, MemStruct DeviceMem, MemStruct HostMem, float timems){
  
  unsigned *d_cnt, *h_cnt;
  gpuErrchk(hipMalloc((void**)&d_cnt, num_bin*sizeof(unsigned)));
  h_cnt = (unsigned*)malloc(num_bin*sizeof(unsigned));
  for(int i=0;i<num_bin;i++){
    reduce_sum_equal<<<gridx, blockx, blockx*sizeof(unsigned)>>>(
                   DeviceMem.tally.cnt+i*gridx*blockx, 
                   DeviceMem.block_spectrum+i*gridx);
  }
  for(int i=0;i<num_bin;i++){
    reduce_sum_equal<<<1, gridx, gridx*sizeof(unsigned)>>>(
                   DeviceMem.block_spectrum+i*gridx, d_cnt+i);
  }
  gpuErrchk(hipMemcpy(h_cnt,d_cnt,sizeof(unsigned)*num_bin, hipMemcpyDeviceToHost));

/*print collision cnt and time*/
  unsigned sum=0;
  for(int j=0;j<num_bin;j++){ 
    sum+=h_cnt[j];
    printf("%4d \n",h_cnt[j]);
  }
  printf("%u\n",HostMem.num_terminated_neutrons[0]);
  printf("time elapsed:%g mus\n", timems*1000/sum);
  
  free(h_cnt);
  gpuErrchk(hipFree(d_cnt));
  FILE *fp=NULL;
  fp = fopen("timelog","a+");
  gpuErrchk(hipMemcpy(HostMem.num_terminated_neutrons, 
		       DeviceMem.num_terminated_neutrons, 
		       sizeof(unsigned int), 
		       hipMemcpyDeviceToHost));
  fprintf(fp,"%-4d,%-4d,%-.6f,%-8d,%-4d,%-2d M\n", gridx, blockx,timems*1000/sum, *HostMem.num_terminated_neutrons, 1, num_src/1000000);
  fclose(fp);
}

void printdevice(){
  hipDeviceProp_t prop; 
  int count;
  hipGetDeviceCount(&count);
  printf("num of devices=%d\n",count);
  for (int i=0; i<count; i++){
    hipGetDeviceProperties( &prop, i );
    printf( "   --- General Information for device %d ---\n", i );
    printf( "Name:  %s\n", prop.name );
    printf( "Compute capability:  %d.%d\n", prop.major, prop.minor );
    printf( "Clock rate:  %d\n", prop.clockRate );
    printf( "Device copy overlap:  " );
    if (prop.deviceOverlap)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n");
    printf( "Kernel execution timeout :  " );
    if (prop.kernelExecTimeoutEnabled)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n" );
    
    printf( "   --- Memory Information for device %d ---\n", i );
    printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
    printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
    printf( "Max mem pitch:  %ld\n", prop.memPitch );
    printf( "Texture Alignment:  %ld\n", prop.textureAlignment );
    
    printf( "   --- MP Information for device %d ---\n", i );
    printf( "Multiprocessor count:  %d\n",
	    prop.multiProcessorCount );
    printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
    printf( "Registers per mp:  %d\n", prop.regsPerBlock );
    printf( "Threads in warp:  %d\n", prop.warpSize );
    printf( "Max threads per block:  %d\n",
	    prop.maxThreadsPerBlock );
    printf( "Max thread dimensions:  (%d, %d, %d)\n",
	    prop.maxThreadsDim[0], prop.maxThreadsDim[1],
	    prop.maxThreadsDim[2] );
    printf( "Max grid dimensions:  (%d, %d, %d)\n",
	    prop.maxGridSize[0], prop.maxGridSize[1],
	    prop.maxGridSize[2] );
    printf( "\n" );
  }


}
