#include "simulation.h"
#include "manmemory.h"

#include "devicebridge.h"
/*
  To compile host and device codes separately, 
  this "main" file works as interface 
  allocating device memory, transfering data and partitioning computation sources
*/
void initialize_neutrons_fix(unsigned gridx, unsigned blockx,MemStruct DeviceMem,float width,int ubat){
  srand (time(NULL));
  int i=0;
  for(i=0;i<ubat;i++){
    fixsrc_sample<<<gridx, blockx>>>(DeviceMem,width,i*gridx*blockx);
  }
  //gpuErrchk(hipDeviceSynchronize());  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_x+gridx*blockx*ubat,DeviceMem.nInfo.pos_x,sizeof(float)*gridx*blockx*ubat, hipMemcpyDeviceToDevice));    
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_y+gridx*blockx*ubat,DeviceMem.nInfo.pos_y,sizeof(float)*gridx*blockx*ubat, hipMemcpyDeviceToDevice));    
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_z+gridx*blockx*ubat,DeviceMem.nInfo.pos_z,sizeof(float)*gridx*blockx*ubat, hipMemcpyDeviceToDevice));    
}


void initialize_neutrons(unsigned gridx, unsigned blockx,MemStruct DeviceMem,float width,int banksize,int ubat, int seed){
  int i=0;
  for(i=0;i<ubat;i++){
  //  printf("init... %d:%d/%d\n",i*gridx*blockx,(i+1)*gridx*blockx,banksize);
    initialize<<<gridx, blockx>>>(DeviceMem,width,banksize,i*gridx*blockx,seed);
  }
  //gpuErrchk(hipDeviceSynchronize());  
#if defined(__3D)
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_x+gridx*blockx*ubat,
		       DeviceMem.nInfo.pos_x,sizeof(float)*gridx*blockx*ubat,
		       hipMemcpyDeviceToDevice));    
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_y+gridx*blockx*ubat,
		       DeviceMem.nInfo.pos_y,sizeof(float)*gridx*blockx*ubat,
		       hipMemcpyDeviceToDevice));    
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_z+gridx*blockx*ubat,
		       DeviceMem.nInfo.pos_z,sizeof(float)*gridx*blockx*ubat,
		       hipMemcpyDeviceToDevice));    
#endif
#if defined(__1D)
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_x+gridx*blockx*ubat,
		       DeviceMem.nInfo.pos_x,sizeof(float)*gridx*blockx*ubat,
		       hipMemcpyDeviceToDevice));    
#endif
#if defined(__MTALLY)||(__FTALLY_UN)
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.imat +gridx*blockx*ubat,
		       DeviceMem.nInfo.imat ,sizeof(int  )*gridx*blockx*ubat,
		       hipMemcpyDeviceToDevice));    
#endif
}

#if defined(__SCATTERPLOT)
void copyinitial(MemStruct DeviceMem, MemStruct HostMem, unsigned gridsize){
  gpuErrchk(hipMemcpy(HostMem.nInfo.pos_x,DeviceMem.nInfo.pos_x,sizeof(float)*gridsize, hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(HostMem.nInfo.pos_y,DeviceMem.nInfo.pos_y,sizeof(float)*gridsize, hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(HostMem.nInfo.pos_z,DeviceMem.nInfo.pos_z,sizeof(float)*gridsize, hipMemcpyDeviceToHost)); 
  gpuErrchk(hipMemcpy(HostMem.nInfo.live,DeviceMem.nInfo.live,sizeof(int)*gridsize, hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(HostMem.nInfo.energy,DeviceMem.nInfo.energy,sizeof(CMPTYPE)*gridsize, hipMemcpyDeviceToHost));  
}
#endif

void resetcount(MemStruct DeviceMem){
  unsigned x=0;
  gpuErrchk(hipMemcpy(DeviceMem.num_terminated_neutrons,&x,sizeof(unsigned), hipMemcpyHostToDevice));  
}
#if defined(__1D)
#if defined(__MTALLY)||(__FTALLY)||(__FTALLY_UN)

//==============================================================================
//====================MTALLY ===================================================
  //(live,imat) = (0, *): absorption
  //              (0,-1): didn't run //won't encourter for i<oldbanksize
  //             (-1, *): leaked
#if defined(__MTALLY)
unsigned setbank(MemStruct DeviceMem, MemStruct HostMem, int gridsize,
		 int oldbanksize, int tnum_bins){

  float* y2 = (float*)malloc(sizeof(float)*gridsize);
  float* x2 = (float*)malloc(sizeof(float)*gridsize);
  int* sid1 = (int*)malloc(sizeof(int)*gridsize);
  int* sid2 = (int*)malloc(sizeof(int)*gridsize);
  memset(sid2, 0xff, sizeof(int)*gridsize);

  gpuErrchk(hipMemcpy(y2,DeviceMem.nInfo.pos_y,sizeof(float)*gridsize, hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(sid1,DeviceMem.nInfo.imat,sizeof(int )*gridsize, hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(HostMem.nInfo.live,DeviceMem.nInfo.live,
		       sizeof(int )*gridsize, hipMemcpyDeviceToHost));  

  int sid;
  float y; 
  int live;
  unsigned j=0;

  for(int i=0;i<oldbanksize;i++){  
    live = HostMem.nInfo.live[i];
    y = y2[i]; sid = sid1[i];
    if(live>=0)
      HostMem.batcnt[sid]++;
    //live<0, leaked, the 'imat' was set online to be (-1)*source_pos
    if(/*(sid<0)&&*/(live<0 ))
	HostMem.leaked[(sid*(-1))]++;
    if((0!=y)&&(live>=0)){
      sid = sid%tnum_bins;
      if(y>0){sid2[j]=sid; x2[j++]=y;  sid2[j]=sid; x2[j++]=y;
	      sid2[j]=sid; x2[j++]=y;}
      else{   sid2[j]=sid; x2[j++]=0-y;sid2[j]=sid; x2[j++]=0-y;}
    }
    if(j>(gridsize)) {printf("live=%d,j=%d,i=%d/%d,overflow\n",
			       live,j,i,gridsize);exit(-1);}
    
  }


  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_x+gridsize,x2,sizeof(float)
		       *gridsize, hipMemcpyHostToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.imat +gridsize,sid2,sizeof(int)
		       *gridsize, hipMemcpyHostToDevice));  
  free(sid2);
  free(x2);
  free(y2);
  free(sid1);
  return j;
}
#endif
//==============================================================================
//====================FTALLY_UN=================================================
#if defined(__FTALLY_UN)
unsigned setbank(MemStruct DeviceMem, MemStruct HostMem, int gridsize,
		 int oldbanksize, int tnum_bins){
  float* y2 = (float*)malloc(sizeof(float)*gridsize);
  float* x2 = (float*)malloc(sizeof(float)*gridsize);
  int* sid1 = (int*)malloc(sizeof(int)*gridsize);
  int* sid2 = (int*)malloc(sizeof(int)*gridsize);
  memset(sid2, 0xff, sizeof(int)*gridsize);

  gpuErrchk(hipMemcpy(y2,DeviceMem.nInfo.pos_y,sizeof(float)*gridsize, hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(sid1,DeviceMem.nInfo.imat,sizeof(int )*gridsize, hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(HostMem.nInfo.live,DeviceMem.nInfo.live,
		       sizeof(int )*gridsize, hipMemcpyDeviceToHost));  
  int sid;
  float y; 
  int live;
  unsigned j=0;
  for(int i=0;i<oldbanksize;i++){  
    live = HostMem.nInfo.live[i];
    y = y2[i]; sid = sid1[i];
    HostMem.batcnt[sid]++;
    //live<0, leaked, the 'imat' was set online to be (-1)*source_pos
    if((0!=y)&&(live>=0)){
      if(y>0){sid2[j]=sid; x2[j++]=y;  sid2[j]=sid; x2[j++]=y;
	      sid2[j]=sid; x2[j++]=y;}
      else{   sid2[j]=sid; x2[j++]=0-y;sid2[j]=sid; x2[j++]=0-y;}
    }
    if(j>(gridsize)) {printf("live=%d,j=%d,i=%d/%d,overflow\n",
			     live,j,i,gridsize);exit(-1);}

  }


  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_x+gridsize,x2,sizeof(float)
		       *gridsize, hipMemcpyHostToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.imat +gridsize,sid2,sizeof(int)
		       *gridsize, hipMemcpyHostToDevice));  
  free(sid2);
  free(x2);
  free(y2);
  free(sid1);
  return j;
}
#endif


//==============================================================================
//====================FTALLY   =================================================
#if defined(__FTALLY)
unsigned setbank(MemStruct DeviceMem, MemStruct HostMem, int gridsize, int tnum_bins){

  float* y2 = (float*)malloc(sizeof(float)*gridsize);
  float* x2 = (float*)malloc(sizeof(float)*gridsize*2);
  int* sid1 = (int*)malloc(sizeof(int)*gridsize);

  gpuErrchk(hipMemcpy(y2,DeviceMem.nInfo.pos_y,sizeof(float)*gridsize, hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(sid1,DeviceMem.nInfo.imat,sizeof(int )*gridsize, hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(HostMem.nInfo.live,DeviceMem.nInfo.live,
		       sizeof(int )*gridsize, hipMemcpyDeviceToHost));  
  int sid;
  float y; 
  int live;
  unsigned j=0;
  for(int i=0;i<gridsize;i++){
    live = HostMem.nInfo.live[i];
    y = y2[i]; sid = sid1[i];

    if((0!=y)&&(live>=0)){
      if(y>0){x2[j++]=y;  x2[j++]=y;  x2[j++]=y;}
      else{   x2[j++]=0-y;x2[j++]=0-y;}
    }
  }

  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_x+gridsize,x2,sizeof(float)
		       *gridsize*2, hipMemcpyHostToDevice));  
  free(x2);
  free(y2);
  free(sid1);
  return j;
}
#endif




#endif //__MTALLY or __FTALLY or __FTALLY_UN



#if defined(__CTALLY)
unsigned setbank(MemStruct DeviceMem, MemStruct HostMem, int gridsize){
  float* y2 = (float*)malloc(sizeof(float)*gridsize);
  float* x2 = (float*)malloc(sizeof(float)*gridsize*2);
  gpuErrchk(hipMemcpy(y2,DeviceMem.nInfo.pos_y,sizeof(float)*gridsize, hipMemcpyDeviceToHost));  
  float y; 
  unsigned j=0;
  for(int i=0;i<gridsize;i++){
    y = y2[i]; 
    if(0!=y){
      if(y>0){
	//number=3;
        x2[j++]=y;
	x2[j++]=y;
	x2[j++]=y;
      }
      else{
	//number=2;
	x2[j++]=0-y;
	x2[j++]=0-y;
      }
    }
  }
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_x+gridsize,x2,sizeof(float)*gridsize*2, hipMemcpyHostToDevice));  
  free(x2);
  free(y2);
  return j;
}
#endif//__CTALLY

#endif//__1D
#if defined(__3D)
//==============================================================================
//================setbank_converge() ===========================================
unsigned setbank_converge(MemStruct DeviceMem, MemStruct HostMem, int gridsize){
  float* x2 = (float*)malloc(sizeof(float)*gridsize*2);
  float* y2 = (float*)malloc(sizeof(float)*gridsize*2);
  float* z2 = (float*)malloc(sizeof(float)*gridsize*2);
  gpuErrchk(hipMemcpy(HostMem.nInfo.pos_x,DeviceMem.nInfo.pos_x,sizeof(float)
		       *gridsize, hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(HostMem.nInfo.pos_y,DeviceMem.nInfo.pos_y,sizeof(float)
		       *gridsize, hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(HostMem.nInfo.pos_z,DeviceMem.nInfo.pos_z,sizeof(float)
		       *gridsize, hipMemcpyDeviceToHost));  
  memset(HostMem.nInfo.live,0,sizeof(int)*gridsize);
  gpuErrchk(hipMemcpy(HostMem.nInfo.live, DeviceMem.nInfo.live ,sizeof(int)
		       *gridsize,   hipMemcpyDeviceToHost));  
  int live;  unsigned j=0;int k=0;
  for(int i=0;i<gridsize;i++){
    live = HostMem.nInfo.live[i];
    //if(live<4){
    for(k=0;k<live;k++){//live=2 or 3
      if(j>(gridsize*2)) {printf("live=%d,j=%d,i=%d/%d,overflow\n",
				 live,j,i,gridsize);exit(-1);}
      //else{
      x2[j]=HostMem.nInfo.pos_x[i];
      y2[j]=HostMem.nInfo.pos_y[i];
      z2[j]=HostMem.nInfo.pos_z[i];
      j++;
      //}
    }
    //}
  }

#if defined(__MTALLY)
  live = j;
  while(j<gridsize){
    k = rand()%live;
    x2[j] = x2[k];
    y2[j] = y2[k];
    z2[j] = z2[k];
    j++;
  }
#endif  

  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_x+gridsize,x2,sizeof(float)
		       *gridsize*2, hipMemcpyHostToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_y+gridsize,y2,sizeof(float)
		       *gridsize*2, hipMemcpyHostToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_z+gridsize,z2,sizeof(float)
		       *gridsize*2, hipMemcpyHostToDevice));  

  free(x2);  free(y2);  free(z2);
  return live;
}
//=====================end function setbank_converge() =========================
void copysrcforwrite(MemStruct HostMem, int num_src, float* x2, float* y2,
		     float* z2){
  int live;  unsigned j=0;int k=0;
  for(int i=0;i<num_src;i++){
    live = HostMem.nInfo.live[i];
    //if(live<4){
    for(k=0;k<live;k++){//live=2 or 3
      if(j>(num_src*2)) {printf("live=%d,j=%d,i=%d/%d,overflow\n",
				live,j,i,num_src);exit(-1);}
      //else{
      x2[j]=HostMem.nInfo.pos_x[i];
      y2[j]=HostMem.nInfo.pos_y[i];
      z2[j]=HostMem.nInfo.pos_z[i];
      j++;
      //}
    }
    //}
  }
}

#if defined(__FTALLY2)
//==============================================================================
//=============setbank() of __FTALLY2 samples to meet the \mu===================
unsigned setbank(MemStruct DeviceMem, MemStruct HostMem, int gridsize,
		 int banksize, unsigned jstart, int shift){
  float* x2 = (float*)malloc(sizeof(float)*gridsize*2);
  float* y2 = (float*)malloc(sizeof(float)*gridsize*2);
  float* z2 = (float*)malloc(sizeof(float)*gridsize*2);
  int* sid1 = (int*)malloc(sizeof(int)*gridsize);
  gpuErrchk(hipMemcpy(sid1,DeviceMem.nInfo.imat,sizeof(int )*gridsize,
		       hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(HostMem.nInfo.pos_x,DeviceMem.nInfo.pos_x+shift,
		       sizeof(float)*gridsize, hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(HostMem.nInfo.pos_y,DeviceMem.nInfo.pos_y+shift,
		       sizeof(float)*gridsize, hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(HostMem.nInfo.pos_z,DeviceMem.nInfo.pos_z+shift,
		       sizeof(float)*gridsize, hipMemcpyDeviceToHost));  
  memset(HostMem.nInfo.live,0,sizeof(int)*gridsize);
  gpuErrchk(hipMemcpy(HostMem.nInfo.live, DeviceMem.nInfo.live +shift,
		       sizeof(int)*gridsize,   hipMemcpyDeviceToHost));  
  //jfor(int i=0;i<gridsize;i++){
  //j  printf("i=%d,live=%d,imat[i]=%d\n",i,HostMem.nInfo.live[i],sid1[i]);
  //j}
  int live;  unsigned j=jstart;int k=0; int sid;
  for(int i=0;i<gridsize;i++){
    live = HostMem.nInfo.live[i];
    sid = sid1[i];
    HostMem.batcnt[sid]+= (1*(0!=live));
    if(live>1){
    x2[j-jstart]=HostMem.nInfo.pos_x[i];
    y2[j-jstart]=HostMem.nInfo.pos_y[i];
    z2[j-jstart]=HostMem.nInfo.pos_z[i];
    j++;
    }
    for(k=0;k<live-1;k++){//live=2 or 3
      if((j-jstart)>(gridsize*2)) {printf("live=%d,j=%d,i=%d/%d,overflow\n",
					  live,j,i,gridsize);exit(-1);}
      //else{
      x2[j-jstart]=rand()*1.0/RAND_MAX*HostMem.wdspp[0];
      y2[j-jstart]=rand()*1.0/RAND_MAX*HostMem.wdspp[0];
      z2[j-jstart]=rand()*1.0/RAND_MAX*HostMem.wdspp[0];
      j++;
      //}
    }
  }
  
  k = (j>banksize)*banksize + (j<=banksize)*j;
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_x+2*banksize+jstart,x2,sizeof(float)*(k-jstart), hipMemcpyHostToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_y+2*banksize+jstart,y2,sizeof(float)*(k-jstart), hipMemcpyHostToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_z+2*banksize+jstart,z2,sizeof(float)*(k-jstart), hipMemcpyHostToDevice));  
  gpuErrchk(hipDeviceSynchronize());  
  free(sid1);
  free(x2);  free(y2);  free(z2);

  return j;
}
//==============================================================================
//=============== setbank() of __FTALLY2 samples to satisfy num_src ============
void setbank2(MemStruct DeviceMem, MemStruct HostMem, int banksize, unsigned jstart){
  jstart = (jstart>=banksize)*banksize + (jstart<banksize)*jstart;
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_x+banksize,DeviceMem.nInfo.pos_x+2*banksize,sizeof(float)*(jstart), hipMemcpyDeviceToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_y+banksize,DeviceMem.nInfo.pos_y+2*banksize,sizeof(float)*(jstart), hipMemcpyDeviceToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_z+banksize,DeviceMem.nInfo.pos_z+2*banksize,sizeof(float)*(jstart), hipMemcpyDeviceToDevice));  
  
  if(jstart>=banksize) return;
  float* x2 = (float*)malloc(sizeof(float)*(banksize-jstart));
  float* y2 = (float*)malloc(sizeof(float)*(banksize-jstart));
  float* z2 = (float*)malloc(sizeof(float)*(banksize-jstart));

  int k; 
  for(k=jstart;k<banksize;k++){
    x2[k-jstart]=rand()*1.0/RAND_MAX*HostMem.wdspp[0];
    y2[k-jstart]=rand()*1.0/RAND_MAX*HostMem.wdspp[0];
    z2[k-jstart]=rand()*1.0/RAND_MAX*HostMem.wdspp[0];
  }

  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_x+banksize+jstart,x2,sizeof(float)*(banksize-jstart), hipMemcpyHostToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_y+banksize+jstart,y2,sizeof(float)*(banksize-jstart), hipMemcpyHostToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_z+banksize+jstart,z2,sizeof(float)*(banksize-jstart), hipMemcpyHostToDevice));  
  free(x2);  free(y2);  free(z2);
}

#endif

#if defined(__MTALLY)||(__FTALLY)
#if defined(__MTALLY)
unsigned setbank(MemStruct DeviceMem, MemStruct HostMem, int gridsize,
		 int oldbanksize, int tnum_bins){
  float* x2 = (float*)malloc(sizeof(float)*gridsize);
  float* y2 = (float*)malloc(sizeof(float)*gridsize);
  float* z2 = (float*)malloc(sizeof(float)*gridsize);
  int* sid1 = (int*)malloc(sizeof(int)*gridsize);
  int* sid2 = (int*)malloc(sizeof(int)*gridsize);
  memset(sid2, 0xff, sizeof(int)*gridsize);
#else
unsigned setbank(MemStruct DeviceMem, MemStruct HostMem, int gridsize,
		 int tnum_bins){
  float* x2 = (float*)malloc(sizeof(float)*gridsize*2);
  float* y2 = (float*)malloc(sizeof(float)*gridsize*2);
  float* z2 = (float*)malloc(sizeof(float)*gridsize*2);
  int* sid1 = (int*)malloc(sizeof(int)*gridsize);
#endif


  gpuErrchk(hipMemcpy(sid1,DeviceMem.nInfo.imat,sizeof(int )*gridsize,
		       hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(HostMem.nInfo.pos_x,DeviceMem.nInfo.pos_x,sizeof(float)
		       *gridsize, hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(HostMem.nInfo.pos_y,DeviceMem.nInfo.pos_y,sizeof(float)
		       *gridsize, hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(HostMem.nInfo.pos_z,DeviceMem.nInfo.pos_z,sizeof(float)
		       *gridsize, hipMemcpyDeviceToHost));  
  memset(HostMem.nInfo.live,0,sizeof(int)*gridsize);
  gpuErrchk(hipMemcpy(HostMem.nInfo.live, DeviceMem.nInfo.live ,sizeof(int)
		       *gridsize,   hipMemcpyDeviceToHost));  
  int live;  unsigned j=0;int k=0; int sid;
  /*
  for(int i=0;i<gridsize;i++){
    printf("%d ",HostMem.nInfo.live[i]);
    if(0==i%100) printf("\n");
  }
  printf("\n");
  */
#if defined(__MTALLY)
  for(int i=0;i<oldbanksize;i++){
#else
  for(int i=0;i<gridsize;i++){
#endif
    live = HostMem.nInfo.live[i];
    sid = sid1[i];
    HostMem.batcnt[sid]+= 1;
    //if(live<4){
    for(k=0;k<live;k++){//live=2 or 3
#if !defined(__MTALLY)
      if(j>(gridsize*2)) 
#else
      if(j>(gridsize)) 
#endif
	{printf("live=%d,j=%d,i=%d/%d,overflow\n",
		live,j,i,gridsize);exit(-1);}
      //else{
      x2[j]=HostMem.nInfo.pos_x[i];
      y2[j]=HostMem.nInfo.pos_y[i];
      z2[j]=HostMem.nInfo.pos_z[i];
#if defined(__MTALLY)
      sid2[j]=sid%tnum_bins;
#endif
      j++;
      //}
    }
    //}
  }
#if !defined(__MTALLY)
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_x+gridsize,x2,sizeof(float)
		       *gridsize*2, hipMemcpyHostToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_y+gridsize,y2,sizeof(float)
		       *gridsize*2, hipMemcpyHostToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_z+gridsize,z2,sizeof(float)
		       *gridsize*2, hipMemcpyHostToDevice));  
#else
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_x+gridsize,x2,sizeof(float)
		       *gridsize, hipMemcpyHostToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_y+gridsize,y2,sizeof(float)
		       *gridsize, hipMemcpyHostToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_z+gridsize,z2,sizeof(float)
		       *gridsize, hipMemcpyHostToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.imat +gridsize,sid2,sizeof(int)
		       *gridsize, hipMemcpyHostToDevice));  
  free(sid2);
#endif
  free(sid1);
  free(x2);  free(y2);  free(z2);
  return j;
}
#endif
#if defined(__CTALLY)
unsigned setbank(MemStruct DeviceMem, MemStruct HostMem, int gridsize){
  float* x2 = (float*)malloc(sizeof(float)*gridsize*2);
  float* y2 = (float*)malloc(sizeof(float)*gridsize*2);
  float* z2 = (float*)malloc(sizeof(float)*gridsize*2);
  gpuErrchk(hipMemcpy(HostMem.nInfo.pos_x,DeviceMem.nInfo.pos_x,sizeof(float)
		       *gridsize, hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(HostMem.nInfo.pos_y,DeviceMem.nInfo.pos_y,sizeof(float)
		       *gridsize, hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(HostMem.nInfo.pos_z,DeviceMem.nInfo.pos_z,sizeof(float)
		       *gridsize, hipMemcpyDeviceToHost));  
  memset(HostMem.nInfo.live,0,sizeof(int)*gridsize);
  gpuErrchk(hipMemcpy(HostMem.nInfo.live, DeviceMem.nInfo.live ,sizeof(int)
		       *gridsize,   hipMemcpyDeviceToHost));  
  int live;  unsigned j=0;int k=0;
  /*
  for(int i=0;i<gridsize;i++){
    printf("%d ",HostMem.nInfo.live[i]);
    if(0==i%100) printf("\n");
  }
  printf("\n");
  */
  for(int i=0;i<gridsize;i++){
    live = HostMem.nInfo.live[i];
    //if(live<4){
#if defined(__CTALLY2)
    if(live>0){
    x2[j]=HostMem.nInfo.pos_x[i];
    y2[j]=HostMem.nInfo.pos_y[i];
    z2[j]=HostMem.nInfo.pos_z[i];
    j++;
    }
    for(k=0;k<live-1;k++){//live=2 or 3
      if(j>(gridsize*2)) {printf("live=%d,j=%d,i=%d/%d,overflow\n",live,j,i,gridsize);exit(-1);}
      //else{
      x2[j]=rand()*1.0/RAND_MAX*HostMem.wdspp[0];
      y2[j]=rand()*1.0/RAND_MAX*HostMem.wdspp[0];
      z2[j]=rand()*1.0/RAND_MAX*HostMem.wdspp[0];
      j++;
      //}
    }
    for(k=j;k<gridsize;k++){
      x2[k]=rand()*1.0/RAND_MAX*HostMem.wdspp[0];
      y2[k]=rand()*1.0/RAND_MAX*HostMem.wdspp[0];
      z2[k]=rand()*1.0/RAND_MAX*HostMem.wdspp[0];
    }
#else
    for(k=0;k<live;k++){//live=2 or 3
      if(j>(gridsize*2)) {printf("live=%d,j=%d,i=%d/%d,overflow\n",live,j,i,gridsize);exit(-1);}
      //else{
      x2[j]=HostMem.nInfo.pos_x[i];
      y2[j]=HostMem.nInfo.pos_y[i];
      z2[j]=HostMem.nInfo.pos_z[i];
      j++;
      //}
    }
#endif
    //}
  }
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_x+gridsize,x2,sizeof(float)*gridsize*2, hipMemcpyHostToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_y+gridsize,y2,sizeof(float)*gridsize*2, hipMemcpyHostToDevice));  
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_z+gridsize,z2,sizeof(float)*gridsize*2, hipMemcpyHostToDevice));  
  free(x2);  free(y2);  free(z2);
  return j;
}
#endif
#endif


int count_pop(int *live, int gridsize){
  int sum = 0;
  for(int i=0;i<gridsize;i++)
    sum += (0!=live[i]);
  return sum;
}
#if defined(__1D)
void start_neutrons(unsigned gridx, unsigned blockx, MemStruct DeviceMem, unsigned ubat,unsigned num_src,unsigned banksize, unsigned tnum_bin){
  int i=0;
  for(i=0;i<ubat;i++){//num_src is important as loop index, but useless in history<<<>>>
    history<<<gridx, blockx/*, blockx*sizeof(unsigned)*/>>>(DeviceMem, num_src,i*gridx*blockx,banksize);
  }
  gpuErrchk(hipDeviceSynchronize());  
}
#endif

#if defined(__3D)
#if defined(__FTALLY2)
unsigned start_neutrons(unsigned gridx, unsigned blockx, MemStruct DeviceMem, unsigned ubat,unsigned num_src,unsigned banksize, unsigned tnum_bin, MemStruct HostMem){
  int j=0;
#else
void     start_neutrons(unsigned gridx, unsigned blockx, MemStruct DeviceMem, unsigned ubat,unsigned num_src,unsigned banksize, unsigned tnum_bin){
#endif
  int i=0;
  for(i=0;i<ubat;i++){//num_src is important as loop index, but useless in history<<<>>>
#if defined(__CTALLY2)
    gpuErrchk(hipMemset(DeviceMem.cnt2_t, 0, tnum_bin*gridx*blockx*sizeof(int)));
#endif
#if defined(__FTALLY2)
    gpuErrchk(hipMemset(DeviceMem.nInfo.imat, 0, gridx*blockx*sizeof(int)));
#endif
    history<<<gridx, blockx>>>(DeviceMem, num_src,i*gridx*blockx,banksize);
    gpuErrchk(hipDeviceSynchronize()); 
    //printf("%d/%d done\n",i,ubat);
#if defined(__FTALLY2)
    j = setbank(DeviceMem, HostMem, gridx*blockx,gridx*blockx*ubat,j,i*gridx*blockx);
#endif
  }
#if defined(__FTALLY2) 
  setbank2(DeviceMem, HostMem, num_src, j);
  return j;
#endif
}
#endif

void check(unsigned gridx, unsigned blockx, MemStruct DeviceMem, int ubat){
  int i=0;
  printf("start of check\n");
  for(i=0;i<ubat;i++){
    preview_live<<<gridx, blockx>>>(DeviceMem, i*gridx*blockx);
  }
}

//Original branches of start_neutron() for 1D,3D,ref,vac and steady, transient
//void start_neutrons(unsigned gridx, unsigned blockx, MemStruct DeviceMem, unsigned num_src,unsigned active,unsigned banksize){
//#if defined(__3D)&&!defined(__TRAN)
//  history_3d_ref<<<gridx, blockx/*, blockx*sizeof(unsigned)*/>>>(DeviceMem, num_src,active,banksize);
//#endif
//} 
//
//
unsigned count_neutrons(unsigned gridx, unsigned blockx, MemStruct DeviceMem, MemStruct HostMem, unsigned num_src){
//count terminated neutrons 
  unsigned active;
  reduce_sum_plus<<<1, gridx, gridx*sizeof(int)>>>(DeviceMem.block_terminated_neutrons, DeviceMem.num_terminated_neutrons);
  gpuErrchk(hipMemcpy(HostMem.num_terminated_neutrons,DeviceMem.num_terminated_neutrons,sizeof(int), hipMemcpyDeviceToHost));
  active = HostMem.num_terminated_neutrons[0] + gridx*blockx < num_src;  
#if defined(__PRINTTRACK__)
  printf("[active]%d terminated\n",HostMem.num_terminated_neutrons[0]);
#endif
  return active;
}

unsigned count_lives(unsigned gridx, unsigned blockx, MemStruct DeviceMem, MemStruct HostMem){
//count neutrons still marked "live"
  int active;
  reduce_sum_equal<<<gridx,blockx,blockx*sizeof(int)>>>(DeviceMem.nInfo.live, DeviceMem.block_terminated_neutrons);
  //I made a mistake to reuse block_terminated_neutrons here. 
  //However, as long as blockx<=gridx(size of block_terminated_neutrons), there would be no problem
  reduce_sum_equal<<<1,gridx, gridx*sizeof(int)>>>(DeviceMem.block_terminated_neutrons, DeviceMem.num_live_neutrons);
  gpuErrchk(hipMemcpy(&active, DeviceMem.num_live_neutrons, sizeof(int), hipMemcpyDeviceToHost));  
  return active;
}

void save_results(unsigned ibat, unsigned gridx, unsigned blockx, unsigned num_bin, MemStruct DeviceMem, MemStruct HostMem){
  for(int i=0;i<num_bin;i++){
    reduce_sum_equal<<<gridx, blockx, blockx*sizeof(CMPTYPE)>>>(
                   DeviceMem.tally.cnt+i*gridx*blockx, 
                   DeviceMem.block_spectrum+i*gridx);
  }
  for(int i=0;i<num_bin;i++){
    reduce_sum_equal<<<1, gridx, gridx*sizeof(CMPTYPE)>>>(
                   DeviceMem.block_spectrum+i*gridx, DeviceMem.batcnt+i);
  }
  //printf("%s\n", hipGetErrorString(hipPeekAtLastError()));
  //printf("%s\n", hipGetErrorString(hipDeviceSynchronize()));
  gpuErrchk(hipMemcpy(HostMem.batcnt,DeviceMem.batcnt,sizeof(CMPTYPE)*num_bin, hipMemcpyDeviceToHost));

#if defined(__CTALLY2)
  for(int i=0;i<num_bin;i++){
    reduce_sum_equal<<<gridx, blockx, blockx*sizeof(CMPTYPE)>>>(
                   DeviceMem.tally.cnt2+i*gridx*blockx, 
                   DeviceMem.block_spectrum+i*gridx);
  }
  for(int i=0;i<num_bin;i++){
    reduce_sum_equal<<<1, gridx, gridx*sizeof(CMPTYPE)>>>(
                   DeviceMem.block_spectrum+i*gridx, DeviceMem.batcnt2+i);
  }
  //printf("%s\n", hipGetErrorString(hipPeekAtLastError()));
  //printf("%s\n", hipGetErrorString(hipDeviceSynchronize()));
  gpuErrchk(hipMemcpy(HostMem.batcnt2,DeviceMem.batcnt2,sizeof(CMPTYPE)*num_bin, hipMemcpyDeviceToHost));

#endif
/*print collision cnt and time*/
/*
  unsigned sum=0;
  for(int j=0;j<num_bin;j++){ 
    sum+=h_cnt[j];
    printf("%6d ",h_cnt[j]);
  }
  printf("|||%u +++ %u\n",HostMem.num_terminated_neutrons[0],sum);
*/
}

void print_results(unsigned meshes, unsigned nbat, double *tally){
  int im,ib;
  for(ib=0;ib<nbat;ib++){
    for(im=0;im<meshes;im++){
      printf("%.5f ",tally[ib*meshes+im]);
    }
    printf("\n");
  }
}
void printdevice(){
  hipDeviceProp_t prop; 
  int count;
  hipGetDeviceCount(&count);
  printf("num of devices=%d\n",count);
  for (int i=0; i<count; i++){
    hipGetDeviceProperties( &prop, i );
    printf( "   --- General Information for device %d ---\n", i );
    printf( "Name:  %s\n", prop.name );
    printf( "Compute capability:  %d.%d\n", prop.major, prop.minor );
    printf( "Clock rate:  %d\n", prop.clockRate );
    printf( "Device copy overlap:  " );
    if (prop.deviceOverlap)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n");
    printf( "Kernel execution timeout :  " );
    if (prop.kernelExecTimeoutEnabled)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n" );
    
    printf( "   --- Memory Information for device %d ---\n", i );
    printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
    printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
    printf( "Max mem pitch:  %ld\n", prop.memPitch );
    printf( "Texture Alignment:  %ld\n", prop.textureAlignment );
    
    printf( "   --- MP Information for device %d ---\n", i );
    printf( "Multiprocessor count:  %d\n",
	    prop.multiProcessorCount );
    printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
    printf( "Registers per mp:  %d\n", prop.regsPerBlock );
    printf( "Threads in warp:  %d\n", prop.warpSize );
    printf( "Max threads per block:  %d\n",
	    prop.maxThreadsPerBlock );
    printf( "Max thread dimensions:  (%d, %d, %d)\n",
	    prop.maxThreadsDim[0], prop.maxThreadsDim[1],
	    prop.maxThreadsDim[2] );
    printf( "Max grid dimensions:  (%d, %d, %d)\n",
	    prop.maxGridSize[0], prop.maxGridSize[1],
	    prop.maxGridSize[2] );
    printf( "\n" );
  }


}
