#include "CPUComplex.h"
#include "simulation.h"
#include "manmemory.h"

#include "devicebridge.h"
/*
  To compile host and device codes separately, 
  this "main" file works as interface 
  allocating device memory, transfering data and partitioning computation sources
*/
void print_results(unsigned num_src, unsigned num_bin,  MemStruct HostMem, float timems){
  
/*print collision cnt and time*/
  unsigned sum=0;
  for(int j=0;j<num_bin;j++){ 
    sum+=HostMem.spectrum[j];
    printf("%4d \n",HostMem.spectrum[j]);
  }
  printf("%u\n",HostMem.num_terminated_neutrons);
  printf("time elapsed:%g mus\n", timems*1000/sum);
  
  FILE *fp=NULL;
  fp = fopen("timelog","a+");
  fprintf(fp,"%-.6f,%-8d,%-4d,%-2d M\n", timems*1000/sum, HostMem.num_terminated_neutrons, 1, num_src/1000000);
  fclose(fp);
}

#if defined(__XS_GPU)
#include "CComplex.h"
void eval_xs(multipole mp_para,unsigned int* iS_h,unsigned int* iS_d, unsigned numIso, CMPTYPE E, CMPTYPE sqrtKT, 
                               CMPTYPE *sigTs_h, CMPTYPE *sigAs_h, CMPTYPE *sigFs_h,
                               CMPTYPE *sigTs_d, CMPTYPE *sigAs_d, CMPTYPE *sigFs_d){
  gpuErrchk(hipMemcpy(iS_d, iS_h, sizeof(unsigned int), hipMemcpyHostToDevice));  
  device_xs_eval<<<1,numIso>>>(mp_para,iS_d,E,sqrtKT,sigTs_d, sigAs_d, sigFs_d);
  gpuErrchk(hipMemcpy(sigTs_h, sigTs_d, sizeof(CMPTYPE), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(sigAs_h, sigAs_d, sizeof(CMPTYPE), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(sigFs_h, sigFs_d, sizeof(CMPTYPE), hipMemcpyDeviceToHost));
}

void printdevice(){
  hipDeviceProp_t prop; 
  int count;
  hipGetDeviceCount(&count);
  printf("num of devices=%d\n",count);
  for (int i=0; i<count; i++){
    hipGetDeviceProperties( &prop, i );
    printf( "   --- General Information for device %d ---\n", i );
    printf( "Name:  %s\n", prop.name );
    printf( "Compute capability:  %d.%d\n", prop.major, prop.minor );
    printf( "Clock rate:  %d\n", prop.clockRate );
    printf( "Device copy overlap:  " );
    if (prop.deviceOverlap)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n");
    printf( "Kernel execution timeout :  " );
    if (prop.kernelExecTimeoutEnabled)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n" );
    
    printf( "   --- Memory Information for device %d ---\n", i );
    printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
    printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
    printf( "Max mem pitch:  %ld\n", prop.memPitch );
    printf( "Texture Alignment:  %ld\n", prop.textureAlignment );
    
    printf( "   --- MP Information for device %d ---\n", i );
    printf( "Multiprocessor count:  %d\n",
	    prop.multiProcessorCount );
    printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
    printf( "Registers per mp:  %d\n", prop.regsPerBlock );
    printf( "Threads in warp:  %d\n", prop.warpSize );
    printf( "Max threads per block:  %d\n",
	    prop.maxThreadsPerBlock );
    printf( "Max thread dimensions:  (%d, %d, %d)\n",
	    prop.maxThreadsDim[0], prop.maxThreadsDim[1],
	    prop.maxThreadsDim[2] );
    printf( "Max grid dimensions:  (%d, %d, %d)\n",
	    prop.maxGridSize[0], prop.maxGridSize[1],
	    prop.maxGridSize[2] );
    printf( "\n" );
  }
}
#endif

#if defined(__W__GPU)
void eval_w(CPUComplex<CMPTYPE>* z_h, void** z_d, CPUComplex<CMPTYPE>* w_h, void** w_d,unsigned window){
  gpuErrchk(hipMemcpy((CComplex<CMPTYPE>*)(*z_d),z_h,sizeof(CMPTYPE)*2*window,hipMemcpyHostToDevice));
  device_w_eval<<<1,window>>>((CComplex<CMPTYPE>*)(*z_d),(CComplex<CMPTYPE>*)(*w_d)); 
  gpuErrchk(hipMemcpy(w_h,(CComplex<CMPTYPE>*)(*w_d),sizeof(CMPTYPE)*2*window,hipMemcpyDeviceToHost));
}
#endif

