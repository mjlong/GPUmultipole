#include "CPUComplex.h"
#include "CComplex.h"
#include "simulation.h"
#include "manmemory.h"

#include "devicebridge.h"
/*
  To compile host and device codes separately, 
  this "main" file works as interface 
  allocating device memory, transfering data and partitioning computation sources
*/

void print_results(unsigned num_src, unsigned num_bin,  MemStruct HostMem, float timems){
  
/*print collision cnt and time*/
  unsigned sum=0;
  for(int j=0;j<num_bin;j++){ 
    sum+=HostMem.spectrum[j];
    printf("%4d \n",HostMem.spectrum[j]);
  }
  printf("%u\n",HostMem.num_terminated_neutrons);
  printf("time elapsed:%g mus\n", timems*1000/sum);
  
  FILE *fp=NULL;
  fp = fopen("timelog","a+");
  fprintf(fp,"%-.6f,%-8d,%-4d,%-2d M\n", timems*1000/sum, HostMem.num_terminated_neutrons, 1, num_src/1000000);
  fclose(fp);
}

void printdevice(){
  hipDeviceProp_t prop; 
  int count;
  hipGetDeviceCount(&count);
  printf("num of devices=%d\n",count);
  for (int i=0; i<count; i++){
    hipGetDeviceProperties( &prop, i );
    printf( "   --- General Information for device %d ---\n", i );
    printf( "Name:  %s\n", prop.name );
    printf( "Compute capability:  %d.%d\n", prop.major, prop.minor );
    printf( "Clock rate:  %d\n", prop.clockRate );
    printf( "Device copy overlap:  " );
    if (prop.deviceOverlap)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n");
    printf( "Kernel execution timeout :  " );
    if (prop.kernelExecTimeoutEnabled)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n" );
    
    printf( "   --- Memory Information for device %d ---\n", i );
    printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
    printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
    printf( "Max mem pitch:  %ld\n", prop.memPitch );
    printf( "Texture Alignment:  %ld\n", prop.textureAlignment );
    
    printf( "   --- MP Information for device %d ---\n", i );
    printf( "Multiprocessor count:  %d\n",
	    prop.multiProcessorCount );
    printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
    printf( "Registers per mp:  %d\n", prop.regsPerBlock );
    printf( "Threads in warp:  %d\n", prop.warpSize );
    printf( "Max threads per block:  %d\n",
	    prop.maxThreadsPerBlock );
    printf( "Max thread dimensions:  (%d, %d, %d)\n",
	    prop.maxThreadsDim[0], prop.maxThreadsDim[1],
	    prop.maxThreadsDim[2] );
    printf( "Max grid dimensions:  (%d, %d, %d)\n",
	    prop.maxGridSize[0], prop.maxGridSize[1],
	    prop.maxGridSize[2] );
    printf( "\n" );
  }


}
