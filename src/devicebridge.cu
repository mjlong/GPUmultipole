#include "simulation.h"
#include "manmemory.h"

#include "devicebridge.h"
/*
  To compile host and device codes separately, 
  this "main" file works as interface 
  allocating device memory, transfering data and partitioning computation sources
*/

void initialize_neutrons(unsigned gridx, unsigned blockx,MemStruct DeviceMem,float width){
  initialize<<<gridx, blockx>>>(DeviceMem,width);
}

void resetcount(MemStruct DeviceMem){
  unsigned x=0;
  gpuErrchk(hipMemcpy(DeviceMem.num_terminated_neutrons,&x,sizeof(unsigned), hipMemcpyHostToDevice));  
}
unsigned setbank(MemStruct DeviceMem, unsigned gridsize){
  float* y2 = (float*)malloc(sizeof(float)*gridsize);
  float* x2 = (float*)malloc(sizeof(float)*gridsize*3);
  gpuErrchk(hipMemcpy(y2,DeviceMem.nInfo.pos_y,sizeof(float)*gridsize, hipMemcpyDeviceToHost));  
  float y;
  unsigned j=0;
  for(int i=0;i<gridsize;i++){
    y = y2[i];
    if(0!=y){
      if(y>0){
	//number=3;
	x2[j++]=y;
	x2[j++]=y;
	x2[j++]=y;
      }
      else{
	//number=2;
	x2[j++]=0-y;
	x2[j++]=0-y;
      }
    }
  }
  gpuErrchk(hipMemcpy(DeviceMem.nInfo.pos_x,x2,sizeof(float)*gridsize*3, hipMemcpyHostToDevice));  
  free(x2);
  free(y2);
  return j;
}

void start_neutrons(unsigned gridx, unsigned blockx, MemStruct DeviceMem, unsigned num_src,unsigned active,unsigned banksize){
  history<<<gridx, blockx/*, blockx*sizeof(unsigned)*/>>>(DeviceMem, num_src,active,banksize);
} 

unsigned count_neutrons(unsigned gridx, unsigned blockx, MemStruct DeviceMem, MemStruct HostMem, unsigned num_src){
//count terminated neutrons 
  unsigned active;
  reduce_sum_plus<<<1, gridx, gridx*sizeof(unsigned)>>>(DeviceMem.block_terminated_neutrons, DeviceMem.num_terminated_neutrons);
  gpuErrchk(hipMemcpy(HostMem.num_terminated_neutrons,DeviceMem.num_terminated_neutrons,sizeof(unsigned int), hipMemcpyDeviceToHost));
  active = HostMem.num_terminated_neutrons[0] + gridx*blockx < num_src;  
#if defined(__PRINTTRACK__)
  printf("[active]%d terminated\n",HostMem.num_terminated_neutrons[0]);
#endif
  return active;
}

unsigned count_lives(unsigned gridx, unsigned blockx, MemStruct DeviceMem, MemStruct HostMem){
//count neutrons still marked "live"
  unsigned active;
  reduce_sum_equal<<<gridx,blockx,blockx*sizeof(unsigned)>>>(DeviceMem.nInfo.live, DeviceMem.block_terminated_neutrons);
  //I made a mistake to reuse block_terminated_neutrons here. 
  //However, as long as blockx<=gridx(size of block_terminated_neutrons), there would be no problem
  reduce_sum_equal<<<1,gridx, gridx*sizeof(unsigned)>>>(DeviceMem.block_terminated_neutrons, DeviceMem.num_live_neutrons);
  gpuErrchk(hipMemcpy(&active, DeviceMem.num_live_neutrons, sizeof(unsigned), hipMemcpyDeviceToHost));  
  return active;
}

void save_results(unsigned ibat, unsigned gridx, unsigned blockx, unsigned num_src, unsigned num_bin, MemStruct DeviceMem, MemStruct HostMem){
  
  unsigned *d_cnt, *h_cnt;
  gpuErrchk(hipMalloc((void**)&d_cnt, num_bin*sizeof(unsigned)));
  h_cnt = (unsigned*)malloc(num_bin*sizeof(unsigned));
  for(int i=0;i<num_bin;i++){
    reduce_sum_equal<<<gridx, blockx, blockx*sizeof(unsigned)>>>(
                   DeviceMem.tally.cnt+i*gridx*blockx, 
                   DeviceMem.block_spectrum+i*gridx);
  }
  for(int i=0;i<num_bin;i++){
    reduce_sum_equal<<<1, gridx, gridx*sizeof(unsigned)>>>(
                   DeviceMem.block_spectrum+i*gridx, d_cnt+i);
  }
  gpuErrchk(hipMemcpy(h_cnt,d_cnt,sizeof(unsigned)*num_bin, hipMemcpyDeviceToHost));
  copymeans(h_cnt,HostMem.acccnt,num_bin,num_bin*ibat);

/*print collision cnt and time*/
/*
  unsigned sum=0;
  for(int j=0;j<num_bin;j++){ 
    sum+=h_cnt[j];
    printf("%6d ",h_cnt[j]);
  }
  printf("|||%u +++ %u\n",HostMem.num_terminated_neutrons[0],sum);
*/
  free(h_cnt);
  gpuErrchk(hipFree(d_cnt));
  /*
  FILE *fp=NULL;
  fp = fopen("timelog","a+");
  gpuErrchk(hipMemcpy(HostMem.num_terminated_neutrons, 
		       DeviceMem.num_terminated_neutrons, 
		       sizeof(unsigned int), 
		       hipMemcpyDeviceToHost));
  fprintf(fp,"%-4d,%-4d,%-.6f,%-8d,%-4d,%-2d M\n", gridx, blockx,timems*1000/sum, *HostMem.num_terminated_neutrons, 1, num_src/1000000);
  fclose(fp);
  */
}


void print_results(unsigned meshes, unsigned nbat, double *tally){
  int im,ib;
  for(ib=0;ib<nbat;ib++){
    for(im=0;im<meshes;im++){
      printf("%.5f ",tally[ib*meshes+im]);
    }
    printf("\n");
  }
}
void printdevice(){
  hipDeviceProp_t prop; 
  int count;
  hipGetDeviceCount(&count);
  printf("num of devices=%d\n",count);
  for (int i=0; i<count; i++){
    hipGetDeviceProperties( &prop, i );
    printf( "   --- General Information for device %d ---\n", i );
    printf( "Name:  %s\n", prop.name );
    printf( "Compute capability:  %d.%d\n", prop.major, prop.minor );
    printf( "Clock rate:  %d\n", prop.clockRate );
    printf( "Device copy overlap:  " );
    if (prop.deviceOverlap)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n");
    printf( "Kernel execution timeout :  " );
    if (prop.kernelExecTimeoutEnabled)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n" );
    
    printf( "   --- Memory Information for device %d ---\n", i );
    printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
    printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
    printf( "Max mem pitch:  %ld\n", prop.memPitch );
    printf( "Texture Alignment:  %ld\n", prop.textureAlignment );
    
    printf( "   --- MP Information for device %d ---\n", i );
    printf( "Multiprocessor count:  %d\n",
	    prop.multiProcessorCount );
    printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
    printf( "Registers per mp:  %d\n", prop.regsPerBlock );
    printf( "Threads in warp:  %d\n", prop.warpSize );
    printf( "Max threads per block:  %d\n",
	    prop.maxThreadsPerBlock );
    printf( "Max thread dimensions:  (%d, %d, %d)\n",
	    prop.maxThreadsDim[0], prop.maxThreadsDim[1],
	    prop.maxThreadsDim[2] );
    printf( "Max grid dimensions:  (%d, %d, %d)\n",
	    prop.maxGridSize[0], prop.maxGridSize[1],
	    prop.maxGridSize[2] );
    printf( "\n" );
  }


}
