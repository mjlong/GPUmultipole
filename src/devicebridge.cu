#include "CPUComplex.h"
#include "CComplex.h"
#include "simulation.h"
#include "manmemory.h"

#include "devicebridge.h"
/*
  To compile host and device codes separately, 
  this "main" file works as interface 
  allocating device memory, transfering data and partitioning computation sources
*/

void initialize_neutrons(unsigned gridx, unsigned blockx,MemStruct DeviceMem){
  initialize<<<gridx, blockx>>>(DeviceMem, STARTENE);//1.95093e4);
}

void start_neutrons(unsigned gridx, unsigned blockx, unsigned numIsos, multipole mp_data, CMPTYPE* devicearray, MemStruct DeviceMem, unsigned num_src){
#if defined(__TRACK)
    history<<<gridx, blockx, blockx*sizeof(unsigned)>>>(numIsos, mp_data, devicearray, DeviceMem, num_src);
#else
    history<<<gridx, blockx, blockx*sizeof(unsigned)>>>(numIsos, mp_data, DeviceMem, num_src);
#endif
} 

unsigned count_neutrons(unsigned gridx, unsigned blockx, MemStruct DeviceMem, MemStruct HostMem, unsigned num_src){
  unsigned active;
  statistics<<<1, gridx, gridx*sizeof(unsigned)>>>(DeviceMem.block_terminated_neutrons, DeviceMem.num_terminated_neutrons);
  gpuErrchk(hipMemcpy(HostMem.num_terminated_neutrons,DeviceMem.num_terminated_neutrons,sizeof(unsigned int), hipMemcpyDeviceToHost));
  active = HostMem.num_terminated_neutrons[0] + gridx*blockx < num_src;  
  return active;
}

void remain_neutrons(unsigned gridx, unsigned blockx, unsigned numIsos, multipole mp_data, CMPTYPE* devicearray, MemStruct DeviceMem){
  remaining<<<gridx, blockx>>>(numIsos, mp_data, devicearray, DeviceMem);
}

void print_results(unsigned gridx, unsigned blockx, unsigned num_src, MemStruct DeviceMem, MemStruct HostMem, CMPTYPE* hostarray, CMPTYPE* devicearray, unsigned* blockcnt,unsigned* cnt, float timems){
  gpuErrchk(hipMemcpy(hostarray, devicearray, 4*gridx*blockx*sizeof(CMPTYPE), hipMemcpyDeviceToHost));
  
  statistics<<<gridx, blockx, blockx*sizeof(int)>>>(DeviceMem.tally.cnt, blockcnt);
  gpuErrchk(hipMemcpy(cnt, blockcnt, gridx*sizeof(unsigned), hipMemcpyDeviceToHost));

/*print energy & XS (energies for __TRACK)*/
#if !defined(__PLOT)
  for(int i=0;i<gridx*blockx;i++){
    printf(" %.15e %.15e %.15e %.15e",
	   hostarray[4*i],
	   hostarray[4*i+1],
	   hostarray[4*i+2],
	   hostarray[4*i+3]);
    if(hostarray[4*i]<0)
      printf("error-:%d \n",i);
    else{
      if(hostarray[4*i]>20000.0)
	printf("error+:%d \n",i);
      else
	printf("\n");
    }
  }
#endif

/*print collision cnt and time*/
#if !defined(__PROCESS) && !defined(__TRACK) && !defined(__PLOT)
  unsigned sum = 0;
  for (int i=0;i<gridx;i++){
    printf("%4d\n",cnt[i]);
    sum += cnt[i];
  }
  printf("time elapsed:%g mus\n", timems*1000/sum);

  FILE *fp=NULL;
  fp = fopen("timelog","a+");
  gpuErrchk(hipMemcpy(HostMem.num_terminated_neutrons, 
		       DeviceMem.num_terminated_neutrons, 
		       sizeof(unsigned int), 
		       hipMemcpyDeviceToHost));
  fprintf(fp,"%-4d,%-4d,%-.6f,%-8d,%-4d,%-2d M\n", gridx, blockx,timems*1000/sum, *HostMem.num_terminated_neutrons, 1, num_src/1000000);
  fclose(fp);
#endif
}

void printdevice(){
  hipDeviceProp_t prop; 
  int count;
  hipGetDeviceCount(&count);
  printf("num of devices=%d\n",count);
  for (int i=0; i<count; i++){
    hipGetDeviceProperties( &prop, i );
    printf( "   --- General Information for device %d ---\n", i );
    printf( "Name:  %s\n", prop.name );
    printf( "Compute capability:  %d.%d\n", prop.major, prop.minor );
    printf( "Clock rate:  %d\n", prop.clockRate );
    printf( "Device copy overlap:  " );
    if (prop.deviceOverlap)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n");
    printf( "Kernel execution timeout :  " );
    if (prop.kernelExecTimeoutEnabled)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n" );
    
    printf( "   --- Memory Information for device %d ---\n", i );
    printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
    printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
    printf( "Max mem pitch:  %ld\n", prop.memPitch );
    printf( "Texture Alignment:  %ld\n", prop.textureAlignment );
    
    printf( "   --- MP Information for device %d ---\n", i );
    printf( "Multiprocessor count:  %d\n",
	    prop.multiProcessorCount );
    printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
    printf( "Registers per mp:  %d\n", prop.regsPerBlock );
    printf( "Threads in warp:  %d\n", prop.warpSize );
    printf( "Max threads per block:  %d\n",
	    prop.maxThreadsPerBlock );
    printf( "Max thread dimensions:  (%d, %d, %d)\n",
	    prop.maxThreadsDim[0], prop.maxThreadsDim[1],
	    prop.maxThreadsDim[2] );
    printf( "Max grid dimensions:  (%d, %d, %d)\n",
	    prop.maxGridSize[0], prop.maxGridSize[1],
	    prop.maxGridSize[2] );
    printf( "\n" );
  }


}
