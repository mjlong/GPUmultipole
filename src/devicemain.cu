#include "hip/hip_runtime.h"
#include "CPUComplex.h"
#include "CComplex.h"
#include "multipole_data.h"
#include "multipole.h"
#include "simulation.h"

/*
  To compile host and device codes separately, 
  this "main" file works as interface 
  allocating device memory, transfering data and partitioning computation sources
*/


#if defined (__QUICKW)
#include "QuickW.h"
#endif

#if defined (__FOURIERW)
#include "fourierw.h"
__constant__ CMPTYPE a[M+1];
__constant__ CMPTYPE b[M+1];
#endif

#if defined (__QUICKWC) || defined(__INTERPEXP)
__constant__ CMPTYPE2 constwtable[LENGTH*LENGTH];
#endif


void printdevice();

void anyvalue(struct multipoledata data, unsigned setgridx, unsigned setblockx, unsigned num_src, unsigned devstep){
  unsigned gridx, blockx, gridsize;
  unsigned ints=0, sharedmem;
  float timems = 0.0;
  unsigned *cnt, *blockcnt;
  unsigned int active;
  CMPTYPE *hostarray, *devicearray;
  MemStruct HostMem, DeviceMem;
  hipEvent_t start, stop;
  //printdevice();
  gpuErrchk(hipEventCreate(&start));
  gpuErrchk(hipEventCreate(&stop));
  gridx = setgridx;
  blockx = setblockx;
  dim3 dimGrid(gridx, 1);
  dim3 dimBlock(blockx, 1, 1);
  gridsize = gridx*blockx;
  gpuErrchk(hipMalloc((void**)&devicearray, 4*gridsize*sizeof(CMPTYPE)));
  gpuErrchk(hipMemset(devicearray, 0, 4*gridsize*sizeof(CMPTYPE)));

  gpuErrchk(hipMalloc((void**)&(DeviceMem.nInfo.id),       gridsize*sizeof(unsigned)));
  gpuErrchk(hipMalloc((void**)&(DeviceMem.nInfo.rndState), gridsize*sizeof(hiprandState)));
  gpuErrchk(hipMalloc((void**)&(DeviceMem.nInfo.energy),   gridsize*sizeof(CMPTYPE)));

  gpuErrchk(hipMalloc((void**)&(DeviceMem.num_terminated_neutrons), sizeof(unsigned int)));
  gpuErrchk(hipMemset(DeviceMem.num_terminated_neutrons, 0, sizeof(unsigned)));

  gpuErrchk(hipMalloc((void**)&(DeviceMem.block_terminated_neutrons), sizeof(unsigned int)*gridx));
  HostMem.num_terminated_neutrons = (unsigned int *)malloc(sizeof(unsigned int));
  HostMem.num_terminated_neutrons[0] = 0u;
  gpuErrchk(hipMemcpy(DeviceMem.num_terminated_neutrons, HostMem.num_terminated_neutrons, sizeof(unsigned int), hipMemcpyHostToDevice));

  gpuErrchk(hipMalloc((void**)&(DeviceMem.tally.cnt), gridsize*sizeof(unsigned)));
  gpuErrchk(hipMemset(DeviceMem.tally.cnt, 0, gridsize*sizeof(unsigned)));  

  gpuErrchk(hipMalloc((void**)&(blockcnt), gridx*sizeof(unsigned int)));
  gpuErrchk(hipMemset(blockcnt, 0, gridx*sizeof(unsigned int)));

  hostarray = (CMPTYPE*)malloc(4*gridsize*sizeof(CMPTYPE));
  cnt      = (unsigned*)malloc(gridx*sizeof(unsigned));

  //Initialize CUDPP
    CUDPPHandle theCudpp;
    cudppCreate(&theCudpp);
    CUDPPConfiguration config;
    //config.op = CUDPP_ADD;
    config.datatype = CUDPP_DOUBLE;
    config.algorithm = CUDPP_SORT_RADIX;
    //config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;
    //config.options=CUDPP_OPTION_KEYS_ONLY;

    CUDPPHandle sortplan = 0;
    CUDPPResult res = cudppPlan(theCudpp, &sortplan, config, gridsize, 1, 0);

    if (CUDPP_SUCCESS != res)
    {
        printf("Error creating CUDPPPlan\n");
        exit(-1);
    }
 

// construct coefficients a[n] for fourier expansion w
#if defined(__FOURIERW)
  CMPTYPE *da;
  CMPTYPE *db;
  gpuErrchk(hipMalloc((void**)&da, (M+1)*sizeof(CMPTYPE))); 
  gpuErrchk(hipMalloc((void**)&db, (M+1)*sizeof(CMPTYPE))); 
  fill_a<<<1,M+1>>>(da,db); 
  hipMemcpyToSymbol(HIP_SYMBOL(a), da, M*sizeof(CMPTYPE), 0, hipMemcpyDeviceToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(b), db, M*sizeof(CMPTYPE), 0, hipMemcpyDeviceToDevice);
#endif

// fill w function table for quickw
#if defined(__QUICKW)
  CComplex<CMPTYPE> *wtable;
  gpuErrchk(hipMalloc((void**)&wtable, LENGTH*LENGTH * 2 * sizeof(CMPTYPE)));
  fill_w_tabulated<<<LENGTH,LENGTH>>>(wtable);
#if defined(__QUICKWC)
  hipMemcpyToSymbol(HIP_SYMBOL(constwtable), wtable, LENGTH*LENGTH*2*sizeof(CMPTYPE), 0, hipMemcpyDeviceToDevice);
#endif
#if defined(__QUICKWT)
  bindwtable(wtable);
#endif
#endif

#if defined(__QUICKWG)
  multipole U238(data, wtable);
#else
  multipole U238(data);
#endif 

// fill exp(z) table for fourierw
#if defined(__INTERPEXP)
  CComplex<CMPTYPE> *exptable;
  gpuErrchk(hipMalloc((void**)&exptable, LENGTH*LENGTH * 2 * sizeof(CMPTYPE)));
  fill_exp_table<<<LENGTH,LENGTH>>>(exptable);
  hipMemcpyToSymbol(HIP_SYMBOL(constwtable), exptable, LENGTH*LENGTH*2*sizeof(CMPTYPE), 0, hipMemcpyDeviceToDevice);
#endif
  initialize<<<dimGrid, dimBlock>>>(DeviceMem, STARTENE);//1.95093e4);
  //  hipDeviceSynchronize();
  /*
    Note: shared memory size is in unit of Bybe
    And the address can be referred in form of p = pshared + offset
  */
  gpuErrchk(hipEventRecord(start, 0));

#if defined(__PROCESS) //|| defined(__TRACK)
  active = 0u;
#else
  active = 1u;
#endif

  while (active){
#if defined(__TRACK)
    history<<<dimGrid, dimBlock, blockx*sizeof(unsigned)>>>(U238, devicearray, DeviceMem, num_src, devstep);
#else
    history<<<dimGrid, dimBlock, blockx*sizeof(unsigned)>>>(U238, DeviceMem, num_src, devstep);
#endif
    statistics<<<1, dimGrid, gridx*sizeof(unsigned)>>>(DeviceMem.block_terminated_neutrons, DeviceMem.num_terminated_neutrons);
    gpuErrchk(hipMemcpy(HostMem.num_terminated_neutrons, 
		       DeviceMem.num_terminated_neutrons, 
		       sizeof(unsigned int), 
		       hipMemcpyDeviceToHost));
    cudppRadixSort(sortplan, DeviceMem.nInfo.energy, DeviceMem.nInfo.id, gridsize);
    //                       keys,                   values,             numElements
    active = HostMem.num_terminated_neutrons[0] + gridsize < num_src;  
  }

  remaining<<<dimGrid, dimBlock>>>(U238, devicearray, DeviceMem);

  gpuErrchk(hipEventRecord(stop, 0));
  gpuErrchk(hipEventSynchronize(stop));
  gpuErrchk(hipEventElapsedTime(&timems, start, stop));

  printf("time elapsed:%3.1f ms\n", timems);
 
  gpuErrchk(hipMemcpy(hostarray, devicearray, 4*gridsize*sizeof(CMPTYPE), hipMemcpyDeviceToHost));

  
  ints = blockx;
  sharedmem = ints*sizeof(int);
  statistics<<<dimGrid, dimBlock, sharedmem>>>(DeviceMem.tally.cnt, blockcnt);
  gpuErrchk(hipMemcpy(cnt, blockcnt, gridx*sizeof(unsigned), hipMemcpyDeviceToHost));

/*print energy & XS (energies for __TRACK)*/
#if !defined(__PLOT)
  for(int i=0;i<gridsize;i++){
    printf(" %.15e %.15e %.15e %.15e",
	   hostarray[4*i],
	   hostarray[4*i+1],
	   hostarray[4*i+2],
	   hostarray[4*i+3]);
    if(hostarray[4*i]<0)
      printf("error-:%d \n",i);
    else{
      if(hostarray[4*i]>20000.0)
	printf("error+:%d \n",i);
      else
	printf("\n");
    }
  }
#endif

/*print collision cnt and time*/
#if !defined(__PROCESS) && !defined(__TRACK) && !defined(__PLOT)
  unsigned sum = 0;
  for (int i=0;i<gridx;i++){
    printf("%4d\n",cnt[i]);
    sum += cnt[i];
  }
  printf("time elapsed:%g mus\n", timems*1000/sum);

  FILE *fp=NULL;
  fp = fopen("timelog","a+");
  gpuErrchk(hipMemcpy(HostMem.num_terminated_neutrons, 
		       DeviceMem.num_terminated_neutrons, 
		       sizeof(unsigned int), 
		       hipMemcpyDeviceToHost));
  fprintf(fp,"%-4d,%-4d,%-.6f,%-8d,%-4d,%-2d M\n", gridx, blockx,timems*1000/sum, *HostMem.num_terminated_neutrons, devstep, num_src/1000000);
  fclose(fp);
#endif
  //hipEventRecord(stop, 0);
  //hipEventSynchronize(stop);
  //hipEventElapsedTime(&timems, start, stop);

  gpuErrchk(hipEventDestroy(start));
  gpuErrchk(hipEventDestroy(stop));

  gpuErrchk(hipFree(devicearray));
  gpuErrchk(hipFree(DeviceMem.nInfo.id));
  gpuErrchk(hipFree(DeviceMem.nInfo.rndState));
  gpuErrchk(hipFree(DeviceMem.nInfo.energy));
  gpuErrchk(hipFree(DeviceMem.num_terminated_neutrons));
  gpuErrchk(hipFree(DeviceMem.block_terminated_neutrons));
  gpuErrchk(hipFree(DeviceMem.tally.cnt));
  gpuErrchk(hipFree(blockcnt));
#if defined(__QUICKW)
  gpuErrchk(hipFree(wtable));
#endif
#if defined(__FOURIERW)
  gpuErrchk(hipFree(da));
  gpuErrchk(hipFree(db));
#endif
#if defined(__INTERPEXP)
  gpuErrchk(hipFree(exptable));
#endif
 U238.release_pointer();

  free(hostarray);
  free(cnt);
  free(HostMem.num_terminated_neutrons);

  res = cudppDestroyPlan(sortplan);
  if (CUDPP_SUCCESS != res)
  {
      printf("Error destroying CUDPPPlan\n");
      exit(-1);
  }
  // shut down the CUDPP library
  cudppDestroy(theCudpp);


  return;
}


void printdevice(){
  hipDeviceProp_t prop; 
  int count;
  hipGetDeviceCount(&count);
  printf("num of devices=%d\n",count);
  for (int i=0; i<count; i++){
    hipGetDeviceProperties( &prop, i );
    printf( "   --- General Information for device %d ---\n", i );
    printf( "Name:  %s\n", prop.name );
    printf( "Compute capability:  %d.%d\n", prop.major, prop.minor );
    printf( "Clock rate:  %d\n", prop.clockRate );
    printf( "Device copy overlap:  " );
    if (prop.deviceOverlap)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n");
    printf( "Kernel execution timeout :  " );
    if (prop.kernelExecTimeoutEnabled)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n" );
    
    printf( "   --- Memory Information for device %d ---\n", i );
    printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
    printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
    printf( "Max mem pitch:  %ld\n", prop.memPitch );
    printf( "Texture Alignment:  %ld\n", prop.textureAlignment );
    
    printf( "   --- MP Information for device %d ---\n", i );
    printf( "Multiprocessor count:  %d\n",
	    prop.multiProcessorCount );
    printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
    printf( "Registers per mp:  %d\n", prop.regsPerBlock );
    printf( "Threads in warp:  %d\n", prop.warpSize );
    printf( "Max threads per block:  %d\n",
	    prop.maxThreadsPerBlock );
    printf( "Max thread dimensions:  (%d, %d, %d)\n",
	    prop.maxThreadsDim[0], prop.maxThreadsDim[1],
	    prop.maxThreadsDim[2] );
    printf( "Max grid dimensions:  (%d, %d, %d)\n",
	    prop.maxGridSize[0], prop.maxGridSize[1],
	    prop.maxGridSize[2] );
    printf( "\n" );
  }


}
