#include "hip/hip_runtime.h"
#include "CPUComplex.h"
#include "CComplex.h"
#include "multipole_data.h"
#include "multipole.h"
#include "simulation.h"

/*
  To compile host and device codes separately, 
  this "main" file works as interface 
  allocating device memory, transfering data and partitioning computation sources
*/


#if defined (__QUICKW)
#include "QuickW.h"
#endif

#if defined (__FOURIERW)
#include "fourierw.h"
__constant__ CMPTYPE a[M+1];
__constant__ CMPTYPE b[M+1];
#endif

#if defined (__QUICKWC) || defined(__INTERPEXP)
__constant__ CMPTYPE2 constwtable[LENGTH*LENGTH];
#endif


void printdevice();

void anyvalue(struct multipoledata data, unsigned setgridx, unsigned setblockx, unsigned num_src, unsigned devstep){
  unsigned gridx, blockx, gridsize;
  unsigned ints=0, sharedmem;
  float timems = 0.0;
  unsigned *cnt, *blockcnt;
  unsigned int active,i;
  CMPTYPE *hostarray, *devicearray;
  MemStruct HostMem, DeviceMem;
  hipEvent_t start, stop;
  //printdevice();
  gpuErrchk(hipEventCreate(&start));
  gpuErrchk(hipEventCreate(&stop));
  gridx = setgridx;
  blockx = setblockx;
  dim3 dimBlock(gridx, 1);
  dim3 dimGrid(blockx, 1, 1);
  gridsize = gridx*blockx;
  gpuErrchk(hipMalloc((void**)&devicearray, 4*gridsize*sizeof(CMPTYPE)));
  gpuErrchk(hipMemset(devicearray, 0, 4*gridsize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&(DeviceMem.nInfo), gridsize*sizeof(NeutronInfoStruct)));
  gpuErrchk(hipMalloc((void**)&(DeviceMem.thread_active), gridsize*sizeof(unsigned int)));
  HostMem.thread_active = (unsigned int *)malloc(gridsize*sizeof(unsigned int));
  gpuErrchk(hipMalloc((void**)&(DeviceMem.num_terminated_neutrons), sizeof(unsigned int)));
  HostMem.num_terminated_neutrons = (unsigned int *)malloc(sizeof(unsigned int));
  HostMem.num_terminated_neutrons[0] = 0u;
  gpuErrchk(hipMemcpy(DeviceMem.num_terminated_neutrons, HostMem.num_terminated_neutrons, sizeof(unsigned int), hipMemcpyHostToDevice));
  gpuErrchk(hipMalloc((void**)&(DeviceMem.tally), gridsize*sizeof(TallyStruct)));
  gpuErrchk(hipMalloc((void**)&(blockcnt), gridx*sizeof(unsigned int)));
  hostarray = (CMPTYPE*)malloc(4*gridsize*sizeof(CMPTYPE));
  cnt      = (unsigned*)malloc(gridx*sizeof(unsigned));

// construct coefficients a[n] for fourier expansion w
#if defined(__FOURIERW)
  CMPTYPE *da;
  CMPTYPE *db;
  gpuErrchk(hipMalloc((void**)&da, (M+1)*sizeof(CMPTYPE))); 
  gpuErrchk(hipMalloc((void**)&db, (M+1)*sizeof(CMPTYPE))); 
  fill_a<<<1,M+1>>>(da,db); 
  hipMemcpyToSymbol(HIP_SYMBOL(a), da, M*sizeof(CMPTYPE), 0, hipMemcpyDeviceToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(b), db, M*sizeof(CMPTYPE), 0, hipMemcpyDeviceToDevice);
#endif

// fill w function table for quickw
#if defined(__QUICKW)
  CComplex<CMPTYPE> *wtable;
  gpuErrchk(hipMalloc((void**)&wtable, LENGTH*LENGTH * 2 * sizeof(CMPTYPE)));
  fill_w_tabulated<<<LENGTH,LENGTH>>>(wtable);
#if defined(__QUICKWC)
  hipMemcpyToSymbol(HIP_SYMBOL(constwtable), wtable, LENGTH*LENGTH*2*sizeof(CMPTYPE), 0, hipMemcpyDeviceToDevice);
#endif
#if defined(__QUICKWT)
  bindwtable(wtable);
#endif
#endif

#if defined(__QUICKWG)
  multipole U238(data, wtable);
#else
  multipole U238(data);
#endif 

// fill exp(z) table for fourierw
#if defined(__INTERPEXP)
  CComplex<CMPTYPE> *exptable;
  gpuErrchk(hipMalloc((void**)&exptable, LENGTH*LENGTH * 2 * sizeof(CMPTYPE)));
  fill_exp_table<<<LENGTH,LENGTH>>>(exptable);
  hipMemcpyToSymbol(HIP_SYMBOL(constwtable), exptable, LENGTH*LENGTH*2*sizeof(CMPTYPE), 0, hipMemcpyDeviceToDevice);
#endif
  initialize<<<dimBlock, dimGrid>>>(DeviceMem, STARTENE);//1.95093e4);
  //  hipDeviceSynchronize();
  /*
    Note: shared memory size is in unit of Bybe
    And the address can be referred in form of p = pshared + offset
  */
  gpuErrchk(hipEventRecord(start, 0));

#if defined(__PROCESS) //|| defined(__TRACK)
  active = 0u;
#else
  active = 1u;
#endif

  while (active){
#if defined(__TRACK)
    history<<<dimBlock, dimGrid>>>(U238, devicearray, DeviceMem, num_src, devstep);
#else
    history<<<dimBlock, dimGrid>>>(U238, DeviceMem, num_src, devstep);
#endif
    gpuErrchk(hipMemcpy(HostMem.thread_active, DeviceMem.thread_active, gridsize*sizeof(unsigned int), hipMemcpyDeviceToHost));
    active = 0u;
    for (i = 0; i < blockx; i++){
      active += HostMem.thread_active[i];
    }
  }

  remaining<<<dimBlock, dimGrid>>>(U238, devicearray, DeviceMem);

  gpuErrchk(hipEventRecord(stop, 0));
  gpuErrchk(hipEventSynchronize(stop));
  gpuErrchk(hipEventElapsedTime(&timems, start, stop));

  printf("time elapsed:%3.1f ms\n", timems);
 
  gpuErrchk(hipMemcpy(hostarray, devicearray, 4*gridsize*sizeof(CMPTYPE), hipMemcpyDeviceToHost));

  
  ints = blockx;
  sharedmem = ints*sizeof(int);
  statistics<<<dimBlock, dimGrid, sharedmem>>>(DeviceMem.tally, blockcnt);
  gpuErrchk(hipMemcpy(cnt, blockcnt, gridx*sizeof(unsigned), hipMemcpyDeviceToHost));

/*print energy & XS (energies for __TRACK)*/
#if !defined(__PLOT)
  for(int i=0;i<gridsize;i++){
    printf(" %.15e %.15e %.15e %.15e",
	   hostarray[4*i],
	   hostarray[4*i+1],
	   hostarray[4*i+2],
	   hostarray[4*i+3]);
    if(hostarray[4*i]<0)
      printf("error-:%d \n",i);
    else{
      if(hostarray[4*i]>20000.0)
	printf("error+:%d \n",i);
      else
	printf("\n");
    }
  }
#endif

/*print collision cnt and time*/
#if !defined(__PROCESS) && !defined(__TRACK) && !defined(__PLOT)
  unsigned sum = 0;
  for (int i=0;i<gridx;i++){
    printf("%4d\n",cnt[i]);
    sum += cnt[i];
  }
  printf("time elapsed:%g mus\n", timems*1000/sum);

  FILE *fp=NULL;
  fp = fopen("timelog","a+");
  gpuErrchk(hipMemcpy(HostMem.num_terminated_neutrons, 
		       DeviceMem.num_terminated_neutrons, 
		       sizeof(unsigned int), 
		       hipMemcpyDeviceToHost));
  fprintf(fp,"%-4d,%-4d,%-.6f,%-8d,%-4d,%-2d M\n", gridx, blockx,timems*1000/sum, *HostMem.num_terminated_neutrons, devstep, num_src/1000000);
  fclose(fp);
#endif
  //hipEventRecord(stop, 0);
  //hipEventSynchronize(stop);
  //hipEventElapsedTime(&timems, start, stop);

  gpuErrchk(hipEventDestroy(start));
  gpuErrchk(hipEventDestroy(stop));

  gpuErrchk(hipFree(devicearray));
  gpuErrchk(hipFree(DeviceMem.nInfo));
  gpuErrchk(hipFree(DeviceMem.thread_active));
  gpuErrchk(hipFree(DeviceMem.tally));
#if defined(__QUICKW)
  gpuErrchk(hipFree(wtable));
#endif
#if defined(__FOURIERW)
  gpuErrchk(hipFree(da));
  gpuErrchk(hipFree(db));
#endif
  U238.release_pointer();

  free(hostarray);
  free(cnt);
  free(HostMem.thread_active);
  free(HostMem.num_terminated_neutrons);
  return;
}


void printdevice(){
  hipDeviceProp_t prop; 
  int count;
  hipGetDeviceCount(&count);
  printf("num of devices=%d\n",count);
  for (int i=0; i<count; i++){
    hipGetDeviceProperties( &prop, i );
    printf( "   --- General Information for device %d ---\n", i );
    printf( "Name:  %s\n", prop.name );
    printf( "Compute capability:  %d.%d\n", prop.major, prop.minor );
    printf( "Clock rate:  %d\n", prop.clockRate );
    printf( "Device copy overlap:  " );
    if (prop.deviceOverlap)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n");
    printf( "Kernel execution timeout :  " );
    if (prop.kernelExecTimeoutEnabled)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n" );
    
    printf( "   --- Memory Information for device %d ---\n", i );
    printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
    printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
    printf( "Max mem pitch:  %ld\n", prop.memPitch );
    printf( "Texture Alignment:  %ld\n", prop.textureAlignment );
    
    printf( "   --- MP Information for device %d ---\n", i );
    printf( "Multiprocessor count:  %d\n",
	    prop.multiProcessorCount );
    printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
    printf( "Registers per mp:  %d\n", prop.regsPerBlock );
    printf( "Threads in warp:  %d\n", prop.warpSize );
    printf( "Max threads per block:  %d\n",
	    prop.maxThreadsPerBlock );
    printf( "Max thread dimensions:  (%d, %d, %d)\n",
	    prop.maxThreadsDim[0], prop.maxThreadsDim[1],
	    prop.maxThreadsDim[2] );
    printf( "Max grid dimensions:  (%d, %d, %d)\n",
	    prop.maxGridSize[0], prop.maxGridSize[1],
	    prop.maxGridSize[2] );
    printf( "\n" );
  }


}
