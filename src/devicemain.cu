#include "hip/hip_runtime.h"
#include "CPUComplex.h"
#include "CComplex.h"
#include "multipole_data.h"
#include "multipole.h"
#include "simulation.h"

/*
  To compile host and device codes separately, 
  this "main" file works as interface 
  allocating device memory, transfering data and partitioning computation sources
*/


#if defined (__QUICKWC)
#if defined(__CFLOAT)
extern __constant__ float2 table[LENGTH][LENGTH];
#else
extern __constant__ double2 table[LENGTH][LENGTH];
#endif
//extern __constant__ CMPTYPE table[LENGTH*LENGTH*2];
#endif

void printdevice();

void anyvalue(struct multipoledata data, unsigned setgridx, unsigned setblockx, unsigned num_src, unsigned devstep){
  unsigned gridx, blockx, gridsize;
  unsigned ints=0, sharedmem;
  float timems = 0.0;
  unsigned *cnt, *blockcnt;
  unsigned int active,i;
  CMPTYPE *hostarray, *devicearray;
  MemStruct HostMem, DeviceMem;
  hipEvent_t start, stop;
  gpuErrchk(hipEventCreate(&start));
  gpuErrchk(hipEventCreate(&stop));
  // printdevice();
  gridx = setgridx;
  blockx = setblockx;
  dim3 dimBlock(gridx, 1);
  dim3 dimGrid(blockx, 1, 1);
  gridsize = gridx*blockx;
  gpuErrchk(hipMalloc((void**)&devicearray, 4*gridsize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&(DeviceMem.nInfo), gridsize*sizeof(NeutronInfoStruct)));
  gpuErrchk(hipMalloc((void**)&(DeviceMem.thread_active), gridsize*sizeof(unsigned int)));
  HostMem.thread_active = (unsigned int *)malloc(gridsize*sizeof(unsigned int));
  gpuErrchk(hipMalloc((void**)&(DeviceMem.num_terminated_neutrons), sizeof(unsigned int)));
  HostMem.num_terminated_neutrons = (unsigned int *)malloc(sizeof(unsigned int));
  HostMem.num_terminated_neutrons[0] = 0u;
  gpuErrchk(hipMemcpy(DeviceMem.num_terminated_neutrons, HostMem.num_terminated_neutrons, sizeof(unsigned int), hipMemcpyHostToDevice));
  gpuErrchk(hipMalloc((void**)&(DeviceMem.tally), gridsize*sizeof(TallyStruct)));
  gpuErrchk(hipMalloc((void**)&(blockcnt), gridx*sizeof(unsigned int)));
  hostarray = (CMPTYPE*)malloc(4*gridsize*sizeof(CMPTYPE));
  cnt      = (unsigned*)malloc(gridx*sizeof(unsigned));

#if defined(__QUICKW)
  CComplex<CMPTYPE> *wtable;
  gpuErrchk(hipMalloc((void**)&wtable, LENGTH*LENGTH * 2 * sizeof(CMPTYPE)));
  initialize_table<<<LENGTH,LENGTH>>>(wtable);
#if defined(__QUICKWC)
  hipMemcpyToSymbol(HIP_SYMBOL(table), &wtable, sizeof(wtable));
  multipole U238(data);
#else
  multipole U238(data, wtable);
#endif //__QUICKWC
#else
  multipole U238(data); //host multipoledata to device
#endif 


  initialize<<<dimBlock, dimGrid>>>(DeviceMem, 20000.0);//1.95093e4);
  //  hipDeviceSynchronize();
  /*
    Note: shared memory size is in unit of Bybe
    And the address can be referred in form of p = pshared + offset
  */
  gpuErrchk(hipEventRecord(start, 0));

#if defined(__PROCESS)
  active = 0u;
#else
  active = 1u;
#endif

  while (active){
    history<<<dimBlock, dimGrid>>>(U238, DeviceMem, num_src, devstep);
    gpuErrchk(hipMemcpy(HostMem.thread_active, DeviceMem.thread_active, gridsize*sizeof(unsigned int), hipMemcpyDeviceToHost));
    active = 0u;
    for (i = 0; i < blockx; i++){
      active += HostMem.thread_active[i];
    }
  }

  remaining<<<dimBlock, dimGrid>>>(U238, devicearray, DeviceMem);

  gpuErrchk(hipEventRecord(stop, 0));
  gpuErrchk(hipEventSynchronize(stop));
  gpuErrchk(hipEventElapsedTime(&timems, start, stop));

  printf("time elapsed:%3.1f ms\n", timems);
 
  gpuErrchk(hipMemcpy(hostarray, devicearray, 4*gridsize*sizeof(CMPTYPE), hipMemcpyDeviceToHost));

  
  ints = blockx;
  sharedmem = ints*sizeof(int);
  statistics<<<dimBlock, dimGrid, sharedmem>>>(DeviceMem.tally, blockcnt);
  gpuErrchk(hipMemcpy(cnt, blockcnt, gridx*sizeof(unsigned), hipMemcpyDeviceToHost));

  for(int i=0;i<gridsize;i++){
    printf(" %.15e %.15e %.15e %.15e",
	   hostarray[4*i],
	   hostarray[4*i+1],
	   hostarray[4*i+2],
	   hostarray[4*i+3]);
    if(hostarray[4*i]<0)
      printf("error-:%d \n",i);
    else{
      if(hostarray[4*i]>=20000.0)
	printf("error+:%d \n",i);
      else
	printf("\n");
    }
  }

#if !defined(__PROCESS)
  unsigned sum = 0;
  for (int i=0;i<gridx;i++){
    printf("%4d\n",cnt[i]);
    sum += cnt[i];
  }
  printf("time elapsed:%g mus\n", timems*1000/sum);

  FILE *fp=NULL;
  fp = fopen("timelog","a+");
  gpuErrchk(hipMemcpy(HostMem.num_terminated_neutrons, 
		       DeviceMem.num_terminated_neutrons, 
		       sizeof(unsigned int), 
		       hipMemcpyDeviceToHost));
  fprintf(fp,"%-4d,%-4d,%-.6f,%-8d,%-4d,%-2d M\n", gridx, blockx,timems*1000/sum, *HostMem.num_terminated_neutrons, devstep, num_src/1000000);
  fclose(fp);
#endif
  //hipEventRecord(stop, 0);
  //hipEventSynchronize(stop);
  //hipEventElapsedTime(&timems, start, stop);

  gpuErrchk(hipEventDestroy(start));
  gpuErrchk(hipEventDestroy(stop));

  gpuErrchk(hipFree(devicearray));
  gpuErrchk(hipFree(DeviceMem.nInfo));
  gpuErrchk(hipFree(DeviceMem.thread_active));
  gpuErrchk(hipFree(DeviceMem.tally));
#if defined(__QUICKW)
  gpuErrchk(hipFree(wtable));
#endif
  U238.release_pointer();

  free(hostarray);
  free(cnt);
  free(HostMem.thread_active);
  free(HostMem.num_terminated_neutrons);
  return;
}


void printdevice(){
  hipDeviceProp_t prop; 
  int count;
  hipGetDeviceCount(&count);
  for (int i=0; i<count; i++){
    hipGetDeviceProperties( &prop, i );
    printf( "   --- General Information for device %d ---\n", i );
    printf( "Name:  %s\n", prop.name );
    printf( "Compute capability:  %d.%d\n", prop.major, prop.minor );
    printf( "Clock rate:  %d\n", prop.clockRate );
    printf( "Device copy overlap:  " );
    if (prop.deviceOverlap)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n");
    printf( "Kernel execution timeout :  " );
    if (prop.kernelExecTimeoutEnabled)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n" );
    
    printf( "   --- Memory Information for device %d ---\n", i );
    printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
    printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
    printf( "Max mem pitch:  %ld\n", prop.memPitch );
    printf( "Texture Alignment:  %ld\n", prop.textureAlignment );
    
    printf( "   --- MP Information for device %d ---\n", i );
    printf( "Multiprocessor count:  %d\n",
	    prop.multiProcessorCount );
    printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
    printf( "Registers per mp:  %d\n", prop.regsPerBlock );
    printf( "Threads in warp:  %d\n", prop.warpSize );
    printf( "Max threads per block:  %d\n",
	    prop.maxThreadsPerBlock );
    printf( "Max thread dimensions:  (%d, %d, %d)\n",
	    prop.maxThreadsDim[0], prop.maxThreadsDim[1],
	    prop.maxThreadsDim[2] );
    printf( "Max grid dimensions:  (%d, %d, %d)\n",
	    prop.maxGridSize[0], prop.maxGridSize[1],
	    prop.maxGridSize[2] );
    printf( "\n" );
  }


}
