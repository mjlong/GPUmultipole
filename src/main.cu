#include "hip/hip_runtime.h"
#include "CPUComplex.h"
#include "CComplex.h"
#include "multipole_data.h"
#include "global.h"
#include <stdio.h>
#include <string.h>

#define FILENAMELEN 20
#define MAXISOTOPES 10
/*#if defined (__QUICKWC)
#if defined(__CFLOAT)
__constant__ float2 table[LENGTH][LENGTH];
#else
__constant__ double2 table[LENGTH][LENGTH];
#endif
//__constant__ CMPTYPE table[LENGTH*LENGTH*2];
#endif
*/
extern void h5read(struct multipoledata & pole, char filename[]);
extern void anyvalue(struct multipoledata*,unsigned, unsigned, unsigned, unsigned, unsigned);
int init_data(char* input, char filenames[][FILENAMELEN]);

int main(int argc, char **argv){
  int numIso;
  char filenames[MAXISOTOPES][FILENAMELEN];
  numIso = init_data(argv[5],filenames);
  struct multipoledata *isotopes;
  isotopes = (struct multipoledata*)malloc(sizeof(struct multipoledata)*numIso);
  for(int i=0;i<numIso;i++)  
    h5read(isotopes[i],filenames[i]);

  anyvalue(isotopes,numIso, atoi(argv[1]),atoi(argv[2]),atoi(argv[3]),atoi(argv[4]));
  free(isotopes);
 
  return 0;
}

int init_data(char* input, char line[][FILENAMELEN]){
  int numIso=-1;
  FILE *fp = NULL;
  fp = fopen(input,"r");

  if (fp == NULL) {
    fprintf(stderr, "Can't open input file %s!\n",input);
    exit(1);
  }
  int i;
  while(fgets(line[++numIso], FILENAMELEN, fp) != NULL) {
    i=-1;
    printf("%s", line[numIso]);
    while('\n'!=line[numIso][++i]){}
    line[numIso][i]='\0';
    printf("\n"); 
  }
  fclose(fp);
  return numIso;
}
