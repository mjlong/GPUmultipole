#include "hip/hip_runtime.h"
#include "CPUComplex.h"
#include "CComplex.h"
#include "multipole_data.h"
#include "global.h"

#if defined (__QUICKWC)
#if defined(__CFLOAT)
__constant__ float2 table[LENGTH*LENGTH];
#else
__constant__ double2 table[LENGTH*LENGTH];
#endif
//__constant__ CMPTYPE table[LENGTH*LENGTH*2];
#endif

extern void h5read(struct multipoledata & pole, char filename[]);
extern void anyvalue(struct multipoledata, unsigned, unsigned, unsigned, unsigned);
int main(int argc, char *argv[]){
  char h5filename[] = "092238.h5";
  struct multipoledata U238data;
  h5read(U238data, h5filename);
  anyvalue(U238data, atoi(argv[1]),atoi(argv[2]),atoi(argv[3]),atoi(argv[4]));
  return 0;
}
