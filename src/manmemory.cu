#include "manmemory.h"
__constant__ float wdspp[9];
//Simulation memory allocate and deallocate
void initialize_device(){
  hipSetDevice(0);
  gpuErrchk(hipSetDeviceFlags(hipDeviceMapHost | hipDeviceLmemResizeToMax));
}

void copymeans(int *h_cnt, int *batcnt, unsigned meshes, unsigned offset){
  for(int im=0;im<meshes;im++)
    batcnt[offset+im] = h_cnt[im];

}

void copydata(MemStruct DeviceMem, MemStruct HostMem){
  gpuErrchk(hipMemcpy(DeviceMem.wdspp,  HostMem.wdspp,   sizeof(float)*9, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(wdspp), DeviceMem.wdspp, 9*sizeof(float), 0, hipMemcpyDeviceToDevice));
}
void initialize_memory(MemStruct *DeviceMem, MemStruct *HostMem, unsigned numbins, unsigned gridx, unsigned blockx,unsigned ubat,unsigned gridr,unsigned blockr,unsigned ubatr){
  unsigned banksize,gridsizr;
  banksize = gridx*blockx*ubat;
  gridsizr = gridr*blockr;
  //for __TALLY, ubat is used as tranfac

#if defined(__TALLY)
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).spectrum), numbins*sizeof(CMPTYPE)));
  (*HostMem).spectrum = (CMPTYPE*)malloc(sizeof(CMPTYPE)*numbins);  
  (*HostMem).batcnt     = (CMPTYPE*)malloc(sizeof(CMPTYPE)*numbins);
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).batcnt), numbins*sizeof(CMPTYPE)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).block_spectrum), numbins*gridr*sizeof(CMPTYPE)));
  gpuErrchk(hipMemset((*DeviceMem).block_spectrum, 0, numbins*gridr*sizeof(CMPTYPE)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).tally.cnt), gridsizr*numbins*sizeof(CMPTYPE)));
  gpuErrchk(hipMemset((*DeviceMem).tally.cnt, 0, numbins*gridsizr*sizeof(CMPTYPE)));  
#endif
  (*HostMem).wdspp = (float*)malloc(sizeof(float)*9);


  (*HostMem).nInfo.live  = (int*)malloc(sizeof(int)*banksize);

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).wdspp), 9*sizeof(float)));


  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.id),       banksize*sizeof(unsigned)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.live),       banksize*sizeof(int)));
  gpuErrchk(hipMemset((*DeviceMem).nInfo.live, 0, banksize*sizeof(int)));  

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.rndState), banksize*sizeof(hiprandState)));


#if defined(__3D)
  (*HostMem).nInfo.pos_x = (float*)malloc(sizeof(float)*banksize);
  (*HostMem).nInfo.pos_y = (float*)malloc(sizeof(float)*banksize);
  (*HostMem).nInfo.pos_z = (float*)malloc(sizeof(float)*banksize);
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_x),3*banksize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_y),3*banksize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_z),3*banksize*sizeof(float)));
#endif

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).num_terminated_neutrons), sizeof(unsigned int)));
  gpuErrchk(hipMemset((*DeviceMem).num_terminated_neutrons, 0, sizeof(unsigned)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).num_live_neutrons), sizeof(unsigned int)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).block_terminated_neutrons), sizeof(unsigned int)*gridr));
  gpuErrchk(hipHostMalloc((void**)&((*HostMem).num_terminated_neutrons), sizeof(unsigned int)));
  (*HostMem).num_terminated_neutrons[0] = 0u;


  return;
}

void resettally(CMPTYPE *cnt, unsigned totbins){
  gpuErrchk(hipMemset(cnt, 0, totbins*sizeof(CMPTYPE)));}

void release_memory(MemStruct DeviceMem, MemStruct HostMem){
#if defined(__TALLY)
  free(HostMem.spectrum);
#if defined(__PROCESS)
  free(HostMem.batchmeans);
  free(HostMem.accmeans);
#endif
  free(HostMem.batcnt);
  gpuErrchk(hipFree(DeviceMem.batcnt));
  gpuErrchk(hipFree(DeviceMem.spectrum));
  gpuErrchk(hipFree(DeviceMem.block_spectrum));
  gpuErrchk(hipFree(DeviceMem.tally.cnt));
#endif
  free(HostMem.nInfo.live);
  free(HostMem.wdspp);

  gpuErrchk(hipFree(DeviceMem.wdspp));

  gpuErrchk(hipFree(DeviceMem.nInfo.id));
  gpuErrchk(hipFree(DeviceMem.nInfo.live));
  gpuErrchk(hipFree(DeviceMem.nInfo.rndState));

#if defined(__WASTE)
  gpuErrchk(hipFree(DeviceMem.nInfo.energy));
  gpuErrchk(hipFree(DeviceMem.nInfo.sigT));
  gpuErrchk(hipFree(DeviceMem.nInfo.sigA));
  gpuErrchk(hipFree(DeviceMem.nInfo.sigF));
  gpuErrchk(hipFree(DeviceMem.nInfo.isoenergy));
  gpuErrchk(hipFree(DeviceMem.nInfo.imat));
#endif

#if defined(__1D)
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_x));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_y));
#endif

#if defined(__SCATTERPLOT)
  gpuErrchk(hipFree(DeviceMem.nInfo.energy));
  free(HostMem.nInfo.energy);
#endif
#if defined(__3D)
  free(HostMem.nInfo.pos_x);
  free(HostMem.nInfo.pos_y);
  free(HostMem.nInfo.pos_z);
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_x));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_y));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_z));
#endif

  gpuErrchk(hipFree(DeviceMem.num_terminated_neutrons));
  gpuErrchk(hipFree(DeviceMem.num_live_neutrons));
  gpuErrchk(hipFree(DeviceMem.block_terminated_neutrons));
  gpuErrchk(hipHostFree(HostMem.num_terminated_neutrons));


  return;
}

