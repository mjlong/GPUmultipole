#include "manmemory.h"

//Simulation memory allocate and deallocate
void initialize_device(){
  hipSetDevice(0);
  gpuErrchk(hipSetDeviceFlags(hipDeviceMapHost | hipDeviceLmemResizeToMax));
}


void initialize_memory(MemStruct *DeviceMem, MemStruct *HostMem, unsigned numbins, unsigned gridx, unsigned blockx ){
  unsigned gridsize;
  gridsize = gridx*blockx;

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).spectrum), numbins*sizeof(unsigned int)));
  (*HostMem).spectrum = (unsigned*)malloc(sizeof(unsigned)*numbins);  

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).block_spectrum), numbins*gridx*sizeof(unsigned int)));
  gpuErrchk(hipMemset((*DeviceMem).block_spectrum, 0, numbins*gridx*sizeof(unsigned int)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.id),       gridsize*sizeof(unsigned)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.live),       gridsize*sizeof(unsigned)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.rndState), gridsize*sizeof(hiprandState)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.energy),   gridsize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.sigT),   gridsize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.sigA),   gridsize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.sigF),   gridsize*sizeof(CMPTYPE)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.isoenergy),gridsize*sizeof(CMPTYPE)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.imat),  gridsize*sizeof(unsigned)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_x),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_y),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_z),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.dir_polar),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.dir_azimu),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.d_closest ),gridsize*sizeof(float)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).num_terminated_neutrons), sizeof(unsigned int)));
  gpuErrchk(hipMemset((*DeviceMem).num_terminated_neutrons, 0, sizeof(unsigned)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).num_live_neutrons), sizeof(unsigned int)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).block_terminated_neutrons), sizeof(unsigned int)*gridx));
  gpuErrchk(hipHostMalloc((void**)&((*HostMem).num_terminated_neutrons), sizeof(unsigned int)));
  (*HostMem).num_terminated_neutrons[0] = 0u;

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).tally.cnt), gridsize*numbins*sizeof(unsigned)));
  gpuErrchk(hipMemset((*DeviceMem).tally.cnt, 0, numbins*gridsize*sizeof(unsigned)));  

  return;
}

void release_memory(MemStruct DeviceMem, MemStruct HostMem){
  free(HostMem.spectrum);

  gpuErrchk(hipFree(DeviceMem.spectrum));
  gpuErrchk(hipFree(DeviceMem.block_spectrum));

  gpuErrchk(hipFree(DeviceMem.nInfo.id));
  gpuErrchk(hipFree(DeviceMem.nInfo.live));
  gpuErrchk(hipFree(DeviceMem.nInfo.rndState));
  gpuErrchk(hipFree(DeviceMem.nInfo.energy));
  gpuErrchk(hipFree(DeviceMem.nInfo.sigT));
  gpuErrchk(hipFree(DeviceMem.nInfo.sigA));
  gpuErrchk(hipFree(DeviceMem.nInfo.sigF));

  gpuErrchk(hipFree(DeviceMem.nInfo.isoenergy));

  gpuErrchk(hipFree(DeviceMem.nInfo.imat));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_x));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_y));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_z));
  gpuErrchk(hipFree(DeviceMem.nInfo.dir_polar));
  gpuErrchk(hipFree(DeviceMem.nInfo.dir_azimu));
  gpuErrchk(hipFree(DeviceMem.nInfo.d_closest));

  gpuErrchk(hipFree(DeviceMem.num_terminated_neutrons));
  gpuErrchk(hipFree(DeviceMem.num_live_neutrons));
  gpuErrchk(hipFree(DeviceMem.block_terminated_neutrons));
  gpuErrchk(hipHostFree(HostMem.num_terminated_neutrons));

  gpuErrchk(hipFree(DeviceMem.tally.cnt));

  return;
}

