#include "manmemory.h"

//Simulation memory allocate and deallocate
void initialize_device(){
  hipSetDevice(0);
  gpuErrchk(hipSetDeviceFlags(hipDeviceMapHost | hipDeviceLmemResizeToMax));
}

void copymeans(unsigned *h_cnt, unsigned *acccnt, unsigned meshes, unsigned offset){
  for(int im=0;im<meshes;im++)
    acccnt[offset+im] = h_cnt[im];

}

void copydata(MemStruct DeviceMem, MemStruct HostMem){
  gpuErrchk(hipMemcpy(DeviceMem.wdspp,  HostMem.wdspp,   sizeof(float)*5, hipMemcpyHostToDevice));
}
void initialize_memory(MemStruct *DeviceMem, MemStruct *HostMem, unsigned numbins, unsigned gridx, unsigned blockx,unsigned nbat){
  unsigned gridsize;
  gridsize = gridx*blockx;

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).spectrum), numbins*sizeof(unsigned int)));
  (*HostMem).spectrum = (unsigned*)malloc(sizeof(unsigned)*numbins);  
  (*HostMem).batchmeans = (float*)malloc(sizeof(float)*nbat*numbins);
  (*HostMem).accmeans   = (float*)malloc(sizeof(float)*nbat*numbins);
  (*HostMem).acccnt     = (unsigned*)malloc(sizeof(unsigned)*nbat*numbins);
  (*HostMem).wdspp = (float*)malloc(sizeof(float)*5);
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).wdspp), 5*sizeof(float)));
  
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).block_spectrum), numbins*gridx*sizeof(unsigned int)));
  gpuErrchk(hipMemset((*DeviceMem).block_spectrum, 0, numbins*gridx*sizeof(unsigned int)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.id),       gridsize*sizeof(unsigned)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.live),       gridsize*sizeof(unsigned)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.rndState), gridsize*sizeof(hiprandState)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.energy),   gridsize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.sigT),   gridsize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.sigA),   gridsize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.sigF),   gridsize*sizeof(CMPTYPE)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.isoenergy),gridsize*sizeof(CMPTYPE)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.imat),  gridsize*sizeof(unsigned)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_x),3*gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_y),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_z),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.dir_polar),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.dir_azimu),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.d_closest ),gridsize*sizeof(float)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).num_terminated_neutrons), sizeof(unsigned int)));
  gpuErrchk(hipMemset((*DeviceMem).num_terminated_neutrons, 0, sizeof(unsigned)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).num_live_neutrons), sizeof(unsigned int)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).block_terminated_neutrons), sizeof(unsigned int)*gridx));
  gpuErrchk(hipHostMalloc((void**)&((*HostMem).num_terminated_neutrons), sizeof(unsigned int)));
  (*HostMem).num_terminated_neutrons[0] = 0u;

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).tally.cnt), gridsize*numbins*sizeof(unsigned)));
  gpuErrchk(hipMemset((*DeviceMem).tally.cnt, 0, numbins*gridsize*sizeof(unsigned)));  

  return;
}

void release_memory(MemStruct DeviceMem, MemStruct HostMem){
  free(HostMem.spectrum);
  free(HostMem.batchmeans);
  free(HostMem.accmeans);
  free(HostMem.acccnt);
  free(HostMem.wdspp);

  gpuErrchk(hipFree(DeviceMem.wdspp));
  gpuErrchk(hipFree(DeviceMem.spectrum));
  gpuErrchk(hipFree(DeviceMem.block_spectrum));

  gpuErrchk(hipFree(DeviceMem.nInfo.id));
  gpuErrchk(hipFree(DeviceMem.nInfo.live));
  gpuErrchk(hipFree(DeviceMem.nInfo.rndState));
  gpuErrchk(hipFree(DeviceMem.nInfo.energy));
  gpuErrchk(hipFree(DeviceMem.nInfo.sigT));
  gpuErrchk(hipFree(DeviceMem.nInfo.sigA));
  gpuErrchk(hipFree(DeviceMem.nInfo.sigF));

  gpuErrchk(hipFree(DeviceMem.nInfo.isoenergy));

  gpuErrchk(hipFree(DeviceMem.nInfo.imat));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_x));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_y));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_z));
  gpuErrchk(hipFree(DeviceMem.nInfo.dir_polar));
  gpuErrchk(hipFree(DeviceMem.nInfo.dir_azimu));
  gpuErrchk(hipFree(DeviceMem.nInfo.d_closest));

  gpuErrchk(hipFree(DeviceMem.num_terminated_neutrons));
  gpuErrchk(hipFree(DeviceMem.num_live_neutrons));
  gpuErrchk(hipFree(DeviceMem.block_terminated_neutrons));
  gpuErrchk(hipHostFree(HostMem.num_terminated_neutrons));

  gpuErrchk(hipFree(DeviceMem.tally.cnt));

  return;
}

