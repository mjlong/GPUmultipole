#include "manmemory.h"

//Simulation memory allocate and deallocate

void initialize_memory(MemStruct *DeviceMem, MemStruct *HostMem, CMPTYPE** devicearray, CMPTYPE** hostarray, unsigned **cnt, unsigned** blockcnt, unsigned gridx, unsigned blockx ){
  unsigned gridsize;
  gridsize = gridx*blockx;

  *hostarray = (CMPTYPE*)malloc(4*gridsize*sizeof(CMPTYPE));
  *cnt      = (unsigned*)malloc(gridx*sizeof(unsigned));

  gpuErrchk(hipMalloc((void**)devicearray, 4*gridsize*sizeof(CMPTYPE)));
  gpuErrchk(hipMemset(*devicearray, 0, 4*gridsize*sizeof(CMPTYPE)));

  gpuErrchk(hipMalloc((void**)(blockcnt), gridx*sizeof(unsigned int)));
  gpuErrchk(hipMemset(*blockcnt, 0, gridx*sizeof(unsigned int)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.id),       gridsize*sizeof(unsigned)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.rndState), gridsize*sizeof(hiprandState)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.energy),   gridsize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.isotope),  gridsize*sizeof(unsigned)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.isoenergy),gridsize*sizeof(CMPTYPE)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.icell),  gridsize*sizeof(unsigned)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_x),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_y),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_z),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.dir_polar),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.dir_azimu),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.d_closest ),gridsize*sizeof(float)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).num_terminated_neutrons), sizeof(unsigned int)));
  gpuErrchk(hipMemset((*DeviceMem).num_terminated_neutrons, 0, sizeof(unsigned)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).block_terminated_neutrons), sizeof(unsigned int)*gridx));
  gpuErrchk(hipHostMalloc((void**)&((*HostMem).num_terminated_neutrons), sizeof(unsigned int)));
  (*HostMem).num_terminated_neutrons[0] = 0u;

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).tally.cnt), gridsize*sizeof(unsigned)));
  gpuErrchk(hipMemset((*DeviceMem).tally.cnt, 0, gridsize*sizeof(unsigned)));  


  return;
}

void release_memory(MemStruct DeviceMem, MemStruct HostMem, CMPTYPE* devicearray, CMPTYPE* hostarray, unsigned *cnt, unsigned* blockcnt ){
  free(hostarray);
  free(cnt);
  gpuErrchk(hipFree(devicearray));
  gpuErrchk(hipFree(blockcnt));

  gpuErrchk(hipFree(DeviceMem.nInfo.id));
  gpuErrchk(hipFree(DeviceMem.nInfo.rndState));
  gpuErrchk(hipFree(DeviceMem.nInfo.energy));
  gpuErrchk(hipFree(DeviceMem.nInfo.isotope));
  gpuErrchk(hipFree(DeviceMem.nInfo.isoenergy));

  gpuErrchk(hipFree(DeviceMem.nInfo.icell));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_x));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_y));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_z));
  gpuErrchk(hipFree(DeviceMem.nInfo.dir_polar));
  gpuErrchk(hipFree(DeviceMem.nInfo.dir_azimu));
  gpuErrchk(hipFree(DeviceMem.nInfo.d_closest));

  gpuErrchk(hipFree(DeviceMem.num_terminated_neutrons));
  gpuErrchk(hipFree(DeviceMem.block_terminated_neutrons));
  gpuErrchk(hipHostFree(HostMem.num_terminated_neutrons));

  gpuErrchk(hipFree(DeviceMem.tally.cnt));
  return;
}

