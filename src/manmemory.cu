#include "manmemory.h"

//Simulation memory allocate and deallocate
void initialize_device(){
  hipSetDevice(0);
  gpuErrchk(hipSetDeviceFlags(hipDeviceMapHost | hipDeviceLmemResizeToMax));
}

void copymeans(int *h_cnt, int *batcnt, unsigned meshes, unsigned offset){
  for(int im=0;im<meshes;im++)
#if defined(__TRAN)
    batcnt[offset+im] += h_cnt[im];
#else
    batcnt[offset+im] = h_cnt[im];
#endif

}

void copydata(MemStruct DeviceMem, MemStruct HostMem){
  gpuErrchk(hipMemcpy(DeviceMem.wdspp,  HostMem.wdspp,   sizeof(float)*5, hipMemcpyHostToDevice));
}
void initialize_memory(MemStruct *DeviceMem, MemStruct *HostMem, unsigned numbins, unsigned gridx, unsigned blockx,unsigned nbat,unsigned ubat){
  unsigned gridsize;
  gridsize = gridx*blockx;

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).spectrum), numbins*sizeof(int)));
  (*HostMem).spectrum = (int*)malloc(sizeof(int)*numbins);  
  (*HostMem).batchmeans = (double*)malloc(sizeof(double)*nbat*numbins);
  (*HostMem).accmeans   = (double*)malloc(sizeof(double)*(nbat-ubat)*numbins);
  (*HostMem).batcnt     = (int*)malloc(sizeof(int)*nbat*numbins);
#if defined(__TRAN)
  memset((*HostMem).batcnt, 0, sizeof(int)*nbat*numbins);
#endif
  (*HostMem).wdspp = (float*)malloc(sizeof(float)*5);


  (*HostMem).nInfo.live  = (int*)malloc(sizeof(int)*gridsize);

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).wdspp), 5*sizeof(float)));
  
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).block_spectrum), numbins*gridx*sizeof(unsigned int)));
  gpuErrchk(hipMemset((*DeviceMem).block_spectrum, 0, numbins*gridx*sizeof(unsigned int)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.id),       gridsize*sizeof(unsigned)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.live),       gridsize*sizeof(unsigned)));
  gpuErrchk(hipMemset((*DeviceMem).nInfo.live, 0, gridsize*sizeof(int)));  

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.rndState), gridsize*sizeof(hiprandState)));

#if defined(__WASTE)
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.energy),   gridsize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.sigT),   gridsize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.sigA),   gridsize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.sigF),   gridsize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.isoenergy),gridsize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.imat),  gridsize*sizeof(int)));
#endif

#if defined(__1D)
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_x),3*gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_y),gridsize*sizeof(float)));
#endif 

#if defined(__3D)
  (*HostMem).nInfo.pos_x = (float*)malloc(sizeof(float)*gridsize);
  (*HostMem).nInfo.pos_y = (float*)malloc(sizeof(float)*gridsize);
  (*HostMem).nInfo.pos_z = (float*)malloc(sizeof(float)*gridsize);
  (*HostMem).nInfo.dir_polar = (float*)malloc(sizeof(float)*gridsize);
  (*HostMem).nInfo.dir_azimu = (float*)malloc(sizeof(float)*gridsize);
  (*HostMem).nInfo.d_closest = (float*)malloc(sizeof(float)*gridsize);

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_x),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_y),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_z),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.dir_polar),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.dir_azimu),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.d_closest ),gridsize*sizeof(float)));
  gpuErrchk(hipMemset((*DeviceMem).nInfo.d_closest, 0, gridsize*sizeof(float)));  //use as time
#endif

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).num_terminated_neutrons), sizeof(unsigned int)));
  gpuErrchk(hipMemset((*DeviceMem).num_terminated_neutrons, 0, sizeof(unsigned)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).num_live_neutrons), sizeof(unsigned int)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).block_terminated_neutrons), sizeof(unsigned int)*gridx));
  gpuErrchk(hipHostMalloc((void**)&((*HostMem).num_terminated_neutrons), sizeof(unsigned int)));
  (*HostMem).num_terminated_neutrons[0] = 0u;

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).tally.cnt), gridsize*numbins*sizeof(int)));
  gpuErrchk(hipMemset((*DeviceMem).tally.cnt, 0, numbins*gridsize*sizeof(int)));  

  return;
}

void resettally(int *cnt, unsigned totbins){
  gpuErrchk(hipMemset(cnt, 0, totbins*sizeof(int)));}

void release_memory(MemStruct DeviceMem, MemStruct HostMem){
  free(HostMem.nInfo.live);
  free(HostMem.spectrum);
  free(HostMem.batchmeans);
  free(HostMem.accmeans);
  free(HostMem.batcnt);
  free(HostMem.wdspp);

  gpuErrchk(hipFree(DeviceMem.wdspp));
  gpuErrchk(hipFree(DeviceMem.spectrum));
  gpuErrchk(hipFree(DeviceMem.block_spectrum));

  gpuErrchk(hipFree(DeviceMem.nInfo.id));
  gpuErrchk(hipFree(DeviceMem.nInfo.live));
  gpuErrchk(hipFree(DeviceMem.nInfo.rndState));

#if defined(__WASTE)
  gpuErrchk(hipFree(DeviceMem.nInfo.energy));
  gpuErrchk(hipFree(DeviceMem.nInfo.sigT));
  gpuErrchk(hipFree(DeviceMem.nInfo.sigA));
  gpuErrchk(hipFree(DeviceMem.nInfo.sigF));
  gpuErrchk(hipFree(DeviceMem.nInfo.isoenergy));
  gpuErrchk(hipFree(DeviceMem.nInfo.imat));
#endif

#if defined(__1D)
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_x));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_y));
#endif

#if defined(__3D)
  free(HostMem.nInfo.pos_x);
  free(HostMem.nInfo.pos_y);
  free(HostMem.nInfo.pos_z);
  free(HostMem.nInfo.dir_polar);
  free(HostMem.nInfo.dir_azimu);
  free(HostMem.nInfo.d_closest);

  gpuErrchk(hipFree(DeviceMem.nInfo.pos_x));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_y));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_z));
  gpuErrchk(hipFree(DeviceMem.nInfo.dir_polar));
  gpuErrchk(hipFree(DeviceMem.nInfo.dir_azimu));
  gpuErrchk(hipFree(DeviceMem.nInfo.d_closest));
#endif

  gpuErrchk(hipFree(DeviceMem.num_terminated_neutrons));
  gpuErrchk(hipFree(DeviceMem.num_live_neutrons));
  gpuErrchk(hipFree(DeviceMem.block_terminated_neutrons));
  gpuErrchk(hipHostFree(HostMem.num_terminated_neutrons));

  gpuErrchk(hipFree(DeviceMem.tally.cnt));

  return;
}

