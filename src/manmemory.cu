#include "manmemory.h"
__constant__ float wdspp[9];
//Simulation memory allocate and deallocate
void initialize_device(){
  hipSetDevice(0);
  gpuErrchk(hipSetDeviceFlags(hipDeviceMapHost | hipDeviceLmemResizeToMax));
}

void copymeans(int *h_cnt, int *batcnt, unsigned meshes, unsigned offset){
  for(int im=0;im<meshes;im++)
    batcnt[offset+im] = h_cnt[im];

}


void initialize_memory_data(MemStruct *DeviceMem, MemStruct *HostMem){
  (*HostMem).wdspp = (float*)malloc(sizeof(float)*9);
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).wdspp), 9*sizeof(float)));
}
void copydata(MemStruct DeviceMem, MemStruct HostMem){
  gpuErrchk(hipMemcpy(DeviceMem.wdspp,  HostMem.wdspp,   sizeof(float)*9, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(wdspp), DeviceMem.wdspp, 9*sizeof(float), 0, hipMemcpyDeviceToDevice));
}
void release_memory_data(MemStruct DeviceMem, MemStruct HostMem){
  free(HostMem.wdspp);
  gpuErrchk(hipFree(DeviceMem.wdspp));
}
//==============================================================================
//=====memory_converge() allocates memory for the phase of source convergence===
//1. memory_converge() need not have tally arrays, which exist current for debug
//==============================================================================
void allocate_memory_converge(MemStruct *DeviceMem, MemStruct *HostMem, unsigned numbins, unsigned gridx, unsigned blockx,unsigned num_seg){
  unsigned gridsize,banksize;
  gridsize = gridx*blockx;
  banksize = gridx*blockx*num_seg;

#if defined(__TALLY)
#if defined(__FTALLY)||(__FTALLY2) //Fission source tally
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.imat),  banksize*3*sizeof(int)));
  (*HostMem).nInfo.live  = (int*)malloc(sizeof(int)*banksize);
  (*HostMem).batcnt     = (CMPTYPE*)malloc(sizeof(CMPTYPE)*numbins);
  memset((*HostMem).batcnt, 0, sizeof(CMPTYPE)*numbins);
#endif
#if defined(__CTALLY) //Collision density (flux) tally
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).spectrum), numbins*sizeof(CMPTYPE)));
  (*HostMem).spectrum = (CMPTYPE*)malloc(sizeof(CMPTYPE)*numbins);  

  (*HostMem).batcnt     = (CMPTYPE*)malloc(sizeof(CMPTYPE)*numbins);
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).batcnt), numbins*sizeof(CMPTYPE)));
#if defined(__CTALLY2)
  (*HostMem).batcnt2    = (CMPTYPE*)malloc(sizeof(CMPTYPE)*numbins);
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).batcnt2),numbins*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).cnt2_t),numbins*gridsize*sizeof(int)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).tally.cnt2), gridsize*numbins*sizeof(CMPTYPE)));
  gpuErrchk(hipMemset((*DeviceMem).tally.cnt2,0, numbins*gridsize*sizeof(CMPTYPE)));  
#endif
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).block_spectrum), numbins*gridx*sizeof(CMPTYPE)));
  gpuErrchk(hipMemset((*DeviceMem).block_spectrum, 0, numbins*gridx*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).tally.cnt), gridsize*numbins*sizeof(CMPTYPE)));
  gpuErrchk(hipMemset((*DeviceMem).tally.cnt, 0, numbins*gridsize*sizeof(CMPTYPE)));  
#endif

#endif

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.live),       banksize*sizeof(int)));
  gpuErrchk(hipMemset((*DeviceMem).nInfo.live, 0, banksize*sizeof(int)));  

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.rndState), gridsize*sizeof(hiprandState)));

#if defined(__1D)
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_x),3*banksize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_y),banksize*sizeof(float)));
#endif 

#if defined(__3D)
  (*HostMem).nInfo.pos_x = (float*)malloc(sizeof(float)*banksize);
  (*HostMem).nInfo.pos_y = (float*)malloc(sizeof(float)*banksize);
  (*HostMem).nInfo.pos_z = (float*)malloc(sizeof(float)*banksize);
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_x),3*banksize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_y),3*banksize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_z),3*banksize*sizeof(float)));
#endif

  return;
}


void release_memory_converge(MemStruct DeviceMem, MemStruct HostMem){
#if defined(__TALLY)
#if defined(__FTALLY)||(__FTALLY2) //Fission source tally
  gpuErrchk(hipFree(DeviceMem.nInfo.imat));
  free(HostMem.nInfo.live);
  free(HostMem.batcnt);
#endif
#if defined(__CTALLY) //Collision density (flux) tally
  gpuErrchk(hipFree(DeviceMem.spectrum));
  free(HostMem.spectrum);
  free(HostMem.spectrum);  

  free(HostMem.batcnt);
  gpuErrchk(hipFree(DeviceMem.batcnt));
#if defined(__CTALLY2)
  free(HostMem.batcnt2);
  gpuErrchk(hipFree(DeviceMem.batcnt2));
  gpuErrchk(hipFree(DeviceMem.cnt2_t));
  gpuErrchk(hipFree(DeviceMem.tally.cnt2));
#endif
  gpuErrchk(hipFree(DeviceMem.block_spectrum));
  gpuErrchk(hipFree(DeviceMem.tally.cnt));
#endif
#endif
  gpuErrchk(hipFree(DeviceMem.nInfo.live));
  gpuErrchk(hipFree(DeviceMem.nInfo.rndState));

#if defined(__1D)
  gpuErrchk(hipFree(DeviceMem).nInfo.pos_x),3*banksize*sizeof(float)));
  gpuErrchk(hipFree(DeviceMem).nInfo.pos_y),banksize*sizeof(float)));
#endif 

#if defined(__3D)
  free(HostMem.nInfo.pos_x);
  free(HostMem.nInfo.pos_y);
  free(HostMem.nInfo.pos_z);
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_x));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_y));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_z));
#endif
  return;
}


//==============================================================================
//====================initialize memory for  delayed bank ======================
//==============================================================================
void initialize_memory_bank(MemStruct *HostMem, unsigned banksize){
  (*HostMem).bank.x = (float*)malloc(sizeof(float)*banksize);
  (*HostMem).bank.y = (float*)malloc(sizeof(float)*banksize);
  (*HostMem).bank.z = (float*)malloc(sizeof(float)*banksize);
  (*HostMem).bank.available = (unsigned*)malloc(sizeof(unsigned)*banksize);

  (*HostMem).bank.size   = (unsigned*)malloc(sizeof(unsigned));
  (*HostMem).bank.cursor_end       = (unsigned*)malloc(sizeof(unsigned));

  memset((*HostMem).bank.cursor_available, 0, sizeof(unsigned)*banksize);
  ((*HostMem).bank.size)[0]    = banksize;
  ((*HostMem).bank.cursor_end)[0] = 0; 
}

void release_memory_bank(MemStruct HostMem){
  free(HostMem.bank.x);
  free(HostMem.bank.y);
  free(HostMem.bank.z);

  free(HostMem.bank.size);
  free(HostMem.bank.cursor_end);
  free(HostMem.bank.cursor_available);
}


//==============================================================================
//=====memory_active() allocates memory for the phase of source convergence===
//1. memory_active() takes num_seg as parameter, which varies with phases ====
//2. memory_active() have tally array
//==============================================================================
void allocate_memory_active(MemStruct *DeviceMem, MemStruct *HostMem, unsigned numbins, unsigned gridx, unsigned blockx,unsigned num_seg){
  unsigned gridsize,banksize;
  gridsize = gridx*blockx;
  banksize = gridx*blockx*num_seg;

#if defined(__TALLY)
#if defined(__FTALLY)||(__FTALLY2) //Fission source tally
#if defined(__FTALLY)
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.imat),  banksize*3*sizeof(int)));
  (*HostMem).nInfo.live  = (int*)malloc(sizeof(int)*banksize);
#else //(__FTALLY2)
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.imat),  gridsize*3*sizeof(int)));
  (*HostMem).nInfo.live  = (int*)malloc(sizeof(int)*gridsize);
#endif
  (*HostMem).batcnt     = (CMPTYPE*)malloc(sizeof(CMPTYPE)*numbins);
  memset((*HostMem).batcnt, 0, sizeof(CMPTYPE)*numbins);
#endif
#if defined(__CTALLY) //Collision density (flux) tally
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).spectrum), numbins*sizeof(CMPTYPE)));
  (*HostMem).spectrum = (CMPTYPE*)malloc(sizeof(CMPTYPE)*numbins);  

  (*HostMem).batcnt     = (CMPTYPE*)malloc(sizeof(CMPTYPE)*numbins);
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).batcnt), numbins*sizeof(CMPTYPE)));
#if defined(__CTALLY2)
  (*HostMem).batcnt2    = (CMPTYPE*)malloc(sizeof(CMPTYPE)*numbins);
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).batcnt2),numbins*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).cnt2_t),numbins*gridsize*sizeof(int)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).tally.cnt2), gridsize*numbins*sizeof(CMPTYPE)));
  gpuErrchk(hipMemset((*DeviceMem).tally.cnt2,0, numbins*gridsize*sizeof(CMPTYPE)));  
#endif
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).block_spectrum), numbins*gridx*sizeof(CMPTYPE)));
  gpuErrchk(hipMemset((*DeviceMem).block_spectrum, 0, numbins*gridx*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).tally.cnt), gridsize*numbins*sizeof(CMPTYPE)));
  gpuErrchk(hipMemset((*DeviceMem).tally.cnt, 0, numbins*gridsize*sizeof(CMPTYPE)));  
#endif
#endif

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.live),       banksize*sizeof(int)));
  gpuErrchk(hipMemset((*DeviceMem).nInfo.live, 0, banksize*sizeof(int)));  

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.rndState), gridsize*sizeof(hiprandState)));

#if defined(__1D)
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_x),3*banksize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_y),banksize*sizeof(float)));
#endif 

#if defined(__3D)
#if defined(__FTALLY2)
  (*HostMem).nInfo.pos_x = (float*)malloc(sizeof(float)*gridsize);
  (*HostMem).nInfo.pos_y = (float*)malloc(sizeof(float)*gridsize);
  (*HostMem).nInfo.pos_z = (float*)malloc(sizeof(float)*gridsize);
#else
  (*HostMem).nInfo.pos_x = (float*)malloc(sizeof(float)*banksize);
  (*HostMem).nInfo.pos_y = (float*)malloc(sizeof(float)*banksize);
  (*HostMem).nInfo.pos_z = (float*)malloc(sizeof(float)*banksize);
#endif
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_x),3*banksize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_y),3*banksize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_z),3*banksize*sizeof(float)));
#endif

  return;
}


void release_memory_active(MemStruct DeviceMem, MemStruct HostMem){
#if defined(__TALLY)
#if defined(__FTALLY)||(__FTALLY2) //Fission source tally
  gpuErrchk(hipFree(DeviceMem.nInfo.imat));
  free(HostMem.nInfo.live);
  free(HostMem.batcnt);
#endif
#if defined(__CTALLY) //Collision density (flux) tally
  gpuErrchk(hipFree(DeviceMem.spectrum));
  free(HostMem.spectrum);  

  free(HostMem.batcnt);
  gpuErrchk(hipFree(DeviceMem.batcnt));
#if defined(__CTALLY2)
  free(HostMem.batcnt2);
  gpuErrchk(hipFree(DeviceMem.batcnt2));
  gpuErrchk(hipFree(DeviceMem.cnt2_t));
  gpuErrchk(hipFree(DeviceMem.tally.cnt2));
#endif
  gpuErrchk(hipFree(DeviceMem.block_spectrum));
  gpuErrchk(hipFree(DeviceMem.tally.cnt));
#endif
#endif

  gpuErrchk(hipFree(DeviceMem.nInfo.live));
  gpuErrchk(hipFree(DeviceMem.nInfo.rndState));

#if defined(__1D)
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_x));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_y));
#endif 

#if defined(__3D)
  free(HostMem.nInfo.pos_x);
  free(HostMem.nInfo.pos_y);
  free(HostMem.nInfo.pos_z);
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_x));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_y));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_z));
#endif

  return;
}

void initialize_memory(MemStruct *DeviceMem, MemStruct *HostMem, unsigned numbins, unsigned gridx, unsigned blockx,unsigned nbat,unsigned ubat){
  unsigned gridsize,banksize;
  gridsize = gridx*blockx;
  banksize = gridx*blockx*ubat;
  //for __TALLY, ubat is used as tranfac

#if defined(__TALLY)
#if defined(__FTALLY)||(__FTALLY2) //Fission source tally
#if defined(__FTALLY)
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.imat),  banksize*3*sizeof(int)));
  (*HostMem).nInfo.live  = (int*)malloc(sizeof(int)*banksize);
#else //(__FTALLY2)
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.imat),  gridsize*3*sizeof(int)));
  (*HostMem).nInfo.live  = (int*)malloc(sizeof(int)*gridsize);
  printf("live allocated:%d\n",gridsize);
#endif
  (*HostMem).batcnt     = (CMPTYPE*)malloc(sizeof(CMPTYPE)*numbins);
  memset((*HostMem).batcnt, 0, sizeof(CMPTYPE)*numbins);
#endif
#if defined(__CTALLY) //Collision density (flux) tally
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).spectrum), numbins*sizeof(CMPTYPE)));
  (*HostMem).spectrum = (CMPTYPE*)malloc(sizeof(CMPTYPE)*numbins);  

  (*HostMem).batcnt     = (CMPTYPE*)malloc(sizeof(CMPTYPE)*numbins);
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).batcnt), numbins*sizeof(CMPTYPE)));
#if defined(__CTALLY2)
  (*HostMem).batcnt2    = (CMPTYPE*)malloc(sizeof(CMPTYPE)*numbins);
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).batcnt2),numbins*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).cnt2_t),numbins*gridsize*sizeof(int)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).tally.cnt2), gridsize*numbins*sizeof(CMPTYPE)));
  gpuErrchk(hipMemset((*DeviceMem).tally.cnt2,0, numbins*gridsize*sizeof(CMPTYPE)));  
#endif
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).block_spectrum), numbins*gridx*sizeof(CMPTYPE)));
  gpuErrchk(hipMemset((*DeviceMem).block_spectrum, 0, numbins*gridx*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).tally.cnt), gridsize*numbins*sizeof(CMPTYPE)));
  gpuErrchk(hipMemset((*DeviceMem).tally.cnt, 0, numbins*gridsize*sizeof(CMPTYPE)));  
#endif
#if defined(__PROCESS)
  (*HostMem).batchmeans = (double*)malloc(sizeof(double)*nbat*numbins);
  (*HostMem).accmeans   = (double*)malloc(sizeof(double)*(nbat-ubat)*numbins);
#endif
#endif
#if defined(__TRAN)&&defined(__TALLY)
  memset((*HostMem).batcnt, 0, sizeof(CMPTYPE)*numbins);
#endif
  (*HostMem).wdspp = (float*)malloc(sizeof(float)*9);



  gpuErrchk(hipMalloc((void**)&((*DeviceMem).wdspp), 9*sizeof(float)));


  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.live),       banksize*sizeof(int)));
  gpuErrchk(hipMemset((*DeviceMem).nInfo.live, 0, banksize*sizeof(int)));  

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.rndState), gridsize*sizeof(hiprandState)));

#if defined(__WASTE)
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.id),       banksize*sizeof(unsigned)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.energy),   banksize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.sigT),   banksize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.sigA),   banksize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.sigF),   banksize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.isoenergy),banksize*sizeof(CMPTYPE)));
#endif

#if defined(__1D)
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_x),3*banksize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_y),banksize*sizeof(float)));
#endif 

#if defined(__3D)
#if defined(__FTALLY2)
  (*HostMem).nInfo.pos_x = (float*)malloc(sizeof(float)*gridsize);
  (*HostMem).nInfo.pos_y = (float*)malloc(sizeof(float)*gridsize);
  (*HostMem).nInfo.pos_z = (float*)malloc(sizeof(float)*gridsize);
#else
  (*HostMem).nInfo.pos_x = (float*)malloc(sizeof(float)*banksize);
  (*HostMem).nInfo.pos_y = (float*)malloc(sizeof(float)*banksize);
  (*HostMem).nInfo.pos_z = (float*)malloc(sizeof(float)*banksize);
#endif
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_x),3*banksize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_y),3*banksize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_z),3*banksize*sizeof(float)));
#endif

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).num_terminated_neutrons), sizeof(unsigned int)));
  gpuErrchk(hipMemset((*DeviceMem).num_terminated_neutrons, 0, sizeof(unsigned)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).num_live_neutrons), sizeof(unsigned int)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).block_terminated_neutrons), sizeof(unsigned int)*gridx));
  gpuErrchk(hipHostMalloc((void**)&((*HostMem).num_terminated_neutrons), sizeof(unsigned int)));
  (*HostMem).num_terminated_neutrons[0] = 0u;


  return;
}

void resettally(CMPTYPE *cnt, unsigned totbins){
  gpuErrchk(hipMemset(cnt, 0, totbins*sizeof(CMPTYPE)));
}

void resettally(int *cnt, unsigned totbins){
  gpuErrchk(hipMemset(cnt, 0, totbins*sizeof(int)));
}

void release_memory(MemStruct DeviceMem, MemStruct HostMem){
#if defined(__TALLY)
#if defined(__MTALLY)||(__FTALLY)||(__FTALLY2)
  gpuErrchk(hipFree(DeviceMem.nInfo.imat));
#else
  free(HostMem.spectrum);
  gpuErrchk(hipFree(DeviceMem.batcnt));
#if defined(__CTALLY2)
  gpuErrchk(hipFree(DeviceMem.batcnt2));
  free(HostMem.batcnt2);
  gpuErrchk(hipFree(DeviceMem.cnt2_t));
  gpuErrchk(hipFree(DeviceMem.tally.cnt2));
#endif
  gpuErrchk(hipFree(DeviceMem.spectrum));
  gpuErrchk(hipFree(DeviceMem.block_spectrum));
  gpuErrchk(hipFree(DeviceMem.tally.cnt));
#endif
  free(HostMem.batcnt);
#if defined(__PROCESS)
  free(HostMem.batchmeans);
  free(HostMem.accmeans);
#endif
#endif
  free(HostMem.nInfo.live);

  gpuErrchk(hipFree(DeviceMem.nInfo.live));
  gpuErrchk(hipFree(DeviceMem.nInfo.rndState));

#if defined(__WASTE)
  gpuErrchk(hipFree(DeviceMem.nInfo.id));
  gpuErrchk(hipFree(DeviceMem.nInfo.energy));
  gpuErrchk(hipFree(DeviceMem.nInfo.sigT));
  gpuErrchk(hipFree(DeviceMem.nInfo.sigA));
  gpuErrchk(hipFree(DeviceMem.nInfo.sigF));
  gpuErrchk(hipFree(DeviceMem.nInfo.isoenergy));
#endif

#if defined(__1D)
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_x));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_y));
#endif

#if defined(__3D)
  free(HostMem.nInfo.pos_x);
  free(HostMem.nInfo.pos_y);
  free(HostMem.nInfo.pos_z);
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_x));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_y));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_z));
#endif

  gpuErrchk(hipFree(DeviceMem.num_terminated_neutrons));
  gpuErrchk(hipFree(DeviceMem.num_live_neutrons));
  gpuErrchk(hipFree(DeviceMem.block_terminated_neutrons));
  gpuErrchk(hipHostFree(HostMem.num_terminated_neutrons));


  return;
}

