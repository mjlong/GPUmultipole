#include "manmemory.h"
__constant__ float wdspp[9];
//Simulation memory allocate and deallocate
void initialize_device(){
  hipSetDevice(0);
  gpuErrchk(hipSetDeviceFlags(hipDeviceMapHost | hipDeviceLmemResizeToMax));
}

void copymeans(int *h_cnt, int *batcnt, unsigned meshes, unsigned offset){
  for(int im=0;im<meshes;im++)
#if defined(__TRAN)
    batcnt[offset+im] += h_cnt[im];
#else
    batcnt[offset+im] = h_cnt[im];
#endif

}

void copydata(MemStruct DeviceMem, MemStruct HostMem){
  gpuErrchk(hipMemcpy(DeviceMem.wdspp,  HostMem.wdspp,   sizeof(float)*9, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(wdspp), DeviceMem.wdspp, 9*sizeof(float), 0, hipMemcpyDeviceToDevice));
}
void initialize_memory(MemStruct *DeviceMem, MemStruct *HostMem, unsigned numbins, unsigned gridx, unsigned blockx,unsigned nbat,unsigned ubat){
  unsigned gridsize;
#if defined(__TRAN)
  (*HostMem).initial_delayed = (int*)malloc(sizeof(int)*nbat);  
  gridsize = gridx*blockx*ubat;
  //for __TALLY, ubat is used as tranfac

  (*HostMem).nInfo.d_pos_x = (float*)malloc(sizeof(float)*gridsize);
  (*HostMem).nInfo.d_pos_y = (float*)malloc(sizeof(float)*gridsize);
  (*HostMem).nInfo.d_pos_z = (float*)malloc(sizeof(float)*gridsize);
  (*HostMem).nInfo.d_time  = (float*)malloc(sizeof(float)*gridsize);
  (*HostMem).nInfo.d_igen  = (int*)malloc(sizeof(int)*gridsize);
  (*HostMem).nInfo.d_nu    = (int*)malloc(sizeof(int)*gridsize);
#else
  gridsize = gridx*blockx;
#endif

#if defined(__TALLY)
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).spectrum), numbins*sizeof(CMPTYPE)));
  (*HostMem).spectrum = (CMPTYPE*)malloc(sizeof(CMPTYPE)*numbins);  
  (*HostMem).batcnt     = (CMPTYPE*)malloc(sizeof(CMPTYPE)*numbins);
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).batcnt), numbins*sizeof(CMPTYPE)));
#if defined(__PROCESS)
  (*HostMem).batchmeans = (double*)malloc(sizeof(double)*nbat*numbins);
  (*HostMem).accmeans   = (double*)malloc(sizeof(double)*(nbat-ubat)*numbins);
#endif
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).block_spectrum), numbins*gridx*sizeof(CMPTYPE)));
  gpuErrchk(hipMemset((*DeviceMem).block_spectrum, 0, numbins*gridx*sizeof(CMPTYPE)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).tally.cnt), gridsize*numbins*sizeof(CMPTYPE)));
  gpuErrchk(hipMemset((*DeviceMem).tally.cnt, 0, numbins*gridsize*sizeof(CMPTYPE)));  
#endif
#if defined(__TRAN)&&defined(__TALLY)
  memset((*HostMem).batcnt, 0, sizeof(CMPTYPE)*numbins);
#endif
  (*HostMem).wdspp = (float*)malloc(sizeof(float)*9);


  (*HostMem).nInfo.live  = (int*)malloc(sizeof(int)*gridsize);

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).wdspp), 9*sizeof(float)));


  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.id),       gridsize*sizeof(unsigned)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.live),       gridsize*sizeof(unsigned)));
  gpuErrchk(hipMemset((*DeviceMem).nInfo.live, 0, gridsize*sizeof(int)));  

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.rndState), gridsize*sizeof(hiprandState)));

#if defined(__WASTE)
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.energy),   gridsize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.sigT),   gridsize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.sigA),   gridsize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.sigF),   gridsize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.isoenergy),gridsize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.imat),  gridsize*sizeof(int)));
#endif

#if defined(__SCATTERPLOT)
  (*HostMem).nInfo.energy  = (CMPTYPE*)malloc(sizeof(CMPTYPE)*gridsize);
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.energy),   gridsize*sizeof(CMPTYPE))); //use as initial z position for plot
#endif

#if defined(__1D)
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_x),3*gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_y),gridsize*sizeof(float)));
#endif 

#if defined(__3D)
  (*HostMem).nInfo.pos_x = (float*)malloc(sizeof(float)*gridsize);
  (*HostMem).nInfo.pos_y = (float*)malloc(sizeof(float)*gridsize);
  (*HostMem).nInfo.pos_z = (float*)malloc(sizeof(float)*gridsize);
#if defined(__TRAN)
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_x),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_y),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_z),gridsize*sizeof(float)));

  (*HostMem).nInfo.dir_polar = (float*)malloc(sizeof(float)*gridsize);
  (*HostMem).nInfo.dir_azimu = (float*)malloc(sizeof(float)*gridsize);
  (*HostMem).nInfo.d_closest = (float*)malloc(sizeof(float)*gridsize);
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.dir_polar),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.dir_azimu),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.d_closest ),gridsize*sizeof(float)));
  gpuErrchk(hipMemset((*DeviceMem).nInfo.d_closest, 0, gridsize*sizeof(float)));  //use as time
#else
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_x),3*gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_y),3*gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_z),3*gridsize*sizeof(float)));
#endif
#endif

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).num_terminated_neutrons), sizeof(unsigned int)));
  gpuErrchk(hipMemset((*DeviceMem).num_terminated_neutrons, 0, sizeof(unsigned)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).num_live_neutrons), sizeof(unsigned int)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).block_terminated_neutrons), sizeof(unsigned int)*gridx));
  gpuErrchk(hipHostMalloc((void**)&((*HostMem).num_terminated_neutrons), sizeof(unsigned int)));
  (*HostMem).num_terminated_neutrons[0] = 0u;


  return;
}

void resettally(int *cnt, unsigned totbins){
  gpuErrchk(hipMemset(cnt, 0, totbins*sizeof(int)));}

void release_memory(MemStruct DeviceMem, MemStruct HostMem){
#if defined(__TALLY)
  free(HostMem.spectrum);
#if defined(__PROCESS)
  free(HostMem.batchmeans);
  free(HostMem.accmeans);
#endif
  free(HostMem.batcnt);
  gpuErrchk(hipFree(DeviceMem.batcnt));
  gpuErrchk(hipFree(DeviceMem.spectrum));
  gpuErrchk(hipFree(DeviceMem.block_spectrum));
  gpuErrchk(hipFree(DeviceMem.tally.cnt));
#endif
  free(HostMem.nInfo.live);
  free(HostMem.wdspp);

  gpuErrchk(hipFree(DeviceMem.wdspp));

  gpuErrchk(hipFree(DeviceMem.nInfo.id));
  gpuErrchk(hipFree(DeviceMem.nInfo.live));
  gpuErrchk(hipFree(DeviceMem.nInfo.rndState));

#if defined(__WASTE)
  gpuErrchk(hipFree(DeviceMem.nInfo.energy));
  gpuErrchk(hipFree(DeviceMem.nInfo.sigT));
  gpuErrchk(hipFree(DeviceMem.nInfo.sigA));
  gpuErrchk(hipFree(DeviceMem.nInfo.sigF));
  gpuErrchk(hipFree(DeviceMem.nInfo.isoenergy));
  gpuErrchk(hipFree(DeviceMem.nInfo.imat));
#endif

#if defined(__1D)
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_x));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_y));
#endif

#if defined(__SCATTERPLOT)
  gpuErrchk(hipFree(DeviceMem.nInfo.energy));
  free(HostMem.nInfo.energy);
#endif
#if defined(__3D)
  free(HostMem.nInfo.pos_x);
  free(HostMem.nInfo.pos_y);
  free(HostMem.nInfo.pos_z);
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_x));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_y));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_z));

#if defined(__TRAN)
  free(HostMem.nInfo.d_pos_x);
  free(HostMem.nInfo.d_pos_y);
  free(HostMem.nInfo.d_pos_z);
  free(HostMem.nInfo.d_time );
  free(HostMem.nInfo.d_nu   );
  free(HostMem.nInfo.d_igen );

  free(HostMem.initial_delayed);
  free(HostMem.nInfo.dir_polar);
  free(HostMem.nInfo.dir_azimu);
  free(HostMem.nInfo.d_closest);
  gpuErrchk(hipFree(DeviceMem.nInfo.dir_polar));
  gpuErrchk(hipFree(DeviceMem.nInfo.dir_azimu));
  gpuErrchk(hipFree(DeviceMem.nInfo.d_closest));
#endif
#endif

  gpuErrchk(hipFree(DeviceMem.num_terminated_neutrons));
  gpuErrchk(hipFree(DeviceMem.num_live_neutrons));
  gpuErrchk(hipFree(DeviceMem.block_terminated_neutrons));
  gpuErrchk(hipHostFree(HostMem.num_terminated_neutrons));


  return;
}

