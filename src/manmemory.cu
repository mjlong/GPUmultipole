#include "manmemory.h"
__constant__ float spectrumbins[NUM_BINS+1];

#if defined (__FOURIERW)
#include "fourierw.h"
__constant__ CMPTYPE a[M+1];
__constant__ CMPTYPE b[M+1];
#endif

#if defined (__QUICKW)
#include "QuickW.h"
#endif

#if defined (__QUICKWC) || defined(__INTERPEXP)
__constant__ CMPTYPE2 constwtable[LENGTH*LENGTH];
#endif


//Simulation memory allocate and deallocate
void initialize_device(){
  hipSetDevice(0);
  gpuErrchk(hipSetDeviceFlags(hipDeviceMapHost | hipDeviceLmemResizeToMax));
}

#if defined(__FOURIERW)
void fill_wtables(CMPTYPE** da, CMPTYPE** db){
  gpuErrchk(hipMalloc((void**)da, (M+1)*sizeof(CMPTYPE))); 
  gpuErrchk(hipMalloc((void**)db, (M+1)*sizeof(CMPTYPE))); 
  fill_a<<<1,M+1>>>(*da,*db); 
  hipMemcpyToSymbol(HIP_SYMBOL(a), *da, M*sizeof(CMPTYPE), 0, hipMemcpyDeviceToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(b), *db, M*sizeof(CMPTYPE), 0, hipMemcpyDeviceToDevice);

}
void release_wtables(CMPTYPE* da, CMPTYPE* db){
  gpuErrchk(hipFree(da));
  gpuErrchk(hipFree(db));
}
#endif

#if defined(__INTERPEXP)
void fill_wtables(CComplex<CMPTYPE>** exptable){
  gpuErrchk(hipMalloc((void**)exptable, LENGTH*LENGTH * 2 * sizeof(CMPTYPE)));
  fill_exp_table<<<LENGTH,LENGTH>>>(*exptable);
  hipMemcpyToSymbol(HIP_SYMBOL(constwtable), *exptable, LENGTH*LENGTH*2*sizeof(CMPTYPE), 0, hipMemcpyDeviceToDevice);
}
void release_wtables(CComplex<CMPTYPE>* exptable){
  gpuErrchk(hipFree(exptable));
}
#endif

#if defined(__QUICKW)
void fill_wtables(CComplex<CMPTYPE>** wtable){
  gpuErrchk(hipMalloc((void**)wtable, LENGTH*LENGTH * 2 * sizeof(CMPTYPE)));
  fill_w_tabulated<<<LENGTH,LENGTH>>>(*wtable);
#if defined(__QUICKWC)
  hipMemcpyToSymbol(HIP_SYMBOL(constwtable), *wtable, LENGTH*LENGTH*2*sizeof(CMPTYPE), 0, hipMemcpyDeviceToDevice);
#endif
#if defined(__QUICKWT)
  bindwtable(*wtable);
#endif
}
void release_wtables(CComplex<CMPTYPE>* wtable){
  gpuErrchk(hipFree(wtable));
}
#endif

void initialize_memory(MemStruct *HostMem, unsigned numbins){
  (*HostMem).spectrum = (unsigned*)malloc(sizeof(unsigned)*numbins);  
  memset((*HostMem).spectrum, 0, sizeof(unsigned)*numbins);
  return;
}

void release_memory(MemStruct HostMem){
  free(HostMem.spectrum);
  return;
}

#if defined(__XS_GPU)
void allocate_buffer(unsigned maxiso, unsigned** iS_d, 
                     CMPTYPE** sigTs_h, CMPTYPE** sigAs_h, CMPTYPE** sigFs_h,
                     CMPTYPE** sigTs_d, CMPTYPE** sigAs_d, CMPTYPE** sigFs_d){
  *sigTs_h = (CMPTYPE*)malloc(sizeof(CMPTYPE)*maxiso);
  *sigAs_h = (CMPTYPE*)malloc(sizeof(CMPTYPE)*maxiso);
  *sigFs_h = (CMPTYPE*)malloc(sizeof(CMPTYPE)*maxiso);
  gpuErrchk(hipMalloc((void**)iS_d, maxiso*sizeof(unsigned)));
  gpuErrchk(hipMalloc((void**)sigTs_d, maxiso*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)sigAs_d, maxiso*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)sigFs_d, maxiso*sizeof(CMPTYPE)));
}

void release_buffer(unsigned* iS_d, 
                    CMPTYPE* sigTs_h, CMPTYPE* sigAs_h, CMPTYPE* sigFs_h, 
                    CMPTYPE* sigTs_d, CMPTYPE* sigAs_d, CMPTYPE* sigFs_d){
  free(sigTs_d);
  free(sigAs_d);
  free(sigFs_d);

  gpuErrchk(hipFree(iS_d));
  gpuErrchk(hipFree(sigTs_d));
  gpuErrchk(hipFree(sigAs_d));
  gpuErrchk(hipFree(sigFs_d));
}
#endif
