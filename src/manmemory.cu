#include "manmemory.h"
__constant__ float wdspp[9];
//Simulation memory allocate and deallocate
void initialize_device(){
  hipSetDevice(0);
  gpuErrchk(hipSetDeviceFlags(hipDeviceMapHost | hipDeviceLmemResizeToMax));
}

void copymeans(int *h_cnt, int *batcnt, unsigned meshes, unsigned offset){
  for(int im=0;im<meshes;im++)
    batcnt[offset+im] = h_cnt[im];

}

void copydata(MemStruct DeviceMem, MemStruct HostMem){
  gpuErrchk(hipMemcpy(DeviceMem.wdspp,  HostMem.wdspp,   sizeof(float)*9, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(wdspp), DeviceMem.wdspp, 9*sizeof(float), 0, hipMemcpyDeviceToDevice));
}

void delayed_memory(int nbat, int num_srcp, int csize,MemStruct* HostMem){
  memset((*HostMem).newly_delayed,   0, sizeof(int)*nbat);
  (*HostMem).nInfo.d_pos_x = (float*)malloc(sizeof(float)*csize);
  (*HostMem).nInfo.d_pos_y = (float*)malloc(sizeof(float)*csize);
  (*HostMem).nInfo.d_pos_z = (float*)malloc(sizeof(float)*csize);
  (*HostMem).nInfo.d_igen  = (int*)malloc(sizeof(int)*csize);
  (*HostMem).nInfo.d_nu    = (int*)malloc(sizeof(int)*csize);
  for(int i=0;i<csize;i++)
    ((*HostMem).nInfo.d_igen)[i] = -1;
  (*HostMem).nInfo.dbank_x = (float*)malloc(sizeof(float)*num_srcp);
  (*HostMem).nInfo.dbank_y = (float*)malloc(sizeof(float)*num_srcp);
  (*HostMem).nInfo.dbank_z = (float*)malloc(sizeof(float)*num_srcp);
}

void initialize_memory(MemStruct *DeviceMem, MemStruct *HostMem, unsigned numbins, unsigned gridx, unsigned blockx,unsigned nbat,unsigned ubat){
  unsigned gridsize,banksize;
  gridsize = gridx*blockx;
  banksize = gridx*blockx*ubat;
  //for __TALLY, ubat is used as tranfac

  (*HostMem).newly_delayed = (int*)malloc(sizeof(int)*nbat);
#if defined(__TALLY)
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).spectrum), numbins*sizeof(CMPTYPE)));
  (*HostMem).spectrum = (CMPTYPE*)malloc(sizeof(CMPTYPE)*numbins);  
  (*HostMem).batcnt     = (CMPTYPE*)malloc(sizeof(CMPTYPE)*numbins);
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).batcnt), numbins*sizeof(CMPTYPE)));
#if defined(__PROCESS)
  (*HostMem).batchmeans = (double*)malloc(sizeof(double)*nbat*numbins);
  (*HostMem).accmeans   = (double*)malloc(sizeof(double)*(nbat-ubat)*numbins);
#endif
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).block_spectrum), numbins*gridx*sizeof(CMPTYPE)));
  gpuErrchk(hipMemset((*DeviceMem).block_spectrum, 0, numbins*gridx*sizeof(CMPTYPE)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).tally.cnt), gridsize*numbins*sizeof(CMPTYPE)));
  gpuErrchk(hipMemset((*DeviceMem).tally.cnt, 0, numbins*gridsize*sizeof(CMPTYPE)));  
#endif
#if defined(__TRAN)&&defined(__TALLY)
  memset((*HostMem).batcnt, 0, sizeof(CMPTYPE)*numbins);
#endif
  (*HostMem).wdspp = (float*)malloc(sizeof(float)*9);


  (*HostMem).nInfo.live  = (int*)malloc(sizeof(int)*banksize);

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).wdspp), 9*sizeof(float)));


  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.id),       banksize*sizeof(unsigned)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.live),       banksize*sizeof(int)));
  gpuErrchk(hipMemset((*DeviceMem).nInfo.live, 0, banksize*sizeof(int)));  

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.rndState), banksize*sizeof(hiprandState)));

#if defined(__WASTE)
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.energy),   banksize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.sigT),   banksize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.sigA),   banksize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.sigF),   banksize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.isoenergy),banksize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.imat),  banksize*sizeof(int)));
#endif

#if defined(__SCATTERPLOT)
  (*HostMem).nInfo.energy  = (CMPTYPE*)malloc(sizeof(CMPTYPE)*banksize);
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.energy),   banksize*sizeof(CMPTYPE))); //use as initial z position for plot
#endif

#if defined(__1D)
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_x),3*banksize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_y),banksize*sizeof(float)));
#endif 

#if defined(__3D)
  (*HostMem).nInfo.pos_x = (float*)malloc(sizeof(float)*banksize);
  (*HostMem).nInfo.pos_y = (float*)malloc(sizeof(float)*banksize);
  (*HostMem).nInfo.pos_z = (float*)malloc(sizeof(float)*banksize);
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_x),3*banksize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_y),3*banksize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_z),3*banksize*sizeof(float)));
#endif

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).num_terminated_neutrons), sizeof(unsigned int)));
  gpuErrchk(hipMemset((*DeviceMem).num_terminated_neutrons, 0, sizeof(unsigned)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).num_live_neutrons), sizeof(unsigned int)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).block_terminated_neutrons), sizeof(unsigned int)*gridx));
  gpuErrchk(hipHostMalloc((void**)&((*HostMem).num_terminated_neutrons), sizeof(unsigned int)));
  (*HostMem).num_terminated_neutrons[0] = 0u;


  return;
}

void resettally(CMPTYPE *cnt, unsigned totbins){
  gpuErrchk(hipMemset(cnt, 0, totbins*sizeof(CMPTYPE)));}

void release_memory(MemStruct DeviceMem, MemStruct HostMem){
  free(HostMem.nInfo.dbank_x);
  free(HostMem.nInfo.dbank_y);
  free(HostMem.nInfo.dbank_z);

  free(HostMem.nInfo.d_pos_x);
  free(HostMem.nInfo.d_pos_y);
  free(HostMem.nInfo.d_pos_z);
  free(HostMem.nInfo.d_nu   );
  free(HostMem.nInfo.d_igen );
  free(HostMem.newly_delayed);
#if defined(__TALLY)
  free(HostMem.spectrum);
#if defined(__PROCESS)
  free(HostMem.batchmeans);
  free(HostMem.accmeans);
#endif
  free(HostMem.batcnt);
  gpuErrchk(hipFree(DeviceMem.batcnt));
  gpuErrchk(hipFree(DeviceMem.spectrum));
  gpuErrchk(hipFree(DeviceMem.block_spectrum));
  gpuErrchk(hipFree(DeviceMem.tally.cnt));
#endif
  free(HostMem.nInfo.live);
  free(HostMem.wdspp);

  gpuErrchk(hipFree(DeviceMem.wdspp));

  gpuErrchk(hipFree(DeviceMem.nInfo.id));
  gpuErrchk(hipFree(DeviceMem.nInfo.live));
  gpuErrchk(hipFree(DeviceMem.nInfo.rndState));

#if defined(__WASTE)
  gpuErrchk(hipFree(DeviceMem.nInfo.energy));
  gpuErrchk(hipFree(DeviceMem.nInfo.sigT));
  gpuErrchk(hipFree(DeviceMem.nInfo.sigA));
  gpuErrchk(hipFree(DeviceMem.nInfo.sigF));
  gpuErrchk(hipFree(DeviceMem.nInfo.isoenergy));
  gpuErrchk(hipFree(DeviceMem.nInfo.imat));
#endif

#if defined(__1D)
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_x));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_y));
#endif

#if defined(__SCATTERPLOT)
  gpuErrchk(hipFree(DeviceMem.nInfo.energy));
  free(HostMem.nInfo.energy);
#endif
#if defined(__3D)
  free(HostMem.nInfo.pos_x);
  free(HostMem.nInfo.pos_y);
  free(HostMem.nInfo.pos_z);
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_x));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_y));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_z));
#endif

  gpuErrchk(hipFree(DeviceMem.num_terminated_neutrons));
  gpuErrchk(hipFree(DeviceMem.num_live_neutrons));
  gpuErrchk(hipFree(DeviceMem.block_terminated_neutrons));
  gpuErrchk(hipHostFree(HostMem.num_terminated_neutrons));


  return;
}

