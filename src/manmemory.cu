#include "manmemory.h"
__constant__ float spectrumbins[NUM_BINS+1];

#if defined (__FOURIERW)
#include "fourierw.h"
__constant__ CMPTYPE a[M+1];
__constant__ CMPTYPE b[M+1];
#endif

#if defined (__QUICKW)
#include "QuickW.h"
#endif

#if defined (__QUICKWC) || defined(__INTERPEXP)
__constant__ CMPTYPE2 constwtable[LENGTH*LENGTH];
#endif


//Simulation memory allocate and deallocate
void initialize_device(){
  hipSetDevice(0);
  gpuErrchk(hipSetDeviceFlags(hipDeviceMapHost | hipDeviceLmemResizeToMax));
}

#if defined(__FOURIERW)
void fill_wtables(CMPTYPE** da, CMPTYPE** db){
  gpuErrchk(hipMalloc((void**)da, (M+1)*sizeof(CMPTYPE))); 
  gpuErrchk(hipMalloc((void**)db, (M+1)*sizeof(CMPTYPE))); 
  fill_a<<<1,M+1>>>(*da,*db); 
  hipMemcpyToSymbol(HIP_SYMBOL(a), *da, M*sizeof(CMPTYPE), 0, hipMemcpyDeviceToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(b), *db, M*sizeof(CMPTYPE), 0, hipMemcpyDeviceToDevice);

}
void release_wtables(CMPTYPE* da, CMPTYPE* db){
  gpuErrchk(hipFree(da));
  gpuErrchk(hipFree(db));
}
#endif

#if defined(__INTERPEXP)
void fill_wtables(CComplex<CMPTYPE>** exptable){
  gpuErrchk(hipMalloc((void**)exptable, LENGTH*LENGTH * 2 * sizeof(CMPTYPE)));
  fill_exp_table<<<LENGTH,LENGTH>>>(*exptable);
  hipMemcpyToSymbol(HIP_SYMBOL(constwtable), *exptable, LENGTH*LENGTH*2*sizeof(CMPTYPE), 0, hipMemcpyDeviceToDevice);
}
void release_wtables(CComplex<CMPTYPE>* exptable){
  gpuErrchk(hipFree(exptable));
}
#endif

#if defined(__QUICKW)
void fill_wtables(CComplex<CMPTYPE>** wtable){
  gpuErrchk(hipMalloc((void**)wtable, LENGTH*LENGTH * 2 * sizeof(CMPTYPE)));
  fill_w_tabulated<<<LENGTH,LENGTH>>>(*wtable);
#if defined(__QUICKWC)
  hipMemcpyToSymbol(HIP_SYMBOL(constwtable), *wtable, LENGTH*LENGTH*2*sizeof(CMPTYPE), 0, hipMemcpyDeviceToDevice);
#endif
#if defined(__QUICKWT)
  bindwtable(*wtable);
#endif
}
void release_wtables(CComplex<CMPTYPE>* wtable){
  gpuErrchk(hipFree(wtable));
}
#endif

void assign_tallybins(float *h_tallybins, float **d_tallybins,unsigned nbinedge){
  gpuErrchk(hipMalloc((void**)(d_tallybins), nbinedge*sizeof(float)));
  gpuErrchk(hipMemcpy(*d_tallybins,h_tallybins,nbinedge*sizeof(float),hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(spectrumbins), *d_tallybins, nbinedge*sizeof(float), 0, hipMemcpyDeviceToDevice));
}

void initialize_memory(MemStruct *DeviceMem, MemStruct *HostMem, unsigned numbins, unsigned gridx, unsigned blockx ){
  unsigned gridsize;
  gridsize = gridx*blockx;

  assign_tallybins((*HostMem).tallybins, &((*DeviceMem).tallybins), numbins+1);

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).spectrum), numbins*sizeof(unsigned int)));
  (*HostMem).spectrum = (unsigned*)malloc(sizeof(unsigned)*numbins);  

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).block_spectrum), numbins*gridx*sizeof(unsigned int)));
  gpuErrchk(hipMemset((*DeviceMem).block_spectrum, 0, numbins*gridx*sizeof(unsigned int)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.id),       gridsize*sizeof(unsigned)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.live),       gridsize*sizeof(unsigned)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.rndState), gridsize*sizeof(hiprandState)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.energy),   gridsize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.sigT),   gridsize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.sigA),   gridsize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.sigF),   gridsize*sizeof(CMPTYPE)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.isoenergy),gridsize*sizeof(CMPTYPE)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.imat),  gridsize*sizeof(unsigned)));
  gpuErrchk(hipMemset((*DeviceMem).nInfo.imat, 0, sizeof(unsigned)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_x),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_y),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_z),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.dir_polar),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.dir_azimu),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.d_closest ),gridsize*sizeof(float)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).num_terminated_neutrons), sizeof(unsigned int)));
  gpuErrchk(hipMemset((*DeviceMem).num_terminated_neutrons, 0, sizeof(unsigned)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).num_live_neutrons), sizeof(unsigned int)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).block_terminated_neutrons), sizeof(unsigned int)*gridx));
  gpuErrchk(hipHostMalloc((void**)&((*HostMem).num_terminated_neutrons), sizeof(unsigned int)));
  (*HostMem).num_terminated_neutrons[0] = 0u;

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).tally.cnt), gridsize*numbins*sizeof(unsigned)));
  gpuErrchk(hipMemset((*DeviceMem).tally.cnt, 0, numbins*gridsize*sizeof(unsigned)));  

  return;
}

void release_memory(MemStruct DeviceMem, MemStruct HostMem){
  free(HostMem.spectrum);

  gpuErrchk(hipFree(DeviceMem.tallybins));
  gpuErrchk(hipFree(DeviceMem.spectrum));
  gpuErrchk(hipFree(DeviceMem.block_spectrum));

  gpuErrchk(hipFree(DeviceMem.nInfo.id));
  gpuErrchk(hipFree(DeviceMem.nInfo.live));
  gpuErrchk(hipFree(DeviceMem.nInfo.rndState));
  gpuErrchk(hipFree(DeviceMem.nInfo.energy));
  gpuErrchk(hipFree(DeviceMem.nInfo.sigT));
  gpuErrchk(hipFree(DeviceMem.nInfo.sigA));
  gpuErrchk(hipFree(DeviceMem.nInfo.sigF));

  gpuErrchk(hipFree(DeviceMem.nInfo.isoenergy));

  gpuErrchk(hipFree(DeviceMem.nInfo.imat));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_x));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_y));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_z));
  gpuErrchk(hipFree(DeviceMem.nInfo.dir_polar));
  gpuErrchk(hipFree(DeviceMem.nInfo.dir_azimu));
  gpuErrchk(hipFree(DeviceMem.nInfo.d_closest));

  gpuErrchk(hipFree(DeviceMem.num_terminated_neutrons));
  gpuErrchk(hipFree(DeviceMem.num_live_neutrons));
  gpuErrchk(hipFree(DeviceMem.block_terminated_neutrons));
  gpuErrchk(hipHostFree(HostMem.num_terminated_neutrons));

  gpuErrchk(hipFree(DeviceMem.tally.cnt));

  return;
}

