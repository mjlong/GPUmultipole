#include "manmemory.h"
__constant__ unsigned spectrumbins[NUM_BINS+1];

#if defined (__FOURIERW)
#include "fourierw.h"
__constant__ CMPTYPE a[M+1];
__constant__ CMPTYPE b[M+1];
#endif

#if defined (__QUICKW)
#include "QuickW.h"
#endif

#if defined (__QUICKWC) || defined(__INTERPEXP)
__constant__ CMPTYPE2 constwtable[LENGTH*LENGTH];
#endif


//Simulation memory allocate and deallocate
void initialize_device(){
  hipSetDevice(0);
  gpuErrchk(hipSetDeviceFlags(hipDeviceMapHost | hipDeviceLmemResizeToMax));
}

#if defined(__FOURIERW)
void fill_wtables(CMPTYPE** da, CMPTYPE** db){
  gpuErrchk(hipMalloc((void**)da, (M+1)*sizeof(CMPTYPE))); 
  gpuErrchk(hipMalloc((void**)db, (M+1)*sizeof(CMPTYPE))); 
  fill_a<<<1,M+1>>>(*da,*db); 
  hipMemcpyToSymbol(HIP_SYMBOL(a), *da, M*sizeof(CMPTYPE), 0, hipMemcpyDeviceToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(b), *db, M*sizeof(CMPTYPE), 0, hipMemcpyDeviceToDevice);

}
void release_wtables(CMPTYPE* da, CMPTYPE* db){
  gpuErrchk(hipFree(da));
  gpuErrchk(hipFree(db));
}
#endif

#if defined(__INTERPEXP)
void fill_wtables(CComplex<CMPTYPE>** exptable){
  gpuErrchk(hipMalloc((void**)exptable, LENGTH*LENGTH * 2 * sizeof(CMPTYPE)));
  fill_exp_table<<<LENGTH,LENGTH>>>(*exptable);
  hipMemcpyToSymbol(HIP_SYMBOL(constwtable), *exptable, LENGTH*LENGTH*2*sizeof(CMPTYPE), 0, hipMemcpyDeviceToDevice);
}
void release_wtables(CComplex<CMPTYPE>* exptable){
  gpuErrchk(hipFree(exptable));
}
#endif

#if defined(__QUICKW)
void fill_wtables(CComplex<CMPTYPE>** wtable){
  gpuErrchk(hipMalloc((void**)wtable, LENGTH*LENGTH * 2 * sizeof(CMPTYPE)));
  fill_w_tabulated<<<LENGTH,LENGTH>>>(*wtable);
#if defined(__QUICKWC)
  hipMemcpyToSymbol(HIP_SYMBOL(constwtable), *wtable, LENGTH*LENGTH*2*sizeof(CMPTYPE), 0, hipMemcpyDeviceToDevice);
#endif
#if defined(__QUICKWT)
  bindwtable(*wtable);
#endif
}
void release_wtables(CComplex<CMPTYPE>* wtable){
  gpuErrchk(hipFree(wtable));
}
#endif

void assign_tallybins(double *h_tallybins, double **d_tallybins){
  gpuErrchk(hipMalloc((void**)(d_tallybins), (NUM_BINS+1)*sizeof(double)));
  gpuErrchk(hipMemcpy(*d_tallybins,h_tallybins,(NUM_BINS+1)*sizeof(double),hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(spectrumbins), *d_tallybins, (NUM_BINS+1)*sizeof(double), 0, hipMemcpyDeviceToDevice));
}

void initialize_memory(MemStruct *DeviceMem, MemStruct *HostMem, unsigned **h_blockcnt, unsigned** d_blockcnt, unsigned gridx, unsigned blockx ){
  unsigned gridsize;
  gridsize = gridx*blockx;

  *h_blockcnt      = (unsigned*)malloc(gridx*sizeof(unsigned));

  gpuErrchk(hipMalloc((void**)(d_blockcnt), gridx*sizeof(unsigned int)));
  gpuErrchk(hipMemset(*d_blockcnt, 0, gridx*sizeof(unsigned int)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.id),       gridsize*sizeof(unsigned)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.live),       gridsize*sizeof(unsigned)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.rndState), gridsize*sizeof(hiprandState)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.energy),   gridsize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.sigT),   gridsize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.sigA),   gridsize*sizeof(CMPTYPE)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.sigF),   gridsize*sizeof(CMPTYPE)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.isoenergy),gridsize*sizeof(CMPTYPE)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.imat),  gridsize*sizeof(unsigned)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_x),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_y),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.pos_z),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.dir_polar),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.dir_azimu),gridsize*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).nInfo.d_closest ),gridsize*sizeof(float)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).num_terminated_neutrons), sizeof(unsigned int)));
  gpuErrchk(hipMemset((*DeviceMem).num_terminated_neutrons, 0, sizeof(unsigned)));
  gpuErrchk(hipMalloc((void**)&((*DeviceMem).num_live_neutrons), sizeof(unsigned int)));

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).block_terminated_neutrons), sizeof(unsigned int)*gridx));
  gpuErrchk(hipHostMalloc((void**)&((*HostMem).num_terminated_neutrons), sizeof(unsigned int)));
  (*HostMem).num_terminated_neutrons[0] = 0u;

  gpuErrchk(hipMalloc((void**)&((*DeviceMem).tally.cnt), gridsize*sizeof(unsigned)));
  gpuErrchk(hipMemset((*DeviceMem).tally.cnt, 0, gridsize*sizeof(unsigned)));  

  return;
}

void release_memory(MemStruct DeviceMem, MemStruct HostMem, unsigned *h_blockcnt, unsigned* d_blockcnt, double* d_tallybins ){
  free(h_blockcnt);
  gpuErrchk(hipFree(d_blockcnt));
  gpuErrchk(hipFree(d_tallybins));

  gpuErrchk(hipFree(DeviceMem.nInfo.id));
  gpuErrchk(hipFree(DeviceMem.nInfo.live));
  gpuErrchk(hipFree(DeviceMem.nInfo.rndState));
  gpuErrchk(hipFree(DeviceMem.nInfo.energy));
  gpuErrchk(hipFree(DeviceMem.nInfo.sigT));
  gpuErrchk(hipFree(DeviceMem.nInfo.sigA));
  gpuErrchk(hipFree(DeviceMem.nInfo.sigF));

  gpuErrchk(hipFree(DeviceMem.nInfo.isoenergy));

  gpuErrchk(hipFree(DeviceMem.nInfo.imat));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_x));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_y));
  gpuErrchk(hipFree(DeviceMem.nInfo.pos_z));
  gpuErrchk(hipFree(DeviceMem.nInfo.dir_polar));
  gpuErrchk(hipFree(DeviceMem.nInfo.dir_azimu));
  gpuErrchk(hipFree(DeviceMem.nInfo.d_closest));

  gpuErrchk(hipFree(DeviceMem.num_terminated_neutrons));
  gpuErrchk(hipFree(DeviceMem.num_live_neutrons));
  gpuErrchk(hipFree(DeviceMem.block_terminated_neutrons));
  gpuErrchk(hipHostFree(HostMem.num_terminated_neutrons));

  gpuErrchk(hipFree(DeviceMem.tally.cnt));
  return;
}

