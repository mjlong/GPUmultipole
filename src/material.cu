#include "material.h"

material::material(struct matdata *pmat, unsigned numIso){
  gpuErrchk(hipMalloc((void**)&offsets,sizeof(unsigned)*pmat->numMat)); 
  gpuErrchk(hipMalloc((void**)&N_tot, sizeof(float)*pmat->numMat));
  gpuErrchk(hipMalloc((void**)&densities, sizeof(float)*numIso));
  gpuErrchk(hipMalloc((void**)&isotopes, sizeof(unsigned)*numIso));

  gpuErrchk(hipMemcpy(offsets, pmat->offsets, sizeof(unsigned)*pmat->numMat, hipMemcpyHostToDevice)); 
  gpuErrchk(hipMemcpy(N_tot, pmat->N_tot, sizeof(float)*pmat->numMat, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(densities, pmat->densities, sizeof(float)*numIso, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(isotopes, pmat->isotopes, sizeof(unsigned)*numIso, hipMemcpyHostToDevice));
}

material::~material(){
}

void material::release_pointer(){
  gpuErrchk(hipFree(offsets));
  gpuErrchk(hipFree(N_tot));
  gpuErrchk(hipFree(densities));
  gpuErrchk(hipFree(isotopes));
}
