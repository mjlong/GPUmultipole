#include "hip/hip_runtime.h"
#include "material.h"

material::material(struct matdata *pmat, unsigned numIso){
  unsigned numMat = pmat->numMat;
  gpuErrchk(hipMalloc((void**)&offsets,sizeof(unsigned)*(numMat+1))); 
  gpuErrchk(hipMalloc((void**)&N_tot, sizeof(float)*numMat));
  gpuErrchk(hipMalloc((void**)&densities, sizeof(float)*numIso));
  gpuErrchk(hipMalloc((void**)&isotopes, sizeof(unsigned)*numIso));

  gpuErrchk(hipMemcpy(offsets, pmat->offsets, sizeof(unsigned)*numMat, hipMemcpyHostToDevice)); 
  shiftoffset<<<1,numMat>>>(offsets,numMat-1-(pmat->offsets[numMat-1]));
  gpuErrchk(hipMemcpy(N_tot, pmat->N_tot, sizeof(float)*numMat, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(densities, pmat->densities, sizeof(float)*numIso, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(isotopes, pmat->isotopes, sizeof(unsigned)*numIso, hipMemcpyHostToDevice));
}

material::~material(){
}

void material::release_pointer(){
  gpuErrchk(hipFree(offsets));
  gpuErrchk(hipFree(N_tot));
  gpuErrchk(hipFree(densities));
  gpuErrchk(hipFree(isotopes));
}

__global__ void shiftoffset(unsigned* offsets, unsigned shift)
{
  int id = threadIdx.x;
  offsets[id]+=shift;
}
