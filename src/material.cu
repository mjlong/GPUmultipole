#include "material.h"

material::material(struct matdata *pmat, unsigned numIso){
  unsigned numMat = pmat->numMat;
  gpuErrchk(hipMalloc((void**)&offsets,sizeof(unsigned)*(numMat+1)));
  gpuErrchk(hipMalloc((void**)&N_tot, sizeof(float)*numMat));
  gpuErrchk(hipMalloc((void**)&densities, sizeof(float)*numIso));
  gpuErrchk(hipMalloc((void**)&isotopes, sizeof(unsigned)*numIso));

  gpuErrchk(hipMemcpy(offsets, pmat->offsets, sizeof(unsigned)*numMat, hipMemcpyHostToDevice)); 
  gpuErrchk(hipMemcpy(N_tot, pmat->N_tot, sizeof(float)*numMat, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(densities, pmat->densities, sizeof(float)*numIso, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(isotopes, pmat->isotopes, sizeof(unsigned)*numIso, hipMemcpyHostToDevice));

  gpuErrchk(hipMemcpy(offsets+numMat, &numIso, sizeof(unsigned),hipMemcpyHostToDevice)); 
}

material::~material(){
}

void material::release_pointer(){
  gpuErrchk(hipFree(offsets));
  gpuErrchk(hipFree(N_tot));
  gpuErrchk(hipFree(densities));
  gpuErrchk(hipFree(isotopes));
}

