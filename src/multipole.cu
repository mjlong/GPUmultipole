#include "hip/hip_runtime.h"
#include "multipole.h"

/*multipole::multipole(){
  printf("Hello, i'm constructing\n");
}*/

#if defined(__QUICKWG)
multipole::multipole(struct multipoledata *data, int numIso, CComplex<CMPTYPE>* wtable){
#else 
multipole::multipole(struct multipoledata *data, int numIso){
#endif //Only __QUICWG needs a global wtable

  size_t size;
  int i;
  int * h_offset = (int*)malloc(sizeof(int)*numIso);
  int * h_size   = (int*)malloc(sizeof(int)*numIso);
  // allocate array of offsets
  size = sizeof(int)*numIso*NUMOFFS;
  gpuErrchk(hipMalloc((void**)&offsets, size));
  /*
    allocate and assign integers
  */
  size = sizeof(unsigned);
  gpuErrchk(hipMalloc((void**)&dev_integers, DEVINTS*size*numIso));
  for(i=0;i<numIso;i++){
    hipMemcpy(dev_integers+i*DEVINTS+MODE,        &(data[i].mode), size, hipMemcpyHostToDevice);
    hipMemcpy(dev_integers+i*DEVINTS+FITORDER,    &(data[i].fitorder), size, hipMemcpyHostToDevice);
    hipMemcpy(dev_integers+i*DEVINTS+NUML,        &(data[i].numL), size, hipMemcpyHostToDevice);
    hipMemcpy(dev_integers+i*DEVINTS+FISSIONABLE, &(data[i].fissionable), size, hipMemcpyHostToDevice);
    hipMemcpy(dev_integers+i*DEVINTS+WINDOWS,     &(data[i].windows),size, hipMemcpyHostToDevice);
  }
    hipMalloc((void**)&dev_numIso,sizeof(int));
    hipMemcpy(dev_numIso, &numIso, sizeof(int), hipMemcpyHostToDevice);
  /*
    allocate and assign doubles
  */
  size = sizeof(CMPTYPE);
  hipMalloc((void**)&dev_doubles,  DEVREALS*size*numIso);
  for(i=0;i<numIso;i++){
    hipMemcpy(dev_doubles+i*DEVREALS+STARTE,  &(data[i].startE),  size, hipMemcpyHostToDevice);
    hipMemcpy(dev_doubles+i*DEVREALS+SPACING ,&(data[i].spacing), size, hipMemcpyHostToDevice);
    hipMemcpy(dev_doubles+i*DEVREALS+SQRTAWR, &(data[i].sqrtAWR), size, hipMemcpyHostToDevice);
    hipMemcpy(dev_doubles+i*DEVREALS+ENDE,    &(data[i].endE   ), size, hipMemcpyHostToDevice);
  }

  /*
    allocate and assign arrays
  */
  // mpdata
  size = 0;
  h_offset[0] = 0;
  for(i=0;i<numIso-1;i++){
    h_size[i] = data[i].length*(MP_RF+data[i].fissionable);
    h_offset[i+1]=h_offset[i] + h_size[i];
    h_size[i]*=2*sizeof(CMPTYPE);
    size += h_size[i];
  }
    h_size[i] = data[i].length*(MP_RF+data[i].fissionable)*2*sizeof(CMPTYPE);
    size += h_size[i];
  hipMalloc((void**)&mpdata, size);
  for(i=0;i<numIso;i++){
    hipMemcpy(mpdata + h_offset[i], data[i].mpdata, h_size[i], hipMemcpyHostToDevice);
  }
  hipMemcpy(offsets+PMPDATA*numIso, h_offset, sizeof(int)*numIso, hipMemcpyHostToDevice);

  // l_value
  size = 0;
  h_offset[0] = 0;
  for(i=0;i<numIso-1;i++){
    h_size[i] = data[i].length;
    h_offset[i+1]=h_offset[i] + h_size[i];
    h_size[i]*=sizeof(unsigned);
    size += h_size[i];
  }
    h_size[i] = data[i].length*sizeof(unsigned);
    size += h_size[i];
  hipMalloc((void**)&l_value, size);
  for(i=0;i<numIso;i++){
    hipMemcpy(l_value + h_offset[i], data[i].l_value, h_size[i], hipMemcpyHostToDevice);
  }
  hipMemcpy(offsets+PLVAL*numIso, h_offset, sizeof(int)*numIso, hipMemcpyHostToDevice);

  // pseudo_rho
  size = 0;
  h_offset[0] = 0;
  for(i=0;i<numIso-1;i++){
    h_size[i] = data[i].numL;
    h_offset[i+1]=h_offset[i] + h_size[i];
    h_size[i]*=sizeof(CMPTYPE);
    size += h_size[i];
  }
    h_size[i] = data[i].numL*sizeof(CMPTYPE);
    size += h_size[i];
  hipMalloc((void**)&pseudo_rho, size);
  for(i=0;i<numIso;i++){
    hipMemcpy(pseudo_rho + h_offset[i], data[i].pseudo_rho, h_size[i], hipMemcpyHostToDevice);
  }
  hipMemcpy(offsets+PPRHO*numIso, h_offset, sizeof(int)*numIso, hipMemcpyHostToDevice);

  // w_start and w_end
  size = 0;
  h_offset[0] = 0;
  for(i=0;i<numIso-1;i++){
    h_size[i] = data[i].windows;
    h_offset[i+1]=h_offset[i] + h_size[i];
    h_size[i]*=sizeof(int);
    size += h_size[i];
  }
    h_size[i] = data[i].windows*sizeof(int);
    size += h_size[i];
  hipMalloc((void**)&w_start, size);
  hipMalloc((void**)&w_end,   size);
  for(i=0;i<numIso;i++){
    hipMemcpy(w_start + h_offset[i], data[i].w_start, h_size[i], hipMemcpyHostToDevice);
    hipMemcpy(w_end   + h_offset[i], data[i].w_end,   h_size[i], hipMemcpyHostToDevice);
  }
  hipMemcpy(offsets+PWIND*numIso, h_offset, sizeof(int)*numIso, hipMemcpyHostToDevice);

  // fitT fitA and fitF
  size = 0;
  h_offset[0] = 0;
  for(i=0;i<numIso-1;i++){
    h_size[i] = data[i].windows*(data[i].fitorder+1);
    h_offset[i+1]=h_offset[i] + h_size[i];
    h_size[i]*=sizeof(CMPTYPE);
    size += h_size[i];
  }
    h_size[i] = data[i].windows*(data[i].fitorder+1)*2*sizeof(CMPTYPE);
    size += h_size[i];
  hipMalloc((void**)&fitT, size);
  hipMalloc((void**)&fitA, size);
  hipMemcpy(offsets+PFITS*numIso, h_offset, sizeof(int)*numIso, hipMemcpyHostToDevice);

  CMPTYPE *h_fitT;
  CMPTYPE *h_fitA; 
  unsigned ic, iw;
  for(i=0;i<numIso;i++){
    size = h_size[i]; 
    h_fitT = (CMPTYPE*)malloc(size);
    h_fitA = (CMPTYPE*)malloc(size);
    for(ic=0;ic<=data[i].fitorder;ic++){
      for(iw=0;iw<data[i].windows;iw++){
        h_fitT[ic*data[i].windows+iw] = data[i].fit[findex(iw,ic,FIT_T,data[i].fitorder+1,FIT_F+data[i].fissionable)]; 
     }
    }
    for(ic=0;ic<=data[i].fitorder;ic++){
      for(iw=0;iw<data[i].windows;iw++){
        h_fitA[ic*data[i].windows+iw] = data[i].fit[findex(iw,ic,FIT_A,data[i].fitorder+1,FIT_F+data[i].fissionable)]; 
     }
    }
    gpuErrchk(hipMemcpy(fitT+h_offset[i],h_fitT,size,hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(fitA+h_offset[i],h_fitA,size,hipMemcpyHostToDevice));
    free(h_fitT);
    free(h_fitA);
  }
//fitF 
  size = 0;
  h_offset[0] = 0;
  for(i=0;i<numIso-1;i++){
    h_size[i] = data[i].fissionable? data[i].windows*(data[i].fitorder+1):0;
    h_offset[i+1]=h_offset[i] + h_size[i];
    h_size[i]*=sizeof(CMPTYPE);
    size += h_size[i];
  }
    h_size[i] = data[i].fissionable? data[i].windows*(data[i].fitorder+1)*sizeof(CMPTYPE):0;
    size += h_size[i];
  hipMalloc((void**)&fitF, size);
  hipMemcpy(offsets+PFITF*numIso, h_offset, sizeof(int)*numIso, hipMemcpyHostToDevice);


  CMPTYPE *h_fitF ;
  for(i=0;i<numIso;i++){
    size = h_size[i]; 
    if(0!=size){
    h_fitF = (CMPTYPE*)malloc(size);
    for(ic=0;ic<=data[i].fitorder;ic++){
      for(iw=0;iw<data[i].windows;iw++){
        h_fitF[ic*data[i].windows+iw] = data[i].fit[findex(iw,ic,FIT_F,data[i].fitorder+1,FIT_F+data[i].fissionable)]; 
     }
    }
    gpuErrchk(hipMemcpy(fitF+h_offset[i],h_fitF,size,hipMemcpyHostToDevice));
    free(h_fitF);
    }
  }


  free(h_offset); 
  free(h_size);
 
#if defined(__QUICKWG)
  mtable = wtable;  
#endif
}


multipole::~multipole(){
}

void multipole::release_pointer(){
  gpuErrchk(hipFree(offsets));
  gpuErrchk(hipFree(dev_numIso));
  gpuErrchk(hipFree(dev_integers));
  gpuErrchk(hipFree(dev_doubles));
  gpuErrchk(hipFree(mpdata));
  gpuErrchk(hipFree(l_value));
  gpuErrchk(hipFree(pseudo_rho));
  gpuErrchk(hipFree(w_start));
  gpuErrchk(hipFree(w_end));
  gpuErrchk(hipFree(fitT));
  gpuErrchk(hipFree(fitA));
  gpuErrchk(hipFree(fitF));

#if defined(__QUICKWT)
  unbindwtable();
#endif
}

/*
__device__ void broaden_n_polynomials(double En, double DOPP, double* factors, unsigned n){
//!translated from mit-crpg/WHOPPER
  double sqrtE = sqrt(En);
  double beta  = sqrtE*DOPP;  
  double halfinvDOPP2 = 0.5/(DOPP*DOPP);
  double quarterinvDOPP4 = 0.25/(DOPP*DOPP*DOPP*DOPP);
  double erfBeta, exp_m_beta2;
  if(beta>6.){
    erfBeta = 1.0;
    exp_m_beta2 = 0.0;
  }
  else{
    erfBeta = erf(beta); 
    exp_m_beta2 = exp(-beta*beta);
  }  
  factors[0] = erfBeta/En;
  factors[1] = 1.0/sqrtE;
  factors[2] = erfBeta/En*(halfinvDOPP2+En)+exp_m_beta2/(beta*sqrt(PI));
  if(n>=4){
    factors[3] = 1.0/sqrtE*(En+3.0*halfinvDOPP2);
    for(int i=1;i<n-3;i++){
      factors[i+3] = -factors[i-1]*i*(i+1)*quarterinvDOPP4
                     +factors[i+1]*(En+(3+2*i)*halfinvDOPP2); 
    } 
  }
}
*/
int multipole::findex(int iW, int iC, int type, int orders, int types){
  return iW*orders*types + iC*types + type; 
}

__host__ __device__ int multipole::pindex(int iP, int type){
  return iP*4 + type;
}


