#include "hip/hip_runtime.h"
#include "multipole.h"
#if defined(__QUICKWT)
#if defined(__CFLOAT)
texture<float2, 2> tex_wtable;
#else
texture<double2, 2> tex_wtable;
#endif
#endif

#if defined(__QUICKWG) || defined(__QUICKWT)
multipole::multipole(struct multipoledata data, CComplex<CMPTYPE>* wtable){
#else
multipole::multipole(struct multipoledata data){
#endif
  size_t size;
  /*
    allocate and assign integers
  */
  size = sizeof(unsigned);
  hipMalloc((void**)&dev_integers, 4*size);
  hipMemcpy(dev_integers+MODE,    &(data.mode), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_integers+FITORDER, &(data.fitorder), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_integers+NUML, &(data.numL), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_integers+FISSIONABLE, &(data.fissionable), size, hipMemcpyHostToDevice);

  /*
    allocate and assign doubles
  */
  size = sizeof(CMPTYPE);
  hipMalloc((void**)&dev_doubles,  3*size);
  hipMemcpy(dev_doubles+STARTE, &(data.startE), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_doubles+SPACING,&(data.spacing), size, hipMemcpyHostToDevice);
  hipMemcpy(dev_doubles+SQRTAWR, &(data.sqrtAWR), size, hipMemcpyHostToDevice);

  /*
    allocate and assign arrays
  */
  size = data.length*(MP_RF+data.fissionable)*2*sizeof(CMPTYPE);
  hipMalloc((void**)&mpdata, size);
  hipMemcpy(mpdata, data.mpdata, size, hipMemcpyHostToDevice);

  size = data.length*sizeof(unsigned);
  hipMalloc((void**)&l_value, size);
  hipMemcpy(l_value, data.l_value, size, hipMemcpyHostToDevice);

  size = data.numL*sizeof(CMPTYPE);
  hipMalloc((void**)&pseudo_rho, size);
  hipMemcpy(pseudo_rho, data.pseudo_rho, size, hipMemcpyHostToDevice);


  size = data.windows*sizeof(int);
  hipMalloc((void**)&w_start, size);
  hipMemcpy(w_start, data.w_start, size, hipMemcpyHostToDevice);
  hipMalloc((void**)&w_end, size);
  hipMemcpy(w_end, data.w_end, size, hipMemcpyHostToDevice);

  size = (FIT_F+data.fissionable)*(data.fitorder+1)*data.windows*sizeof(CMPTYPE);
  hipMalloc((void**)&fit, size);
  hipMemcpy(fit, data.fit, size, hipMemcpyHostToDevice);
  
#if defined(__QUICKWT)
  //hipBindTexture(NULL, tex_wtable, wtable, LENGTH*LENGTH*sizeof(CMPTYPE)*2);
#if defined(__CFLOAT)
  hipChannelFormatDesc desc = hipCreateChannelDesc<float2>();
#else
  hipChannelFormatDesc desc = hipCreateChannelDesc<double2>();
#endif
  hipBindTexture2D(NULL, tex_wtable, wtable, desc, LENGTH, LENGTH, sizeof(CMPTYPE)*2*LENGTH);
#endif

#if defined(__QUICKWG)
  table = wtable;  
#endif

}


multipole::~multipole(){
}

void multipole::release_pointer(){
  gpuErrchk(hipFree(dev_integers));
  gpuErrchk(hipFree(dev_doubles));
  gpuErrchk(hipFree(mpdata));
  gpuErrchk(hipFree(l_value));
  gpuErrchk(hipFree(pseudo_rho));
  gpuErrchk(hipFree(w_start));
  gpuErrchk(hipFree(w_end));
  gpuErrchk(hipFree(fit));
#if defined(__QUICKWT)
  hipUnbindTexture(tex_wtable);
#endif
}

// xs eval with MIT Faddeeva()
#if defined(__MITW) || defined(__QUICKW)
__device__  void multipole::xs_eval_fast(CMPTYPE E, CMPTYPE sqrtKT, 
			                 CMPTYPE &sigT, CMPTYPE &sigA, CMPTYPE &sigF){

  // Copy variables to local memory for efficiency 
  unsigned mode        = dev_integers[MODE];
  int    iP, iC, iW, startW, endW;
  CMPTYPE spacing = dev_doubles[SPACING];
  CMPTYPE startE  = dev_doubles[STARTE];
  CMPTYPE sqrtE = sqrt(E);
  if(1==mode)
    iW = (int)((sqrtE - sqrt(startE))/spacing);
  else if(2==mode)
    iW = (int)((log(E) - log(startE))/spacing);
  else
    iW = (int)(( E - startE )/spacing);
  unsigned fitorder    = dev_integers[FITORDER];
  unsigned numL        = dev_integers[NUML];
  unsigned fissionable = dev_integers[FISSIONABLE];

  CMPTYPE sqrtAWR = dev_doubles[SQRTAWR];
  CMPTYPE power, DOPP, DOPP_ECOEF;
  CComplex<CMPTYPE> w_val;

  startW = w_start[iW];
  endW   = w_end[iW];
  CComplex<CMPTYPE> sigT_factor[4];
  //CComplex sigtfactor;
  if(startW <= endW)
    fill_factors(sqrtE,numL,sigT_factor);
  sigT = 0.0;
  sigA = 0.0;
  sigF = 0.0;
  //polynomial fitting

  for (iC=0;iC<=fitorder;iC++){
    power = (CMPTYPE)pow((double)E,(double)iC*0.5-1.0);
    sigT += fit[findex(iW,iC,FIT_T,fitorder+1,2+fissionable)]*power;
    sigA += fit[findex(iW,iC,FIT_A,fitorder+1,2+fissionable)]*power;
    if(MP_FISS == fissionable)
      sigF += fit[findex(iW,iC,FIT_F,fitorder+1,2+fissionable)]*power;
  }

  DOPP = sqrtAWR/sqrtKT;
  DOPP_ECOEF = DOPP/E*sqrt(PI);

  for(iP=startW;iP<=endW;iP++){
    //sigtfactor = sigT_factor[l_value[iP-1]-1];
    //w_val = (sqrtE - mpdata[pindex(iP-1,MP_EA)])*DOPP*DOPP_ECOEF;
		       
#if defined(__QUICKWG) 
    w_val = w_function((sqrtE - mpdata[pindex(iP-1,MP_EA)])*DOPP,table)*DOPP_ECOEF;
#endif

#if defined(__QUICKWC) || defined(__QUICKWT)
    w_val = w_function((sqrtE - mpdata[pindex(iP-1,MP_EA)])*DOPP      )*DOPP_ECOEF;
    // __QUICKWT extern texture in QuickW.cu
    // __QUICKWC extern array   in QuickW.cu
#endif


#if defined(__MITW)
#if defined(__CFLOAT)
    CComplex<float>  zfloat  = mpdata[pindex(iP-1,MP_EA)];
    CComplex<double> zdouble = CComplex<double>((double)real(zfloat),(double)imag(zfloat));
    /*CComplex<double> zdouble = CComplex<double>((double)real(mpdata[pindex(iP-1,MP_EA)]),
						(double)imag(mpdata[pindex(iP-1,MP_EA)]));*/
    zdouble = Faddeeva::w(((double)sqrtE - zdouble)*(double)DOPP);
    w_val = CComplex<float>((float)real(zdouble), (float)imag(zdouble))*DOPP_ECOEF;
#else
    w_val = Faddeeva::w((sqrtE - mpdata[pindex(iP-1,MP_EA)])*DOPP,0.0)*DOPP_ECOEF;
#endif
#endif
    sigT += real(mpdata[pindex(iP-1,MP_RT)]*sigT_factor[l_value[iP-1]-1]*w_val);//sigtfactor);	    
    sigA += real(mpdata[pindex(iP-1,MP_RA)]*w_val);                              
    if(MP_FISS == fissionable)
      sigF += real(mpdata[pindex(iP-1,MP_RF)]*w_val);
  }

}
#endif

//xs eval with Quick W()
/*
__device__  void multipole::xs_eval_fast(CMPTYPE E, CMPTYPE sqrtKT, CComplex<CMPTYPE> *table, 
			                 CMPTYPE &sigT, CMPTYPE &sigA, CMPTYPE &sigF){

  // Copy variables to local memory for efficiency 
  unsigned mode        = dev_integers[MODE];
  int    iP, iC, iW, startW, endW;
  CMPTYPE spacing = dev_doubles[SPACING];
  CMPTYPE startE  = dev_doubles[STARTE];
  CMPTYPE sqrtE = sqrt(E);
  if(1==mode)
    iW = (int)((sqrtE - sqrt(startE))/spacing);
  else if(2==mode)
    iW = (int)((log(E) - log(startE))/spacing);
  else
    iW = (int)(( E - startE )/spacing);
  unsigned fitorder    = dev_integers[FITORDER];
  unsigned numL        = dev_integers[NUML];
  unsigned fissionable = dev_integers[FISSIONABLE];

  CMPTYPE sqrtAWR = dev_doubles[SQRTAWR];
  CMPTYPE power, DOPP, DOPP_ECOEF;
  CComplex<CMPTYPE> w_val;

  startW = w_start[iW];
  endW   = w_end[iW];
  CComplex<CMPTYPE> sigT_factor[4];
  //CComplex sigtfactor;
  if(startW <= endW)
    fill_factors(sqrtE,numL,sigT_factor);
  sigT = 0.0;
  sigA = 0.0;
  sigF = 0.0;
  //polynomial fitting

  for (iC=0;iC<=fitorder;iC++){
    power = (CMPTYPE)pow((double)E,(double)iC*0.5-1.0);
    sigT += fit[findex(iW,iC,FIT_T,fitorder+1,2+fissionable)]*power;
    sigA += fit[findex(iW,iC,FIT_A,fitorder+1,2+fissionable)]*power;
    if(MP_FISS == fissionable)
      sigF += fit[findex(iW,iC,FIT_F,fitorder+1,2+fissionable)]*power;
  }

  DOPP = sqrtAWR/sqrtKT;
  DOPP_ECOEF = DOPP/E*sqrt(PI);

  for(iP=startW;iP<=endW;iP++){
    //sigtfactor = sigT_factor[l_value[iP-1]-1];
    //w_val = (sqrtE - mpdata[pindex(iP-1,MP_EA)])*DOPP*DOPP_ECOEF;
    w_val = w_function((sqrtE - mpdata[pindex(iP-1,MP_EA)])*DOPP,table)*DOPP_ECOEF;
    sigT += real(mpdata[pindex(iP-1,MP_RT)]*sigT_factor[l_value[iP-1]-1]*w_val);//sigtfactor);	    
    sigA += real(mpdata[pindex(iP-1,MP_RA)]*w_val);                              
    if(MP_FISS == fissionable)
      sigF += real(mpdata[pindex(iP-1,MP_RF)]*w_val);
  }

}
*/

//xs eval at 0K
__device__  void multipole::xs_eval_fast(CMPTYPE E,  
                        	 	 CMPTYPE &sigT, CMPTYPE &sigA, CMPTYPE &sigF){

  // Copy variables to local memory for efficiency 
  unsigned mode        = dev_integers[MODE];
  int    iP, iC, iW, startW, endW;
  CMPTYPE spacing = dev_doubles[SPACING];
  CMPTYPE startE  = dev_doubles[STARTE];
  CMPTYPE sqrtE = sqrt(E);
  if(1==mode)
    iW = (int)((sqrtE - sqrt(startE))/spacing);
  else if(2==mode)
    iW = (int)((log(E) - log(startE))/spacing);
  else
    iW = (int)(( E - startE )/spacing);
  unsigned fitorder    = dev_integers[FITORDER];
  unsigned fissionable = dev_integers[FISSIONABLE];
  unsigned numL        = dev_integers[NUML];

  CMPTYPE power;
  CComplex<CMPTYPE> PSIIKI, CDUM1, w_val;

 
  startW = w_start[iW];
  endW   = w_end[iW];
  CComplex<CMPTYPE> sigT_factor[4];
  //CComplex sigtfactor;
  if(startW <= endW)
    fill_factors(sqrtE,numL,sigT_factor);
  sigT = 0.0;
  sigA = 0.0;
  sigF = 0.0;
  //polynomial fitting

  for (iC=0;iC<=fitorder;iC++){
    power = (CMPTYPE)pow((double)E,(double)iC*0.5-1.0);
    sigT += fit[findex(iW,iC,FIT_T,fitorder+1,2+fissionable)]*power;
    sigA += fit[findex(iW,iC,FIT_A,fitorder+1,2+fissionable)]*power;
    if(MP_FISS == fissionable)
      sigF += fit[findex(iW,iC,FIT_F,fitorder+1,2+fissionable)]*power;
  }




  for(iP=startW;iP<=endW;iP++){
    //sigtfactor = sigT_factor[l_value[iP-1]-1];
    PSIIKI = -ONEI/(mpdata[pindex(iP-1,MP_EA)] - sqrtE);
    CDUM1  = PSIIKI / E;
    sigT += real(mpdata[pindex(iP-1,MP_RT)]*CDUM1*sigT_factor[l_value[iP-1]-1]);//sigtfactor);
    sigA += real(mpdata[pindex(iP-1,MP_RA)]*CDUM1);
    if(MP_FISS == fissionable)
      sigF += real(mpdata[pindex(iP-1,MP_RF)]*CDUM1);
  }
  
}

//xs eval at 0k but sampled to sqrtKT
/*
__device__  void multipole::xs_eval_fast(CMPTYPE E, CMPTYPE sqrtKT, CMPTYPE rnd, 
                        	 	 CMPTYPE &sigT, CMPTYPE &sigA, CMPTYPE &sigF){

  // Copy variables to local memory for efficiency 
  unsigned mode        = dev_integers[MODE];
  int    iP, iC, iW, startW, endW;
  CMPTYPE spacing = dev_doubles[SPACING];
  CMPTYPE startE  = dev_doubles[STARTE];
  CMPTYPE sqrtAWR = dev_doubles[SQRTAWR];

  E = E + rnd * sqrtKT * sqrt(0.5) / sqrtAWR;
  CMPTYPE sqrtE = sqrt(E);
  if(1==mode)
    iW = (int)((sqrtE - sqrt(startE))/spacing);
  else if(2==mode)
    iW = (int)((log(E) - log(startE))/spacing);
  else
    iW = (int)(( E - startE )/spacing);
  unsigned fitorder    = dev_integers[FITORDER];
  unsigned fissionable = dev_integers[FISSIONABLE];
  unsigned numL        = dev_integers[NUML];

  CMPTYPE power;
  CComplex<CMPTYPE> PSIIKI, CDUM1, w_val;
 
  startW = w_start[iW];
  endW   = w_end[iW];
  CComplex<CMPTYPE> sigT_factor[4];
  //CComplex sigtfactor;
  if(startW <= endW)
    fill_factors(sqrtE,numL,sigT_factor);
  sigT = 0.0;
  sigA = 0.0;
  sigF = 0.0;
  //polynomial fitting

  for (iC=0;iC<=fitorder;iC++){
    power = (CMPTYPE)pow((double)E,(double)iC*0.5-1.0);
    sigT += fit[findex(iW,iC,FIT_T,fitorder+1,2+fissionable)]*power;
    sigA += fit[findex(iW,iC,FIT_A,fitorder+1,2+fissionable)]*power;
    if(MP_FISS == fissionable)
      sigF += fit[findex(iW,iC,FIT_F,fitorder+1,2+fissionable)]*power;
  }




  for(iP=startW;iP<=endW;iP++){
    //sigtfactor = sigT_factor[l_value[iP-1]-1];
    PSIIKI = -ONEI/(mpdata[pindex(iP-1,MP_EA)] - sqrtE);
    CDUM1  = PSIIKI / E;
    sigT += real(mpdata[pindex(iP-1,MP_RT)]*CDUM1*sigT_factor[l_value[iP-1]-1]);//sigtfactor);
    sigA += real(mpdata[pindex(iP-1,MP_RA)]*CDUM1);
    if(MP_FISS == fissionable)
      sigF += real(mpdata[pindex(iP-1,MP_RF)]*CDUM1);
  }
  
}
*/

__host__ __device__ int multipole::findex(int iW, int iC, int type, int orders, int types){
  return iW*orders*types + iC*types + type; 
}

__host__ __device__ int multipole::pindex(int iP, int type){
  return iP*4 + type;
}

__device__ void multipole::fill_factors(CMPTYPE sqrtE, int numL, 
                                        CComplex<CMPTYPE> *sigT_factor){
  int iL;
  CMPTYPE arg;
  CMPTYPE twophi; 
  
  for(iL = 0; iL<numL; iL++){
    twophi = pseudo_rho[iL] * sqrtE; 
    if(1==iL)
      twophi -= atan(twophi);
    else if(2==iL){
      arg = 3.0*twophi / (3.0 - twophi*twophi);
      twophi -= atan(arg);
    }
    else if(3==iL){
      arg = twophi*(15.0 - twophi*twophi)/(15.0 - 6.0*twophi*twophi);
      twophi -= atan(arg);
    }
    twophi *= 2.0;
    sigT_factor[iL] = CComplex<CMPTYPE>(cos(twophi), -sin(twophi));
  }

}
