#include "hip/hip_runtime.h"
#include "process.h"
void cnt2flux(MemStruct HostMem, unsigned numhis, float dx, unsigned meshes, unsigned nbat){
  int im, ib;
  for(ib=0;ib<nbat;ib++){
    for(im=0;im<meshes;im++){
      HostMem.accmeans[ib*meshes+im] = HostMem.acccnt[ib*meshes+im]/(numhis*dx*(ib+1));
    }
  }

  ib = 0;
  for(im=0;im<meshes;im++)
    HostMem.batchmeans[im] = HostMem.accmeans[im];

  for(ib=1;ib<nbat;ib++){
    for(im=0;im<meshes;im++){
      HostMem.batchmeans[ib*meshes+im] = (HostMem.acccnt[ib*meshes+im]-HostMem.acccnt[(ib-1)*meshes+im])/(numhis*dx);
    }
  }

  
}

void getASE(double *accmeans,unsigned meshes, unsigned nbat, unsigned ubat, double ref, double* ASE){
  int ib,im,index;
  for(ib=ubat;ib<nbat;ib++)
    ASE[ib-ubat]=0.0;
  for(ib=ubat;ib<nbat;ib++){
    for(im=0;im<meshes;im++){
      index = ib*meshes+im;
      ASE[ib-ubat] += (accmeans[index]-ref)*(accmeans[index]-ref)/meshes;
    }
  }
}


void getCOR(double *batmeans, unsigned meshes, unsigned nbat, unsigned ubat, unsigned upto,double *COR){
  int im,ib;
  double *batmeans_active = batmeans+ubat*meshes;
  for(im=0;im<meshes;im++){
    for(ib=0;ib<upto;ib++){
      COR[im*upto+ib] = autok(batmeans_active,nbat-ubat,ib+1,meshes,im);
    }
  }
  

}

double autok(double *batmeans, unsigned n, unsigned k, unsigned meshes, unsigned im){
  double sum1=0; double sum2=0; double sum3=0; double sum4=0; double sum5 = 0;
  double xi,xik;
  int ib;
  for(ib=0;ib<n-k;ib++){
    //printf("in autok, ib=%d\n",ib);
    xi = batmeans[ib*meshes+im];
    xik= batmeans[(ib+k)*meshes+im];
    sum1+=xi;
    sum2+=xik;
    sum3+=xi*xik;
    sum4+=xi*xi;
    sum5+=xik*xik;
  }
  
  return ((n-k)*sum3-sum1*sum2)/sqrt(((n-k)*sum4-sum1*sum1)*((n-k)*sum5-sum2*sum2));
}

void fitrho(double* rho, unsigned m, double* rho0, double* q){
  double sum1=0; double sum2=0;
  for(int i=0;i<m;i++){
    sum1 += log(rho[i]);
    sum2 += log(rho[i])*(i+1);
  }
  *rho0 = exp(2.0*(sum1+2.0*m*sum1-3.0*sum2)/(m*(m-1.0)));
  *q = exp((6.0*sum1+6.0*m*sum1-12.0*sum2)/(m*(1.0-m*m)));
  /*
    m = length(ρ);
    A = [m              0.5*m*(m+1);
         0.5*m*(m+1)    1.0/6*m*(m+1)*(2*m+1)];
    c = inv(A)*[sum(log(ρ)), sum(log(ρ).*range(1,m))];
    return exp(c[1]),exp(c[2]); #ρ0, q
  */
}

void fitrho1(double* rho, unsigned m, double* rho0, double* q){
  double sum1 = 0;
  double rho1 = rho[0];
  for(int i=0;i<m;i++)
    sum1 += log(rho[i])*i;
  *q = exp((sum1-m*(m-1.0)*0.5*log(rho1))/(1.0/6*(m-1.0)*m*(2.0*m-1.0)));
  *rho0 = rho1/(*q);
}


void fitall(double *rhos,unsigned upto, unsigned meshes, double *rho0s, double *qs){
  for(int im=0;im<meshes;im++){
    fitrho(rhos+im*upto, upto, rho0s+im, qs+im);
  }
}

void fitall1(double *rhos,unsigned upto, unsigned meshes, double *rho0s, double *qs){
  for(int im=0;im<meshes;im++){
    fitrho1(rhos+im*upto, upto, rho0s+im, qs+im);
  }
}
