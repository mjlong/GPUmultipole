#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "helpers.h"
#include "commonStructs.h"
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include "global.h"

#if defined(__CFLOAT)
#define CMPTYPE float
#define CMPTYPE2 float2
#else
#define CMPTYPE double
#define CMPTYPE2 double2
#endif

using namespace optix;

rtDeclareVariable(float,         var_R1, , );
rtDeclareVariable(float,         var_Hh, , );
rtDeclareVariable(unsigned,      var_num, , );

rtBuffer<unsigned, 1>           mat_offsets;
rtBuffer<unsigned, 1>           mat_isotopes;
rtBuffer<float, 1>              mat_densities;

rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(unsigned int,  only_one_ray_type, , );

rtDeclareVariable(unsigned int, launch_index, rtLaunchIndex, );
rtDeclareVariable(unsigned int, launch_dim,   rtLaunchDim, );

rtCallableProgram(void, xs_eval, (int, CMPTYPE, CMPTYPE, CMPTYPE*,CMPTYPE*,CMPTYPE* ));
rtCallableProgram(void, locate,  (float3, float3, float*, unsigned*, unsigned* ));

RT_PROGRAM void generate_ray()
{
  hiprandState localstate;   
  hiprand_init(0,0,launch_index,&localstate);
  float phi =   2*PI*hiprand_uniform(&localstate);
  float mu  = -1.f+2*hiprand_uniform(&localstate); 
  float3 ray_origin = make_float3(0.5f+0.00*hiprand_uniform(&localstate),
                                  0.5f+0.00*hiprand_uniform(&localstate),
                                  0.5f+0.00*hiprand_uniform(&localstate));
  float3 ray_direction = make_float3(sqrt(1.f-mu*mu)*cos(phi),sqrt(1.f-mu*mu)*sin(phi),mu); 
  float d;
  unsigned icell, imat,isotope,live;

  locate(ray_origin,ray_direction, &d, &imat, &icell);
  imat = imat*(1-(0==icell));
  live = !(0==icell);

  double E=2;
  double sigT,sigA,sigF,
         sigTsum,sigAsum,sigFsum;

  sigTsum = 0;
  sigAsum = 0;
  sigFsum = 0;
  for(isotope=mat_offsets[imat];isotope<mat_offsets[imat+1];isotope++ ){
    xs_eval(mat_isotopes[isotope],E,sqrt(300.*KB),&sigT,&sigA,&sigF); 
    sigTsum += sigT*mat_densities[isotope];
    sigAsum += sigA*mat_densities[isotope];
    sigFsum += sigF*mat_densities[isotope];
  }

  printf("xs_eval(%g)=%g,%g,%g\n",E,sigTsum,sigAsum,sigFsum);

#if defined(__PRINTTRACK__)
  printf("%3d, %3d, %+18.12e,%+18.12e,%+18.12e\n",
         launch_index,icell,ray_origin.x,ray_origin.y,ray_origin.z);
  ray_origin = ray_origin+d*ray_direction;
  printf("%3d, %3d, %+18.12e,%+18.12e,%+18.12e\n",
         launch_index,1111,ray_origin.x,ray_origin.y,ray_origin.z); 
#endif


}

RT_PROGRAM void exception()
{
  const unsigned int code = rtGetExceptionCode();
  rtPrintf( "Caught exception 0x%X at launch index (%d)\n", code, launch_index );
}
