#include "hip/hip_runtime.h"
#include "simulation.h" 

__device__ void launch(NeutronInfoStruct pInfo,int id, CMPTYPE energy){
  pInfo.energy[id] = energy;
}

__global__ void initialize(MemStruct pInfo, CMPTYPE energy){
  //int id = ((blockDim.x*blockDim.y*blockDim.z)*(blockIdx.y*gridDim.x+blockIdx.x)+(blockDim.x*blockDim.y)*threadIdx.z+blockDim.x*threadIdx.y+threadIdx.x);//THREADID;
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  /* Each thread gets same seed, a different sequence number, no offset */
  hiprand_init(1234, id, 0, &(pInfo.nInfo.rndState[id]));
  launch(pInfo.nInfo, id, energy);

  neutron_sample(pInfo.nInfo, id);
  //pInfo[id].energy = energy; //id+1.0; //(id + 1)*1.63*energy*0.001;// 
  pInfo.nInfo.id[id] = id;
  pInfo.tally.cnt[id] = 0;

}

__global__ void update_sort_key(MemStruct DeviceMem, material mat){
  unsigned id = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned isoID = mat.isotopes[mat.offsets[DeviceMem.nInfo.imat[id]]+0];
                                            //matID
  DeviceMem.nInfo.isoenergy[id] = (MAXENERGY*isoID+DeviceMem.nInfo.energy[id])*DeviceMem.nInfo.live[id];
}

__global__ void transport(MemStruct DeviceMem, material mat,unsigned renew){
  int nid = (DeviceMem.nInfo.id[blockDim.x * blockIdx.x + threadIdx.x])%(gridDim.x*blockDim.x);
  unsigned live = DeviceMem.nInfo.live[nid];
  if(live){
    CMPTYPE sigT = DeviceMem.nInfo.sigT[nid];
    float s = -log(hiprand_uniform(&(DeviceMem.nInfo.rndState[nid])))/mat.N_tot[DeviceMem.nInfo.imat[nid]]*sigT;   
    float d = DeviceMem.nInfo.d_closest[nid];
    s = (d<s)*d+(d>=s)*s;
    float mu = DeviceMem.nInfo.dir_polar[nid];
    float phi= DeviceMem.nInfo.dir_azimu[nid];
    DeviceMem.nInfo.pos_x[nid]+=s*sqrt(1-mu*mu)*cos(phi);
    DeviceMem.nInfo.pos_y[nid]+=s*sqrt(1-mu*mu)*sin(phi);
    DeviceMem.nInfo.pos_z[nid]+=s*mu;
  }
  else if(renew){
    neutron_sample(DeviceMem.nInfo,nid);
  }
}

__device__ void neutron_sample(NeutronInfoStruct nInfo, unsigned id){
  nInfo.id[id] += gridDim.x*blockDim.x;
  nInfo.live[id] = 1u;
  hiprandState state = nInfo.rndState[id];
//TODO: source sampling should take settings dependent on geometry
  nInfo.pos_x[id] = 0.5f+hiprand_uniform(&state);
  nInfo.pos_y[id] = 0.5f+hiprand_uniform(&state);
  nInfo.pos_z[id] = 0.5f+hiprand_uniform(&state);
  nInfo.dir_polar[id] = hiprand_uniform(&state)*2-1;
  nInfo.dir_azimu[id] = hiprand_uniform(&state)*PI*2;
  nInfo.rndState[id] = state;
}


__global__ void resurrection(NeutronInfoStruct nInfo){
  //neutron energy has been set in an efficient way after each collison
  //only position and direction are sampled as neutron 
  unsigned nid = (nInfo.id[blockDim.x*blockIdx.x + threadIdx.x])%(gridDim.x*blockDim.x);
  unsigned live = nInfo.live[nid];
  if(!live)
    neutron_sample(nInfo,nid);
}
__global__ void history(material mat, multipole mp_para, MemStruct DeviceMem, unsigned num_src){
  //try others when real simulation structure becomes clear
  int idl = threadIdx.x;
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  int nid = (DeviceMem.nInfo.id[id])%(gridDim.x*blockDim.x);
  unsigned live;
  extern __shared__ unsigned blockTerminated[];
  if(DeviceMem.nInfo.live[nid]){
  unsigned isotopeID;
  CMPTYPE localenergy;
  CMPTYPE rnd;
  CMPTYPE sigTsum, sigAsum, sigFsum, sigT, sigA, sigF;
  sigTsum=0;
  sigAsum=0;
  sigFsum=0;

  /* Copy state to local memory for efficiency */ 
  hiprandState localState = DeviceMem.nInfo.rndState[nid];

  localenergy = DeviceMem.nInfo.energy[nid];
  live = 1u;
  unsigned imat = DeviceMem.nInfo.imat[nid];
  for(isotopeID=mat.offsets[imat];isotopeID<mat.offsets[imat+1];isotopeID++ ){
    rnd = hiprand_uniform(&localState);
    mp_para.xs_eval_fast(mat.isotopes[isotopeID],localenergy, sqrt(300.0*KB), sigT, sigA, sigF);
    sigTsum += sigT*mat.densities[isotopeID];
    sigAsum += sigA*mat.densities[isotopeID];
    sigFsum += sigF*mat.densities[isotopeID];
  }

#if defined(__PRINTTRACK__)
  if(__PRINTTRACK__){
    printf("%7d,%3d,%+.7e, %+.7e, %+.7e, %.14e %.14e %.14e %.14e\n",
            DeviceMem.nInfo.id[id], DeviceMem.nInfo.imat[nid],
            DeviceMem.nInfo.pos_x[nid], DeviceMem.nInfo.pos_y[nid], DeviceMem.nInfo.pos_z[nid],
            localenergy, sigTsum,sigAsum,sigFsum); 
  }
#endif
  localenergy = localenergy * rnd;
  live = (localenergy > 1.0);
  DeviceMem.nInfo.live[nid] = live;  
  //energy can be updated efficiently here, live state is upated after sorting
  localenergy = localenergy*live + STARTENE*(1u - live);
  //terminated += !live;

  blockTerminated[idl] = !live;
  
  /*Note: from now on, live does not indicate neutron but thread active */
  //blockActive[threadIdx.x] = (((terminated*2)*blockDim.x*gridDim.x + atomicAdd(Info.num_terminated_neutrons, terminated)) < num_src);
  //atomicAdd(Info.num_terminated_neutrons,!live);
  //Info.thread_active[id] =  blockDim.x*gridDim.x + *Info.num_terminated_neutrons < num_src;
  /* Copy state back to global memory */ 
  DeviceMem.nInfo.rndState[nid] = localState; 
  DeviceMem.nInfo.energy[nid] = localenergy;
  DeviceMem.nInfo.sigT[nid]=sigTsum;
  DeviceMem.nInfo.sigA[nid]=sigAsum;
  DeviceMem.nInfo.sigF[nid]=sigFsum;
  DeviceMem.tally.cnt[nid] += 1; 
  }//end if live

  else{
    blockTerminated[idl] = 0;
    //those old unlive neutrons must not be counted again
    //so, 0 instead of !live is used 
  }
  __syncthreads();
  live = blockDim.x>>1;
  while(live){
    if(idl<live)
      blockTerminated[idl] += blockTerminated[idl+live];
    __syncthreads();
    live>>=1;
  }
  if(0==idl){
    //reduction scheme depends on tally type
    //following is to count moderation times
    DeviceMem.block_terminated_neutrons[blockIdx.x] = blockTerminated[0];
  }

}

__global__ void reduce_sum_plus(unsigned *threadcnt, unsigned* cnt){
// reduce threadcnt[] to cnt[], cnt is updated by self increase
// this is used to count terminated neurtons
  /*reduce tally*/
  /*TODO:
    alternatives:
    1. only count for a block, saving global memory (acceess)
    2. count for each thread, saving time in thread wait
  */
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned idl = threadIdx.x;
  extern __shared__ unsigned shared[];
  //size of shared[] is given as 3rd parameter while launching the kernel
  int i;
  shared[idl] = threadcnt[id];
  __syncthreads();
  i = blockDim.x>>1;
  while(i){
    if(idl<i)
      shared[idl] += shared[idl+i];
    __syncthreads();
    i=i>>1;
  }
  if(0==idl){
    //reduction scheme depends on tally type
    //following is to count moderation times
    cnt[blockIdx.x] += shared[0];
  }
}

__global__ void reduce_sum_equal(unsigned* thread_active, unsigned* active){
// reduce thread_active to active, active is updated without history
// this is used to count number of "live" threads
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned idl = threadIdx.x;
  extern __shared__ unsigned shared[];
  //size of shared[] is given as 3rd parameter while launching the kernel
  int i;
  shared[idl] = thread_active[id]; 
  __syncthreads();
  i = blockDim.x>>1;
  while(i){
    if(idl<i)
      shared[idl] += shared[idl+i];
    __syncthreads();
    i=i>>1;
  }
  if(0==idl){
    active[blockIdx.x] = shared[0];
  }
}
