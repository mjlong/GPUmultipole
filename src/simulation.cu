#include "hip/hip_runtime.h"
#include "simulation.h" 

__global__ void initialize(MemStruct pInfo,float width){
  //int id = ((blockDim.x*blockDim.y*blockDim.z)*(blockIdx.y*gridDim.x+blockIdx.x)+(blockDim.x*blockDim.y)*threadIdx.z+blockDim.x*threadIdx.y+threadIdx.x);//THREADID;
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  /* Each thread gets same seed, a different sequence number, no offset */
  hiprand_init(1234, id, 0, &(pInfo.nInfo.rndState[id]));

  neutron_sample(pInfo.nInfo, id,width);
  pInfo.nInfo.id[id] = id;
  pInfo.tally.cnt[id] = 0;
}

__device__ void neutron_sample(NeutronInfoStruct nInfo, unsigned id,float width){
  nInfo.live[id] = 1u;
  hiprandState state = nInfo.rndState[id];
  //TODO: source sampling should take settings dependent on geometry
  nInfo.pos_x[id] = width*hiprand_uniform(&state);
  nInfo.pos_y[id] = 0.5f+0.00*hiprand_uniform(&state);
  nInfo.pos_z[id] = 0.5f+0.00*hiprand_uniform(&state);
  nInfo.dir_polar[id] = hiprand_uniform(&state)*2-1;
  nInfo.dir_azimu[id] = hiprand_uniform(&state)*PI*2;
  nInfo.energy[id] = STARTENE;
  nInfo.rndState[id] = state;
}

__device__ unsigned notleak(float x,float a){
  return (x>=0)&&(x<=a);
}

__global__ void history(MemStruct DeviceMem, unsigned num_src,unsigned active,unsigned banksize){
  float width = DeviceMem.wdspp[0];
  float dx = DeviceMem.wdspp[1];
  float mfp = DeviceMem.wdspp[2];
  float Ps = 1-(DeviceMem.wdspp[3]+DeviceMem.wdspp[4]);
  float Pc = Ps+DeviceMem.wdspp[4];
  float s;
  //try others when real simulation structure becomes clear
  int idl = threadIdx.x;
  //id is the thread index
  //nid is the sampled index to get neutron position
  //in this scheme, normalization is realized by forcefully 
  //select gridsize neutrons from banksize neutrons
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  hiprandState localState = DeviceMem.nInfo.rndState[id];
  int nid = int(hiprand_uniform(&localState)*banksize);
  extern __shared__ unsigned blockTerminated[];

  CMPTYPE rnd;
  float x = DeviceMem.nInfo.pos_x[nid];


  int dir = 1-2*int((hiprand_uniform(&localState))<=0.5);
  /* Copy state to local memory for efficiency */ 

  int newneu;
  unsigned live=1;
  //printf("[%2d],x=%.5f,pf=%.5f\n",id,DeviceMem.nInfo.pos_x[nid],pf);
  //for(istep=0;istep<devstep;istep++){
  while(live){
    s = -log(hiprand_uniform(&localState))*mfp;
    x = x+s*dir;

    while(!notleak(x,width)){
      x=((1==dir)*2*width+(-1==dir)*0-x);
      dir = -1*dir;
    }
    DeviceMem.tally.cnt[int(x/dx)*gridDim.x*blockDim.x+id]+=1;
    
    rnd = hiprand_uniform(&localState);
    if(rnd<Ps)
      dir = 1-2*int((hiprand_uniform(&localState))<=0.5);
    else{
      live = 0;
      if(rnd>Pc){ //fission
	rnd = hiprand_uniform(&localState);
	//newneu = 2*(rnd<=0.55)+3*(rand>0.55);
	newneu = 1-2*(rnd<=0.55); //-1 --> 2 fission; +1 --> 3 fission
	DeviceMem.nInfo.pos_y[id] = x*newneu;
      }
      else{  //rnd<Pc, capture, nothing to do
	DeviceMem.nInfo.pos_y[id] = 0;
      }
    }//end collision type
  }//end one history
  //}
  blockTerminated[idl] =1;// !live;
  
  /*Note: from now on, live does not indicate neutron but thread active */
  //blockActive[threadIdx.x] = (((terminated*2)*blockDim.x*gridDim.x + atomicAdd(Info.num_terminated_neutrons, terminated)) < num_src);
  //atomicAdd(Info.num_terminated_neutrons,!live);
  //Info.thread_active[id] =  blockDim.x*gridDim.x + *Info.num_terminated_neutrons < num_src;
  /* Copy state back to global memory */ 
  DeviceMem.nInfo.rndState[id] = localState; 

  /*
  else{
    blockTerminated[idl] = active;//0;
    //those old unlive neutrons must not be counted again
    //so, 0 instead of !live is used 
    //it was incorrect, above senario forgot to count leak neutron as terminated
  }
  */
  //TODO: no need of such within block reduction for remaining()
  __syncthreads();
  live = blockDim.x>>1;
  while(live){
    if(idl<live)
      blockTerminated[idl] += blockTerminated[idl+live];
    __syncthreads();
    live>>=1;
  }
  if(0==idl){
    //reduction scheme depends on tally type
    //following is to count moderation times
    DeviceMem.block_terminated_neutrons[blockIdx.x] = blockTerminated[0];
  }

}

__global__ void reduce_sum_plus(unsigned *threadcnt, unsigned* cnt){
// reduce threadcnt[] to cnt[], cnt is updated by self increase
// this is used to count terminated neurtons
  /*reduce tally*/
  /*TODO:
    alternatives:
    1. only count for a block, saving global memory (acceess)
    2. count for each thread, saving time in thread wait
  */
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned idl = threadIdx.x;
  extern __shared__ unsigned shared[];
  //size of shared[] is given as 3rd parameter while launching the kernel
  int i;
  shared[idl] = threadcnt[id];
  __syncthreads();
  i = blockDim.x>>1;
  while(i){
    if(idl<i)
      shared[idl] += shared[idl+i];
    __syncthreads();
    i=i>>1;
  }
  if(0==idl){
    //reduction scheme depends on tally type
    //following is to count moderation times
    cnt[blockIdx.x] += shared[0];
  }
}

__global__ void reduce_sum_equal(unsigned* thread_active, unsigned* active){
// reduce thread_active to active, active is updated without history
// this is used to count number of "live" threads
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned idl = threadIdx.x;
  extern __shared__ unsigned shared[];
  //size of shared[] is given as 3rd parameter while launching the kernel
  int i;
  shared[idl] = thread_active[id]; 
  __syncthreads();
  i = blockDim.x>>1;
  while(i){
    if(idl<i)
      shared[idl] += shared[idl+i];
    __syncthreads();
    i=i>>1;
  }
  if(0==idl){
    active[blockIdx.x] = shared[0];
  }
}
