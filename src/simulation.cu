#include "hip/hip_runtime.h"
#include "simulation.h" 

__global__ void initialize(MemStruct pInfo,float width){
  //int id = ((blockDim.x*blockDim.y*blockDim.z)*(blockIdx.y*gridDim.x+blockIdx.x)+(blockDim.x*blockDim.y)*threadIdx.z+blockDim.x*threadIdx.y+threadIdx.x);//THREADID;
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  /* Each thread gets same seed, a different sequence number, no offset */
  hiprand_init(1234, id, 0, &(pInfo.nInfo.rndState[id]));

  neutron_sample(pInfo.nInfo, id,width);
  pInfo.nInfo.id[id] = id;
  pInfo.tally.cnt[id] = 0;
}

__device__ void neutron_sample(NeutronInfoStruct nInfo, unsigned id,float width){
  nInfo.live[id] = 1u;
  hiprandState state = nInfo.rndState[id];
  //TODO: source sampling should take settings dependent on geometry
  nInfo.pos_x[id] = width*hiprand_uniform(&state);
  nInfo.pos_y[id] = 0.5f+0.00*hiprand_uniform(&state);
  nInfo.pos_z[id] = 0.5f+0.00*hiprand_uniform(&state);
  nInfo.dir_polar[id] = hiprand_uniform(&state)*2-1;
  nInfo.dir_azimu[id] = hiprand_uniform(&state)*PI*2;
  nInfo.energy[id] = STARTENE;
  nInfo.rndState[id] = state;
}


__global__ void history(MemStruct DeviceMem, unsigned num_src,unsigned active,unsigned devstep,float width, float dx){
  //try others when real simulation structure becomes clear
  int idl = threadIdx.x;
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  int nid = id;
  unsigned live;
  extern __shared__ unsigned blockTerminated[];

  CMPTYPE rnd;
  float x = DeviceMem.nInfo.pos_x[nid];
  int dir = 1-2*int(DeviceMem.nInfo.dir_polar[nid]<=0.5);
  /* Copy state to local memory for efficiency */ 
  hiprandState localState = DeviceMem.nInfo.rndState[nid];

  unsigned istep;
  //printf("[%2d],x=%.5f\n",id,DeviceMem.nInfo.pos_x[nid]);
  for(istep=0;istep<devstep;istep++){
    DeviceMem.tally.cnt[int(x/dx)*gridDim.x*blockDim.x+nid]+=1;
    rnd = hiprand_uniform(&localState);
    

    rnd = hiprand_uniform(&localState);

    DeviceMem.nInfo.live[nid] = live;  
    //energy can be updated efficiently here, live state is upated after sorting
    live = rnd<0.5;
    //terminated += !live;

  }
  blockTerminated[idl] = !live;
  
  /*Note: from now on, live does not indicate neutron but thread active */
  //blockActive[threadIdx.x] = (((terminated*2)*blockDim.x*gridDim.x + atomicAdd(Info.num_terminated_neutrons, terminated)) < num_src);
  //atomicAdd(Info.num_terminated_neutrons,!live);
  //Info.thread_active[id] =  blockDim.x*gridDim.x + *Info.num_terminated_neutrons < num_src;
  /* Copy state back to global memory */ 
  DeviceMem.nInfo.rndState[nid] = localState; 

  /*
  else{
    blockTerminated[idl] = active;//0;
    //those old unlive neutrons must not be counted again
    //so, 0 instead of !live is used 
    //it was incorrect, above senario forgot to count leak neutron as terminated
  }
  */
  //TODO: no need of such within block reduction for remaining()
  __syncthreads();
  live = blockDim.x>>1;
  while(live){
    if(idl<live)
      blockTerminated[idl] += blockTerminated[idl+live];
    __syncthreads();
    live>>=1;
  }
  if(0==idl){
    //reduction scheme depends on tally type
    //following is to count moderation times
    DeviceMem.block_terminated_neutrons[blockIdx.x] = blockTerminated[0];
  }

}

__global__ void reduce_sum_plus(unsigned *threadcnt, unsigned* cnt){
// reduce threadcnt[] to cnt[], cnt is updated by self increase
// this is used to count terminated neurtons
  /*reduce tally*/
  /*TODO:
    alternatives:
    1. only count for a block, saving global memory (acceess)
    2. count for each thread, saving time in thread wait
  */
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned idl = threadIdx.x;
  extern __shared__ unsigned shared[];
  //size of shared[] is given as 3rd parameter while launching the kernel
  int i;
  shared[idl] = threadcnt[id];
  __syncthreads();
  i = blockDim.x>>1;
  while(i){
    if(idl<i)
      shared[idl] += shared[idl+i];
    __syncthreads();
    i=i>>1;
  }
  if(0==idl){
    //reduction scheme depends on tally type
    //following is to count moderation times
    cnt[blockIdx.x] += shared[0];
  }
}

__global__ void reduce_sum_equal(unsigned* thread_active, unsigned* active){
// reduce thread_active to active, active is updated without history
// this is used to count number of "live" threads
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned idl = threadIdx.x;
  extern __shared__ unsigned shared[];
  //size of shared[] is given as 3rd parameter while launching the kernel
  int i;
  shared[idl] = thread_active[id]; 
  __syncthreads();
  i = blockDim.x>>1;
  while(i){
    if(idl<i)
      shared[idl] += shared[idl+i];
    __syncthreads();
    i=i>>1;
  }
  if(0==idl){
    active[blockIdx.x] = shared[0];
  }
}
