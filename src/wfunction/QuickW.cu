#include "hip/hip_runtime.h"
#include "QuickW.h"
#if defined(__QUICKWT)
#if defined(__CFLOAT)
texture<float2> tex_wtable;
static __inline__ __device__ CComplex<float> texfetch_complex(texture<float2> t, int i){
  float2 v = tex1Dfetch(t,i);
  return CComplex<float>(v.x, v.y);
  }
#else
texture<int4> tex_wtable;
static __inline__ __device__ CComplex<double> texfetch_complex(texture<int4> t, int i){
  int4 v = tex1Dfetch(t,i);
  return CComplex<double>(__hiloint2double(v.y, v.x),__hiloint2double(v.w,v.z));
}
#endif
/*
#if defined(__CFLOAT)
texture<float2,2> tex_wtable;
static __inline__ __device__ CComplex<float> texfetch_complex(texture<float2,2> t, int i, int j){
  float2 v = tex2D(t, i, j);
  return CComplex<float>(v.x, v.y);
}
#else
texture<int4,2> tex_wtable;
static __inline__ __device__ CComplex<double> texfetch_complex(texture<int4,2> t, int i, int j){
  int4 v = tex2D(t, i, j);
  return CComplex<double>(__hiloint2double(v.y, v.x),__hiloint2double(v.w,v.z));
}
#endif
*/
#endif

#if defined (__QUICKWC)
extern __constant__ CMPTYPE2 constwtable[];
#endif

//TODO: load constants on shared memory
__constant__ CMPTYPE b = 0.275255128608410950901357962647054304017026259671664935783653;
__constant__ CMPTYPE d = 2.724744871391589049098642037352945695982973740328335064216346;
__constant__ CMPTYPE a = 0.512424224754768462984202823134979415014943561548661637413182;
__constant__ CMPTYPE c = 0.051765358792987823963876628425793170829107067780337219430904;

/*===============================================================================  
 INITIALIZE_W_TABULATED calculates the Faddeeva function on a 62 x 62 grid       
 using libcerf which is based on Faddeeva package                                                 
 (http://ab-initio.mit.edu/wiki/index.php/Faddeeva_Package). The implementation                  
 has accuracy of at least 13 significant digits.                                                 
===============================================================================*/     

/*
__device__ void initialize_w_tabulated(CComplex* w_tabulated){
  int i,j;
  double x,y;
  CComplex z;
  for(i=0;i<LENGTH;i++){
    y = WIDTH*(i-1);
    for(j=0;j<LENGTH;j++){
      x = WIDTH*(j-1);
      z = CComplex(x,y);
      w_tabulated[i*LENGTH+j] = Faddeeva::w(z);
    }
  }
  return;
}
*/


__device__ void fill_w_tabulated(CComplex<CMPTYPE>* w_tabulated, int id){
  double x,y;
  CComplex<double> z;
  y = WIDTH*(id/LENGTH-1);
  x = WIDTH*(id%LENGTH-1);
  z = CComplex<double>(x,y);
#if defined(__CFLOAT)
  z=Faddeeva::w(z);
  w_tabulated[id] = CComplex<float>((float)real(z),(float)imag(z));
#else
  w_tabulated[id] = Faddeeva::w(z);
#endif
  return;
}

#if defined(__QUICKWT)
__host__ void bindwtable(CComplex<CMPTYPE>* wtable){
  hipBindTexture(NULL, tex_wtable, wtable, LENGTH*LENGTH*sizeof(CMPTYPE)*2);
  //hipChannelFormatDesc desc = hipCreateChannelDesc<CMPTYPE2>();
  //hipBindTexture2D(NULL, tex_wtable, wtable, desc, LENGTH, LENGTH, sizeof(CMPTYPE)*2*LENGTH);
}

__host__ void unbindwtable(){
  hipUnbindTexture(tex_wtable);
}
#endif 

/*===============================================================================                   
 W_FUNCTION calculates the Faddeeva function, also known as the complex                
 probability integral, for complex arguments. For |z| < 6, it uses a six-point 
 interpolation scheme based on pre-tabulated data that is accurate to          
 O(10^-3). For |z| > 6, it uses a three-term asymptotic approximation that is                 
 accurate to O(10^-6).                           
===============================================================================*/ 
#if defined(__QUICKWT) 
__device__ CComplex<CMPTYPE> w_function(CComplex<CMPTYPE> z){
  CComplex<CMPTYPE> w;
  if(abs(Norm(z)) < 6.0){
    CMPTYPE p = 10.0*abs(real(z));
    CMPTYPE q = 10.0*imag(z);
    int     l = (int)p + 1;
    int     m = (int)q + 1;
    p = p - (int)p;
    q = q - (int)q;
    CMPTYPE pp = p*p;
    CMPTYPE qq = q*q;
    CMPTYPE pq = p*q;
    //**********************************************************************
    //CComplex<CMPTYPE> w1,w2,w3,w4,w5,w6;
    //**********************************************************************
    //failed test of binding 1d array to 2d texture
    /*
    w1 = texfetch_complex(tex_wtable, m-1, l  );
    w2 = texfetch_complex(tex_wtable, m  , l-1);
    w3 = texfetch_complex(tex_wtable, m  , l  );
    w4 = texfetch_complex(tex_wtable, m  , l+1);
    w5 = texfetch_complex(tex_wtable, m+1, l  );
    w6 = texfetch_complex(tex_wtable, m+1, l+1);
    */
    /*
    w1 = texfetch_complex(tex_wtable, (m-1)*LENGTH+l);
    w2 = texfetch_complex(tex_wtable, m*LENGTH + l-1);
    w3 = texfetch_complex(tex_wtable, m*LENGTH + l  );
    w4 = texfetch_complex(tex_wtable, m*LENGTH + l+1);
    w5 = texfetch_complex(tex_wtable, (m+1)*LENGTH+l);
    w6 = texfetch_complex(tex_wtable, (m+1)*LENGTH+l+1);
    */
    w =  
      (CMPTYPE)0.5*(qq - q)        *texfetch_complex(tex_wtable,(m-1)*LENGTH+l ) + 
      (CMPTYPE)0.5*(pp - p)        *texfetch_complex(tex_wtable,m*LENGTH + l-1 ) +
      (CMPTYPE)(1.0 + pq - pp - qq)*texfetch_complex(tex_wtable,m*LENGTH + l   ) +
      (CMPTYPE)(0.5*(pp + p) - pq) *texfetch_complex(tex_wtable,m*LENGTH + l+1 ) +
      (CMPTYPE)(0.5*(qq + q) - pq) *texfetch_complex(tex_wtable,(m+1)*LENGTH+l ) +
      (CMPTYPE) pq                 *texfetch_complex(tex_wtable,(m+1)*LENGTH+l+1);
      /*
      (CMPTYPE)0.5*(qq - q)        *texfetch_complex(tex_wtable, m-1, l  ) + 
      (CMPTYPE)0.5*(pp - p)        *texfetch_complex(tex_wtable, m  , l-1) +
      (CMPTYPE)(1.0 + pq - pp - qq)*texfetch_complex(tex_wtable, m  , l  ) +
      (CMPTYPE)(0.5*(pp + p) - pq) *texfetch_complex(tex_wtable, m  , l+1) +
      (CMPTYPE)(0.5*(qq + q) - pq) *texfetch_complex(tex_wtable, m+1, l  ) +
      (CMPTYPE) pq                 *texfetch_complex(tex_wtable, m+1, l+1);
      */
      /*
      (CMPTYPE)0.5*(qq - q)        *w1 + 
      (CMPTYPE)0.5*(pp - p)        *w2 +
      (CMPTYPE)(1.0 + pq - pp - qq)*w3 +
      (CMPTYPE)(0.5*(pp + p) - pq) *w4 +
      (CMPTYPE)(0.5*(qq + q) - pq) *w5 +
      (CMPTYPE) pq                 *w6;
      */

    /*    
    if(blockIdx.x==0 && threadIdx.x==18){
      printf("w1=%20.16e + i*%20.16e\n",real(w1),imag(w1));
      printf("w2=%20.16e + i*%20.16e\n",real(w2),imag(w2));
      printf("w3=%20.16e + i*%20.16e\n",real(w3),imag(w3));
      printf("w4=%20.16e + i*%20.16e\n",real(w4),imag(w4));
      printf("w5=%20.16e + i*%20.16e\n",real(w5),imag(w5));
      printf("w6=%20.16e + i*%20.16e\n",real(w6),imag(w6));
      printf("m=%2d, l=%2d\n", m,l);
      printf("p=%16.12e, q=%16.12e, pp=%16.12e, qq=%16.12e, pq=%16.12e\n",p,q,pp,qq,pq);
    }
    */
    if(real(z)<0)
      w = Conjugate(w);
  }
  else
    w = ONEI * z * (a/(z*z - b) + c/(z*z - d));
  return w;
  
}
#endif

#if defined(__QUICKWG) 
__device__ CComplex<CMPTYPE> w_function(CComplex<CMPTYPE> z, CComplex<CMPTYPE>* w_tabulated){
  CMPTYPE  p;           // interpolation factor on real axis                                   
  CMPTYPE  q;           // interpolation factor on imaginary axis                                  
  CMPTYPE  pp, qq, pq;  // products of p and q                                         

  int l;               //interpolation index for real axis
  int m;               //interpolation index for imaginary axis
  
  CComplex<CMPTYPE> w;
  
  if(abs(Norm(z)) < 6.0){
    //CComplex<CMPTYPE> w1,w2,w3,w4,w5,w6;
    // Use interpolation for |z| < 6. The interpolation scheme uses a bivariate         
    // six-point quadrature described in Abramowitz and Stegun 25.2.67. This          
    // interpolation is accurate to O(h^3) = O(10^-3).                           
    //                          
    //     l-1  l  l+1            
    // m+1      +   +                                   
    //          |                         
    // m    +---+---+                         
    //          |                   
    // m-1      +        

    // Determine indices on grid for interpolation and interpolation factors --
    // note that in previous implementations it was necessary to add/subtract
    // two in places because of the indexing on the tabulated function. Because
    // w_tabulated is indexed from -1 to 60, we don't need to do that here

    p = 10.0*abs(real(z));
    q = 10.0*imag(z);
    l = (int)p;
    m = (int)q;
    p = p - l;
    q = q - m;

    //Calculate products
    pp = p*p;
    qq = q*q;
    pq = p*q;

    // Use six-point interpolation to calculate real and imaginary parts
    l++;
    m++;
    /*
    w1 = w_tabulated[(m-1)*LENGTH+l];
    w2 = w_tabulated[m*LENGTH + l-1];
    w3 = w_tabulated[m*LENGTH + l  ];
    w4 = w_tabulated[m*LENGTH + l+1];
    w5 = w_tabulated[(m+1)*LENGTH+l];
    w6 = w_tabulated[(m+1)*LENGTH+l+1];
    */
    w = 
      (CMPTYPE)0.5*(qq - q)        *w_tabulated[(m-1)*LENGTH+l] + 
      (CMPTYPE)0.5*(pp - p)        *w_tabulated[m*LENGTH + l-1] +
      (CMPTYPE)(1.0 + pq - pp - qq)*w_tabulated[m*LENGTH + l  ] +
      (CMPTYPE)(0.5*(pp + p) - pq) *w_tabulated[m*LENGTH + l+1] +
      (CMPTYPE)(0.5*(qq + q) - pq) *w_tabulated[(m+1)*LENGTH+l] +
      (CMPTYPE) pq                 *w_tabulated[(m+1)*LENGTH+l+1];
    /*
    w =  
      (CMPTYPE)0.5*(qq - q)        *w1+
      (CMPTYPE)0.5*(pp - p)        *w2+
      (CMPTYPE)(1.0 + pq - pp - qq)*w3+
      (CMPTYPE)(0.5*(pp + p) - pq) *w4+
      (CMPTYPE)(0.5*(qq + q) - pq) *w5+
      (CMPTYPE) pq                 *w6; 
    if(blockIdx.x==0 && threadIdx.x==18){
      printf("w1=%20.16e + i*%20.16e\n",real(w1),imag(w1));
      printf("w2=%20.16e + i*%20.16e\n",real(w2),imag(w2));
      printf("w3=%20.16e + i*%20.16e\n",real(w3),imag(w3));
      printf("w4=%20.16e + i*%20.16e\n",real(w4),imag(w4));
      printf("w5=%20.16e + i*%20.16e\n",real(w5),imag(w5));
      printf("w6=%20.16e + i*%20.16e\n",real(w6),imag(w6));
      printf("m=%2d, l=%2d\n", m,l);
      printf("p=%16.12e, q=%16.12e, pp=%16.12e, qq=%16.12e, pq=%16.12e\n",p,q,pp,qq,pq);
    }
    */
    if(real(z)<0) 
      w = Conjugate(w);
  }
  else
    w = ONEI * z * (a/(z*z - b) + c/(z*z - d));

  return w;
}
#endif

#if defined(__QUICKWC)
__device__ CComplex<CMPTYPE> w_function(CComplex<CMPTYPE> z){
  CMPTYPE  p;           // interpolation factor on real axis                                   
  CMPTYPE  q;           // interpolation factor on imaginary axis                                  
  CMPTYPE  pp, qq, pq;  // products of p and q                                         

  int l;               //interpolation index for real axis
  int m;               //interpolation index for imaginary axis
  
  CComplex<CMPTYPE> w;
  
  if(abs(Norm(z)) < 6.0){
    //CComplex<CMPTYPE> w1,w2,w3,w4,w5,w6;
    // Use interpolation for |z| < 6. The interpolation scheme uses a bivariate         
    // six-point quadrature described in Abramowitz and Stegun 25.2.67. This          
    // interpolation is accurate to O(h^3) = O(10^-3).                           
    //                          
    //     l-1  l  l+1            
    // m+1      +   +                                   
    //          |                         
    // m    +---+---+                         
    //          |                   
    // m-1      +        

    // Determine indices on grid for interpolation and interpolation factors --
    // note that in previous implementations it was necessary to add/subtract
    // two in places because of the indexing on the tabulated function. Because
    // w_tabulated is indexed from -1 to 60, we don't need to do that here

    p = 10.0*abs(real(z));
    q = 10.0*imag(z);
    l = (int)p;
    m = (int)q;
    p = p - l;
    q = q - m;

    //Calculate products
    pp = p*p;
    qq = q*q;
    pq = p*q;

    // Use six-point interpolation to calculate real and imaginary parts
    l++;
    m++;
    /*
    w1 = CComplex<CMPTYPE>(constwtable[(m-1)*LENGTH+l].x,constwtable[(m-1)*LENGTH+l].y);
    w2 = CComplex<CMPTYPE>(constwtable[m*LENGTH + l-1].x,constwtable[m*LENGTH + l-1].y);
    w3 = CComplex<CMPTYPE>(constwtable[m*LENGTH + l  ].x,constwtable[m*LENGTH + l  ].y);
    w4 = CComplex<CMPTYPE>(constwtable[m*LENGTH + l+1].x,constwtable[m*LENGTH + l+1].y);
    w5 = CComplex<CMPTYPE>(constwtable[(m+1)*LENGTH+l].x,constwtable[(m+1)*LENGTH+l].y);
    w6 = CComplex<CMPTYPE>(constwtable[(m+1)*LENGTH+l+1].x,constwtable[(m+1)*LENGTH+l+1].y);
    */
    w = 
      (CMPTYPE)0.5*(qq - q)        *CComplex<CMPTYPE>(constwtable[(m-1)*LENGTH+l].x,constwtable[(m-1)*LENGTH+l].y)+	  
      (CMPTYPE)0.5*(pp - p)        *CComplex<CMPTYPE>(constwtable[m*LENGTH + l-1].x,constwtable[m*LENGTH + l-1].y)+	  
      (CMPTYPE)(1.0 + pq - pp - qq)*CComplex<CMPTYPE>(constwtable[m*LENGTH + l  ].x,constwtable[m*LENGTH + l  ].y)+	  
      (CMPTYPE)(0.5*(pp + p) - pq) *CComplex<CMPTYPE>(constwtable[m*LENGTH + l+1].x,constwtable[m*LENGTH + l+1].y)+	  
      (CMPTYPE)(0.5*(qq + q) - pq) *CComplex<CMPTYPE>(constwtable[(m+1)*LENGTH+l].x,constwtable[(m+1)*LENGTH+l].y)+	  
      (CMPTYPE) pq                 *CComplex<CMPTYPE>(constwtable[(m+1)*LENGTH+l+1].x,constwtable[(m+1)*LENGTH+l+1].y);
    /*
     w=
      (CMPTYPE)0.5*(qq - q)        *w1+	  
      (CMPTYPE)0.5*(pp - p)        *w2+	  
      (CMPTYPE)(1.0 + pq - pp - qq)*w3+	  
      (CMPTYPE)(0.5*(pp + p) - pq) *w4+	  
      (CMPTYPE)(0.5*(qq + q) - pq) *w5+	  
      (CMPTYPE) pq                 *w6;
    if(blockIdx.x==0 && threadIdx.x==18){
      printf("w1=%20.16e + i*%20.16e\n",real(w1),imag(w1));
      printf("w2=%20.16e + i*%20.16e\n",real(w2),imag(w2));
      printf("w3=%20.16e + i*%20.16e\n",real(w3),imag(w3));
      printf("w4=%20.16e + i*%20.16e\n",real(w4),imag(w4));
      printf("w5=%20.16e + i*%20.16e\n",real(w5),imag(w5));
      printf("w6=%20.16e + i*%20.16e\n",real(w6),imag(w6));
      printf("m=%2d, l=%2d\n", m,l);
      printf("p=%16.12e, q=%16.12e, pp=%16.12e, qq=%16.12e, pq=%16.12e\n",p,q,pp,qq,pq);
    }
    */
    if(real(z)<0) 
      w = Conjugate(w);
  }
  else
    w = ONEI * z * (a/(z*z - b) + c/(z*z - d));

  return w;
}
#endif

