#include "hip/hip_runtime.h"
#include "fourierw.h"

__device__ CComplex<CMPTYPE> w_function(CComplex<CMPTYPE> z){
  CComplex<CMPTYPE> w;
  CComplex<CMPTYPE> A = taom*z;
  CComplex<CMPTYPE> B = exp(ONEI*A);
  CComplex<CMPTYPE> C = A*A;
  w = CComplex<CMPTYPE>((CMPTYPE)0.0,(CMPTYPE)0.0); 
  for(int n=1;n<M;n++){
    //w = w + exp(-n*n*PI*PI/(taom*taom))*( (CMPTYPE)(((n<<31)>>31)|0x00000001)*B - (CMPTYPE)1.0)/(n*n*PI*PI - C);
    w = w + (CMPTYPE)exp(-n*n*PI*PI/(taom*taom))*( -B - (CMPTYPE)1.0)/((CMPTYPE)(n*n*PI*PI) - C);
    n++;
    w = w + (CMPTYPE)exp(-n*n*PI*PI/(taom*taom))*(  B - (CMPTYPE)1.0)/((CMPTYPE)(n*n*PI*PI) - C);
  }
  w = w + (CMPTYPE)exp(-23*23*PI*PI/(taom*taom))*( -B - (CMPTYPE)1.0)/((CMPTYPE)(23*23*PI*PI) - C);

  w = w*2.0*A;
  w = w + ((CMPTYPE)1.0-B)/(taom*z);
  w = ONEI*w;
  return w;
}

