#include "hip/hip_runtime.h"
#include "fourierw.h"

extern __constant__ CMPTYPE a[];

__global__ void fill_a(CMPTYPE *a){
  // since n is supposed to be 1,2,3,...,23
  // by default dimGrid=(1,1,1)
  int n = threadIdx.x+1;
  a[n] = exp(-n*n*PI*PI/(taom*taom));
}

__device__ CComplex<CMPTYPE> w_function(CComplex<CMPTYPE> z){
  CComplex<CMPTYPE> w;
  CComplex<CMPTYPE> A = taom*z;
  CComplex<CMPTYPE> B = exp(ONEI*A);
  CComplex<CMPTYPE> C = A*A;
  w = CComplex<CMPTYPE>((CMPTYPE)0.0,(CMPTYPE)0.0); 
  for(int n=1;n<M;n++){
    //w = w + exp(-n*n*PI*PI/(taom*taom))*( (CMPTYPE)(((n<<31)>>31)|0x00000001)*B - (CMPTYPE)1.0)/(n*n*PI*PI - C);
    w = w + a[n]*( -B - (CMPTYPE)1.0)/((CMPTYPE)(n*n*PI*PI) - C);
    n++;
    w = w + a[n]*(  B - (CMPTYPE)1.0)/((CMPTYPE)(n*n*PI*PI) - C);
  }
  w = w + a[23] *( -B - (CMPTYPE)1.0)/((CMPTYPE)(23*23*PI*PI) - C);

  w = w*2.0*A;
  w = w + ((CMPTYPE)1.0-B)/(taom*z);
  w = ONEI*w;
  return w;
}

